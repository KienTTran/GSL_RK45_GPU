#include "hip/hip_runtime.h"
#include "gpu_rk45.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

GPU_RK45::GPU_RK45(){
    params = new GPU_Parameters();
}

GPU_RK45::~GPU_RK45(){
    params = nullptr;
}

void GPU_RK45::setParameters(GPU_Parameters* params_) {
    params = &(*params_);
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPU Assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
__device__
double seasonal_transmission_factor(GPU_Parameters* gpu_params, double t)
{
    /*


        We're gonna make this thing go for 40 years. 30 years of burn in and 10 years of real modeling.
        We're creating a "10-year model cycle" and need the code below to find a time point's "place" in the "cycle"
        modulus (denoted with % in C++) only works with integers, so need the acrobatics below

     */

    // This is some code that's needed to create the 10-year "cycles" in transmission.

    if(gpu_params->phis_d_length == 0){
        return 1.0;
    }

    int x = (int)t; // This is now to turn a double into an integer
    double remainder = t - (double)x;
    int xx = x % 3650; // int xx = x % NUMDAYSOUTPUT;
    double yy = (double)xx + remainder;
    // put yy into the sine function, let it return the beta value
    t = yy;
    double sine_function_value = 0.0;

    for(int i=0; i<gpu_params->phis_d_length; i++)
    {
        if( std::fabs( t - gpu_params->phis_d[i] ) < (gpu_params->v_d[gpu_params->i_epidur] / 2))
        {
            // sine_function_value = sin( 2.0 * 3.141592653589793238 * (phis[i]-t+91.25) / 365.0 );
            sine_function_value = std::sin( 2.0 * 3.141592653589793238 * (gpu_params->phis_d[i] - t +(gpu_params->v_d[gpu_params->i_epidur] / 2)) / (gpu_params->v_d[gpu_params->i_epidur] * 2));
//            printf("      in loop %1.3f %d  %1.3f %1.3f\n", t, i, gpu_params->phis_d[i], sine_function_value );
        }
    }
//    printf("    %f sine_function_value %1.3f\n",t,sine_function_value);
//    printf("    %f return %1.3f\n",t,1.0 + v[i_amp] * sine_function_value);
    return 1.0 + gpu_params->v_d[gpu_params->i_amp] * sine_function_value;
}

__device__
double seasonal_transmission_factor(GPU_Parameters* gpu_params, int day)
{
    /*


        We're gonna make this thing go for 40 years. 30 years of burn in and 10 years of real modeling.
        We're creating a "10-year model cycle" and need the code below to find a time point's "place" in the "cycle"
        modulus (denoted with % in C++) only works with integers, so need the acrobatics below

     */

    // This is some code that's needed to create the 10-year "cycles" in transmission.

    if(gpu_params->phis_d_length == 0){
        return 1.0;
    }

    double sine_function_value = 0.0;

    for(int i=0; i<gpu_params->phis_d_length; i++)
    {
        if( std::fabs( day - gpu_params->phis_d[i] ) < (gpu_params->v_d[gpu_params->i_epidur] / 2))
        {
            // sine_function_value = sin( 2.0 * 3.141592653589793238 * (phis[i]-t+91.25) / 365.0 );
            sine_function_value = std::sin( 2.0 * 3.141592653589793238 * (gpu_params->phis_d[i] - day +(gpu_params->v_d[gpu_params->i_epidur] / 2)) / (gpu_params->v_d[gpu_params->i_epidur] * 2));
//            printf("      in loop %1.3f %d  %1.3f %1.3f\n", t, i, gpu_params->phis_d[i], sine_function_value );
        }
    }
//    printf("    %f sine_function_value %1.3f\n",t,sine_function_value);
//    printf("    %f return %1.3f\n",t,1.0 + v[i_amp] * sine_function_value);
    return 1.0 + gpu_params->v_d[gpu_params->i_amp] * sine_function_value;
}

__device__
double gpu_pop_sum( double yy[] )
{
    double sum=0.0;
    for(int i=0; i<DIM; i++) sum += yy[i];

    for(int i=STARTJ; i<STARTJ+NUMLOC*NUMSEROTYPES; i++) sum -= yy[i];
    return sum;
}

__device__
void rk45_gpu_adjust_h(double y[], double y_err[], double dydt_out[],
                       double* h, double h_0, int* adjustment_out, int final_step,
                       double r[], double D0[], double r_max[],
                       const int index, GPU_Parameters* params)
                       {
    /* adaptive adjustment */
    /* Available control object constructors.
     *
     * The standard control object is a four parameter heuristic
     * defined as follows:
     *    D0 = eps_abs + eps_rel * (a_y |y| + a_dydt h |y'|)
     *    D1 = |yerr|
     *    q  = consistency order of method (q=4 for 4(5) embedded RK)
     *    S  = safety factor (0.9 say)
     *
     *                      /  (D0/D1)^(1/(q+1))  D0 >= D1
     *    h_NEW = S h_OLD * |
     *                      \  (D0/D1)^(1/q)      D0 < D1
     *
     * This encompasses all the standard error scaling methods.
     *
     * The y method is the standard method with a_y=1, a_dydt=0.
     * The yp method is the standard method with a_y=0, a_dydt=1.
     */
    static double eps_abs = 1e-6;
    static double eps_rel = 0.0;
    static double a_y = 1.0;
    static double a_dydt = 0.0;
    static unsigned int ord = 5;
    const double S = 0.9;
    double h_old;
    if(final_step){
        h_old = h_0;
    }
    else{
        h_old = *h;
    }
//    if(index == 0 || index == params->dimension - 1)
//    {
//        printf("  [gpu_adjust_h] Index = %d D0 = %f r = %f\n", index, D0, r);
//        printf("    IN y[%d] = %f\n",index,y[index]);
//        printf("    IN y_err[%d] = %f\n",index,y_err[index]);
//        printf("    IN dydt_out[%d] = %f\n",index,dydt_out[index]);
//        printf("    eps_rel[%d] = %f\n",index,eps_rel);
//        printf("    a_y[%d] = %f\n",index,a_y);
//        printf("    h_old[%d] = %f\n",index,h_old);
//        printf("    fabs((h_old) * dydt_out_d[%d])) = %f\n",index,fabs((h_old) * dydt_out[index]));
//        printf("    eps_abs[%d] = %f\n",index,eps_abs);
//        printf("    D0[%d] = %f\n",index,D0);
//        printf("    r[%d] = %f\n",index,r);
//    }

    //finding r_max
    r_max[index] = 2.2250738585072014e-308;
    D0[index] = eps_rel * (a_y * fabs(y[index]) + a_dydt * fabs((h_old) * dydt_out[index])) + eps_abs;
    r[index]  = fabs(y_err[index]) / fabs(D0[index]);

//    if(index == 0 || index == params->dimension - 1) {
//        printf("      index = %d D0[%d] = %f\n",index,index,D0[index]);
//        printf("      index = %d r[%d] = %f\n",index,index,r[index]);
//    }

    cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
    for (int i = 0; i < params->dimension; i++)
    {
//        if(index == 0 || index == params->dimension - 1)
//        if(r[i] != 0)
//        {
//            printf("      compare r[%d] = %f with r_max[%d] = %f\n",i,r[i],index,r_max[index]);
//        }
        r_max[index] = max(r[i], r_max[index]);
    }
    block.sync();

//    if(index == 0 || index == params->dimension - 1) {
//        printf("    Index = %d r_max =  %f\n", index, r_max[index]);
//    }

    if (r_max[index] > 1.1) {
        /* decrease step, no more than factor of 5, but a fraction S more
           than scaling suggests (for better accuracy) */
        double r = S / pow(r_max[index], 1.0 / ord);

        if (r < 0.2)
            r = 0.2;

        *h = r * (h_old);

//        if(index == 0 || index == params->dimension - 1) {
//            printf("    Index = %d decrease by %f, h_old is %f new h is %f\n", index, r, h_old, *h);
//        }
        *adjustment_out = -1;
    } else if (r_max[index] < 0.5) {
        /* increase step, no more than factor of 5 */
        double r = S / pow(r_max[index], 1.0 / (ord + 1.0));

        if (r > 5.0)
            r = 5.0;

        if (r < 1.0)  /* don't allow any decrease caused by S<1 */
            r = 1.0;

        *h = r * (h_old);

//        if(index == 0 || index == params->dimension - 1) {
//            printf("    Index = %d increase by %f, h_old is %f new h is %f\n", index, r, h_old, *h);
//        }
        *adjustment_out = 1;
    } else {
        /* no change */
//        if(index == 0 || index == params->dimension - 1) {
//            printf("    Index = %d no change\n", index);
//        }
        *adjustment_out = 0;
    }
//    if(index == 0 || index == params->dimension - 1) {
//        printf("    [adjust h] index = %d end\n",index);
//    }
    return;
}

__device__
void rk45_gpu_step_apply(double t, double h,
                         double y[], double y_tmp[], double y_err[], double dydt_in[], double dydt_out[],
                         double k1[], double k2[], double k3[], double k4[], double k5[], double k6[],
                         const int index, const int day, GPU_Parameters* params)
{
    static const double ah[] = { 1.0/4.0, 3.0/8.0, 12.0/13.0, 1.0, 1.0/2.0 };
    static const double b3[] = { 3.0/32.0, 9.0/32.0 };
    static const double b4[] = { 1932.0/2197.0, -7200.0/2197.0, 7296.0/2197.0};
    static const double b5[] = { 8341.0/4104.0, -32832.0/4104.0, 29440.0/4104.0, -845.0/4104.0};
    static const double b6[] = { -6080.0/20520.0, 41040.0/20520.0, -28352.0/20520.0, 9295.0/20520.0, -5643.0/20520.0};

    static const double c1 = 902880.0/7618050.0;
    static const double c3 = 3953664.0/7618050.0;
    static const double c4 = 3855735.0/7618050.0;
    static const double c5 = -1371249.0/7618050.0;
    static const double c6 = 277020.0/7618050.0;

    static const double ec[] = { 0.0,
                                 1.0 / 360.0,
                                 0.0,
                                 -128.0 / 4275.0,
                                 -2197.0 / 75240.0,
                                 1.0 / 50.0,
                                 2.0 / 55.0
    };
    y_tmp[index] = 0.0;
    y_err[index] = 0.0;
    dydt_out[index] = 0.0;
    k1[index] = 0.0;
    k2[index] = 0.0;
    k3[index] = 0.0;
    k4[index] = 0.0;
    k5[index] = 0.0;
    k6[index] = 0.0;

//    if(index == 0 || index == params->dimension - 1)
//    if(index == 29)
//    {
//        printf("  [gpu_step_apply] Index = %d t = %f h = %f start\n",index,t,h);
//        printf("    IN y[%d] = %f\n",index,y[index]);
//        printf("    IN y_err[%d] = %f\n",index,y_err[index]);
//        printf("    IN dydt_out[%d] = %f\n",index,dydt_out[index]);
//    }

    /* k1 */
    if (dydt_in != NULL)
    {
        k1[index] = dydt_in[index];
//        if(index == 0 || index == params->dimension - 1) {
//            printf("dydt_in != NULL\n");
//        }
    }
    else {
        gpu_func_test(t, y, k1, index, day, params);
        __syncthreads();
    }
//    if(index == 0 || index == params->dimension - 1) {
//        printf("    k1[%d] = %f\n", index, k1[index]);
//    }
    y_tmp[index] = y[index] +  ah[0] * h * k1[index];
    /* k2 */
    gpu_func_test(t + ah[0] * h, y_tmp, k2, index, day, params);
    __syncthreads();
//    if(index == 0 || index == params->dimension - 1) {
//            printf("    k2[%d] = %f\n",index,k2[index]);
//    }
    y_tmp[index] = y[index] + h * (b3[0] * k1[index] + b3[1] * k2[index]);
    /* k3 */
    gpu_func_test(t + ah[1] * h, y_tmp, k3, index, day, params);
    __syncthreads();
//    if(index == 0 || index == params->dimension - 1) {
//            printf("    k3[%d] = %f\n",index,k3[index]);
//    }
    y_tmp[index] = y[index] + h * (b4[0] * k1[index] + b4[1] * k2[index] + b4[2] * k3[index]);
    /* k4 */
    gpu_func_test(t + ah[2] * h, y_tmp, k4, index, day, params);
    __syncthreads();
//    if(index == 0 || index == params->dimension - 1) {
//            printf("    k4[%d] = %f\n",index,k4[index]);
//    }
    y_tmp[index] = y[index] + h * (b5[0] * k1[index] + b5[1] * k2[index] + b5[2] * k3[index] + b5[3] * k4[index]);
    /* k5 */
    gpu_func_test(t + ah[3] * h, y_tmp, k5, index, day, params);
    __syncthreads();
//    if(index == 0 || index == params->dimension - 1) {
//            printf("    k5[%d] = %f\n",index,k5[index]);
//    }
    y_tmp[index] = y[index] + h * (b6[0] * k1[index] + b6[1] * k2[index] + b6[2] * k3[index] + b6[3] * k4[index] + b6[4] * k5[index]);
    /* k6 */
    gpu_func_test(t + ah[4] * h, y_tmp, k6, index, day, params);
    __syncthreads();
    /* final sum */
//    if(index == 0 || index == params->dimension - 1) {
//        printf("    k6[%d] = %f\n", index, k6[index]);
//    }
    const double d_i = c1 * k1[index] + c3 * k3[index] + c4 * k4[index] + c5 * k5[index] + c6 * k6[index];
    y[index] += h * d_i;
    /* Derivatives at output */
    gpu_func_test(t + h, y, dydt_out, index, day, params);
    __syncthreads();
    /* difference between 4th and 5th order */
    y_err[index] = h * (ec[1] * k1[index] + ec[3] * k3[index] + ec[4] * k4[index] + ec[5] * k5[index] + ec[6] * k6[index]);
    //debug printout
//    if(index == 0 || index == params->dimension - 1)
//    if(index == 29)
//    {
//        printf("    OUT y[%d] = %f\n",index,y[index]);
//        printf("    OUT y_err[%d] = %f\n",index,y_err[index]);
//        printf("    OUT dydt_out[%d] = %f\n",index,dydt_out[index]);
//        printf("  [gpu_step_apply] Index = %d t = %f h = %f end\n",index,t,h);
//    }
    return;
}

__global__
void rk45_gpu_evolve_apply(double t, double t_target, double t_delta, double h, double* y,
                           double* y_output,
                           GPU_Parameters* params){
    __shared__ double r_max[DIM];
    __shared__ double D0[DIM];
    __shared__ double r[DIM];
    __shared__ double y_0[DIM];
    __shared__ double y_tmp[DIM];
    __shared__ double y_err[DIM];
    __shared__ double dydt_in[DIM];
    __shared__ double dydt_out[DIM];
    __shared__ double k1[DIM];
    __shared__ double k2[DIM];
    __shared__ double k3[DIM];
    __shared__ double k4[DIM];
    __shared__ double k5[DIM];
    __shared__ double k6[DIM];

//    __shared__ double stf;
//    __shared__ double sum_foi[NUMSEROTYPES*NUMR];
    __shared__ int day;

    int index_gpu = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int index = index_gpu; index < params->dimension; index += stride)
    {
        y_0[index] = 0.0;
        y_tmp[index] = 0.0;
        y_err[index] = 0.0;
        dydt_in[index] = 0.0;
        dydt_out[index] = 0.0;
        k1[index] = 0.0;
        k2[index] = 0.0;
        k3[index] = 0.0;
        k4[index] = 0.0;
        k5[index] = 0.0;
        k6[index] = 0.0;
        r_max[index] = 0.0;
        D0[index] = 0.0;
        r[index] = 0.0;

//        for(int i = 0; i<NUMDAYSOUTPUT; i++){
//            gpu_func_test(t, y, dydt_in, index, day, params);
//            __syncthreads();
//            y[index] = dydt_in[index];
//            dydt_in[index] = 0;
//        }
//        printf("[function] OUT y[%d] = %f f[%d] = %f\n",index,y[index],index,dydt_in[index]);
//        return;

        while(t < t_target)
        {
//            if(index == 0 || index == params->dimension - 1) {
//                printf("[evolve apply] Index = %d t = %f h = %f start one day\n", index, t_start, h[index]);
//            }
            double device_t;
            double device_t1;
            double device_h;
            double device_h_0;
            double device_dt;
            int device_adjustment_out = 999;
            device_t = t;
            device_t1 = device_t + 1.0;
            device_h = h;

//            if(index == 0 || index == params->dimension - 1) {
//                printf("\n  Will run from %f to %f, step %f\n", t, device_t1, h);
//                printf("    t = %f t_1 = %f  h = %f\n", device_t, device_t1, device_h);
//            }
//            cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
            int output_index = day * params->display_dimension + index;
            if(output_index % params->display_dimension == 0){//write to first column the time
//                printf("0 - y_output[%d] = %f\n",output_index,day);
                y_output[output_index] = day;
            }
            if(output_index % params->display_dimension == 1){//write to second column the stf
//                printf("1 - y_output[%d] = %f\n",output_index,params->stf_d[day]);
                if(params->phis_d_length == 0){
                    y_output[output_index] = 1.0;
                }
                else{
                    y_output[output_index] = params->stf_d[day];
                }
            }
            y_output[output_index + 2] = y[index];//write to third column onward y
//            printf("2 - y_output[%d] = %f\n",output_index + 2,y[index]);
//            block.sync();

            while(device_t < device_t1)
            {
                int device_final_step = 0;
                const double device_t_0 = device_t;
                device_h_0 = device_h;
                device_dt = device_t1 - device_t_0;
                y_0[index] = y[index];
//                if(index == 0 || index == params->dimension - 1) {
//                    printf("[evolve apply] Index = %d t = %f t_0 = %f h = %f dt = %f start one iteration\n", index, t, t_0, h,dt);
//                }
//                if(index == 0 || index == params->dimension - 1) {
//                    printf("[evolve apply] Useydt_in\n");
//                }

                gpu_func_test(device_t_0, y, dydt_in, index, day, params);
                __syncthreads();
                while(true)
                {
                    if ((device_dt >= 0.0 && device_h_0 > device_dt) || (device_dt < 0.0 && device_h_0 < device_dt)) {
                        device_h_0 = device_dt;
                        device_final_step = 1;
                    } else {
                        device_final_step = 0;
                    }
                    rk45_gpu_step_apply(device_t_0, device_h_0,
                                        y, y_tmp, y_err, dydt_in, dydt_out,
                                        k1, k2, k3, k4, k5, k6,
                                        index, day, params);
                    if (device_final_step) {
                        device_t = device_t1;
                    } else {
                        device_t = device_t_0 + device_h_0;
                    }
                    double h_old = device_h_0;
                    rk45_gpu_adjust_h(y, y_err, dydt_out,
                                      &device_h, device_h_0, &device_adjustment_out, device_final_step,
                                      r, D0, r_max,
                                      index, params);
                    //Extra step to get data from h
                    device_h_0 = device_h;
                    if (device_adjustment_out == -1)
                    {
                        double t_curr = (device_t);
                        double t_next = (device_t) + device_h_0;

                        if (fabs(device_h_0) < fabs(h_old) && t_next != t_curr) {
                            /* Step was decreased. Undo step, and try again with new h0. */
//                            if(index == 0 || index == params->dimension - 1) {
//                                printf("  [evolve apply] index = %d step decreased, y = y0\n", index);
//                            }
                            y[index] = y_0[index];
                        } else {
//                            if(index == 0 || index == params->dimension - 1) {
//                                printf("  [evolve apply] index = %d step decreased h_0 = h_old\n", index);
//                            }
                            device_h_0 = h_old; /* keep current step size */
                            break;
                        }
                    }
                    else{
//                        if(index == 0 || index == params->dimension - 1) {
//                            printf("  [evolve apply] index = %d step increased or no change\n", index);
//                        }
                        break;
                    }
                }
//                if(index == 0 || index == params->dimension - 1)
//                {
//                    printf("    index = %d t = %f t_0 = %f  h = %f h_0 = %f\n", index, device_t, device_t_0, device_h, device_h_0);
//                    printf("    index = %d y[%d] = %f\n", index, index, y[index]);
//                    printf("\n");
//                    if(device_final_step)
//                    {
//                        if(index == 0 || index == params->dimension - 1) {
//                            if(index == 0)
//                            {
//                                printf("[output] index = %d t = %f t_0 = %f  h = %f h_0 = %f\n", index, device_t,
//                                   device_t_0, device_h, device_h_0);
//                                printf("[output] index = %d y[%d] = %f\n", index, index, y[index]);
//                                printf("\n");
//                            }
//                        }
//                    }
//                    printf("  [evolve apply] index = %d end\n\n",index);
//                }
//                /* Test */
//                t += device_h;
                //1D_index = row*width+col
//                if(index == 0){
//                    printf("Time = %d index = %d 1D index = %d\n",day,index,day*DIM + index);
//                }
                device_h = device_h_0;  /* suggest step size for next time-step */
//                t = device_t;
                h = device_h;
            }
//            if(index == 0) {
//                printf("[evolve apply] Index = %d t = %f h = %f end one day\n", index, t, h);
//            }
            t += t_delta;
            day += 1;
        }
    }
    return;
}

void GPU_RK45::run(){

    auto start_all = std::chrono::high_resolution_clock::now();
    auto start = std::chrono::high_resolution_clock::now();

    const int num_streams = 20;
    hipStream_t streams[num_streams];
    //y
    double *y_d[num_streams];
    double *y_output_d[num_streams];
    double *y_output[num_streams];
    GPU_Parameters* params_d[num_streams];

    for (int i = 0; i < num_streams; ++i) {
        y_output[i] = new double[NUMDAYSOUTPUT * params->display_dimension]();
        gpuErrchk(hipMalloc((void **) &y_d[i], params->dimension * sizeof(double)));
        gpuErrchk(hipMemcpy(y_d[i], params->y, params->dimension * sizeof(double), hipMemcpyHostToDevice));
        gpuErrchk(hipMalloc((void **) &y_output_d[i], NUMDAYSOUTPUT * params->display_dimension * sizeof(double)));
        gpuErrchk(hipMemcpy(y_output_d[i], params->y_output, NUMDAYSOUTPUT * params->display_dimension * sizeof(double),hipMemcpyHostToDevice));
        gpuErrchk(hipMalloc((void **) &params_d[i], sizeof(GPU_Parameters)));
        gpuErrchk(hipMemcpy(params_d[i], params, sizeof(GPU_Parameters), hipMemcpyHostToDevice));
    }
    gpuErrchk(hipDeviceSynchronize());

    int num_SMs;
    gpuErrchk(hipDeviceGetAttribute(&num_SMs, hipDeviceAttributeMultiprocessorCount, 0));
//    int numBlocks = 32*num_SMs; //multiple of 32
    int block_size = 256; //max is 1024
    int num_blocks = (params->dimension + block_size - 1) / block_size;
//    printf("[GSL GPU] SMs = %d block_size = %d num_blocks = %d\n",num_SMs,block_size,num_blocks);
    dim3 dimBlock(block_size, block_size); // so your threads are BLOCK_SIZE*BLOCK_SIZE, 256 in this case
    dim3 dimGrid(num_blocks, num_blocks); // 1*1 blocks in a grid

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for allocate mem CPU to GPU: %ld micro seconds which is %f seconds\n",duration.count(),(duration.count()/1e6));

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(rk45_gpu_evolve_apply), hipFuncCachePreferL1);
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024000*100);

    start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < num_streams; ++i) {
        hipStreamCreate(&streams[i]);
        rk45_gpu_evolve_apply<<<num_blocks, block_size, 0, streams[i]>>>(params->t0, params->t_target, 1.0, params->h,
                                                                      y_d[i],
                                                                      y_output_d[i],
                                                                      params_d[i]);
    }
    gpuErrchk(hipDeviceSynchronize());

    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for compute %d ODE with %d parameters, step %f in %f days on GPU: %ld micro seconds which is %f seconds\n",params->number_of_ode,params->dimension,params->h,params->t_target,duration.count(),(duration.count()/1e6));

    start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < num_streams; ++i) {
        gpuErrchk(hipMemcpy(y_output[i], y_output_d[i], NUMDAYSOUTPUT * params->display_dimension * sizeof(double), hipMemcpyDeviceToHost));
    }

    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for copy data from GPU on CPU: %ld micro seconds which is %f seconds\n",duration.count(),(duration.count()/1e6));

    start = std::chrono::high_resolution_clock::now();
    for(int s = 0; s < num_streams; s++){
        printf("Display on Host stream %d\n",s);
        for(int i = 0; i < NUMDAYSOUTPUT * params->display_dimension; i++){
            printf("%1.5f\t",y_output[s][i]);
            //reverse position from 1D array
            if(i > 0 && (i + 1) % params->display_dimension == 0){
                printf("\n");
            }
        }
        printf("\n");
    }
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for display: %ld micro seconds which is %f seconds\n",duration.count(),(duration.count()/1e6));

    auto stop_all = std::chrono::high_resolution_clock::now();
    auto duration_all = std::chrono::duration_cast<std::chrono::microseconds>(stop_all - start_all);
    printf("[GSL GPU] Time for all: %ld micro seconds which is %f seconds\n",duration_all.count(),(duration_all.count()/1e6));

    return;
}
