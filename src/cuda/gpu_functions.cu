#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__device__
void function(double t, const double y[], double dydt[], const int dim){
    const double m = 5.2;		// Mass of pendulum
    const double g = -9.81;		// g
    const double l = 2;		// Length of pendulum
    const double A = 0.5;		// Amplitude of driving force
    const double wd = 1;		// Angular frequency of driving force
    const double b = 0.5;		// Damping coefficient

    dydt[0] = y[1];
    dydt[1] = -(g / l) * sin(y[0]) + (A * cos(wd * t) - b * y[1]) / (m * l * l);
    return;
}

__device__
void rk45_gsl_gpu_adjust_h(double eps_abs, double eps_rel, double a_y, double a_dydt, unsigned int ord, double scale_abs[],
                           double *h, double h_0, int final_step,
                           double y[],double y_err[], double dydt_out[], int *adjustment_out, int dim){
    /* adaptive adjustment */
    /* Available control object constructors.
     *
     * The standard control object is a four parameter heuristic
     * defined as follows:
     *    D0 = eps_abs + eps_rel * (a_y |y| + a_dydt h |y'|)
     *    D1 = |yerr|
     *    q  = consistency order of method (q=4 for 4(5) embedded RK)
     *    S  = safety factor (0.9 say)
     *
     *                      /  (D0/D1)^(1/(q+1))  D0 >= D1
     *    h_NEW = S h_OLD * |
     *                      \  (D0/D1)^(1/q)      D0 < D1
     *
     * This encompasses all the standard error scaling methods.
     *
     * The y method is the standard method with a_y=1, a_dydt=0.
     * The yp method is the standard method with a_y=0, a_dydt=1.
     */
    const double S = 0.9;
    double h_old;
    if(final_step){
        h_old = h_0;
    }
    else{
        h_old = *h;
    }

    printf("    [adjust h] begin\n");
    for (int i = 0; i < dim; i ++)
    {
        printf("      y[%d] = %.10f\n",i,y[i]);
    }
    for (int i = 0; i < dim; i ++)
    {
        printf("      y_err[%d] = %.10f\n",i,y_err[i]);
    }
    for (int i = 0; i < dim; i ++)
    {
        printf("      dydt_out[%d] = %.10f\n",i,dydt_out[i]);
    }

    double r_max = 0.0;
    for (int i = 0; i < dim; i ++)
    {
        const double D0 = eps_rel * (a_y * fabs(y[i]) + a_dydt * fabs((h_old) * dydt_out[i])) + eps_abs * scale_abs[i];
        const double r  = fabs(y_err[i]) / fabs(D0);
        printf("      compare r = %.10f r_max = %.10f\n",r,r_max);
        r_max = max(r, r_max);
    }

    printf("      r_max = %.10f\n",r_max);

    if (r_max > 1.1) {
        /* decrease step, no more than factor of 5, but a fraction S more
           than scaling suggests (for better accuracy) */
        double r = S / pow(r_max, 1.0 / ord);

        if (r < 0.2)
            r = 0.2;
        *h = r * (h_old);

        printf("      decrease by %.10f, h_old is %.10f new h is %.10f\n", r, h_old, *h);
        *adjustment_out = -1;
    } else if (r_max < 0.5) {
        /* increase step, no more than factor of 5 */
        double r = S / pow(r_max, 1.0 / (ord + 1.0));

        if (r > 5.0)
            r = 5.0;

        if (r < 1.0)  /* don't allow any decrease caused by S<1 */
            r = 1.0;

        *h = r * (h_old);

        printf("      increase by %.10f, h_old is %.10f new h is %.10f\n", r, h_old, *h);
        *adjustment_out = 1;
    } else {
        /* no change */
        printf("      no change\n");
        *adjustment_out = 0;
    }
    printf("    [adjust h] end\n");
    return;
}

__device__
void rk45_gsl_gpu_step_apply(double t, double h,
                             double y[], double y_tmp[], double y_err[], double dydt_out[],
                             double k1[], double k2[], double k3[], double k4[], double k5[], double k6[], double temp[],
                             int dim)
{
    static const double ah[] = { 1.0/4.0, 3.0/8.0, 12.0/13.0, 1.0, 1.0/2.0 };
    static const double b3[] = { 3.0/32.0, 9.0/32.0 };
    static const double b4[] = { 1932.0/2197.0, -7200.0/2197.0, 7296.0/2197.0};
    static const double b5[] = { 8341.0/4104.0, -32832.0/4104.0, 29440.0/4104.0, -845.0/4104.0};
    static const double b6[] = { -6080.0/20520.0, 41040.0/20520.0, -28352.0/20520.0, 9295.0/20520.0, -5643.0/20520.0};

    static const double c1 = 902880.0/7618050.0;
    static const double c3 = 3953664.0/7618050.0;
    static const double c4 = 3855735.0/7618050.0;
    static const double c5 = -1371249.0/7618050.0;
    static const double c6 = 277020.0/7618050.0;

    static const double ec[] = { 0.0,
                                 1.0 / 360.0,
                                 0.0,
                                 -128.0 / 4275.0,
                                 -2197.0 / 75240.0,
                                 1.0 / 50.0,
                                 2.0 / 55.0
    };

    printf("    [step apply] start\n");
    printf("      t = %.10f h = %.10f\n",t,h);
    for (int i = 0; i < dim; i ++)
    {
        printf("      y[%d] = %.10f\n",i,y[i]);
        printf("      y_err[%d] = %.10f\n",i,y_err[i]);
        printf("      dydt_out[%d] = %.10f\n",i,dydt_out[i]);
    }

    /* k1 */
    function(t,y,k1,dim);
//    cudaDeviceSynchronize();
    for (int i = 0; i < dim; i ++)
    {
        printf("      k1[%d] = %.10f\n",i,k1[i]);
        y_tmp[i] = y[i] +  ah[0] * h * k1[i];
    }
    /* k2 */
    function(t + ah[0] * h, y_tmp,k2,dim);
//    cudaDeviceSynchronize();
    for (int i = 0; i < dim; i ++)
    {
        printf("      k2[%d] = %.10f\n",i,k2[i]);
        y_tmp[i] = y[i] + h * (b3[0] * k1[i] + b3[1] * k2[i]);
    }
    /* k3 */
    function(t + ah[1] * h, y_tmp,k3,dim);
//    cudaDeviceSynchronize();
    for (int i = 0; i < dim; i ++)
    {
        printf("      k3[%d] = %.10f\n",i,k3[i]);
        y_tmp[i] = y[i] + h * (b4[0] * k1[i] + b4[1] * k2[i] + b4[2] * k3[i]);
    }
    /* k4 */
    function(t + ah[2] * h, y_tmp,k4,dim);
//    cudaDeviceSynchronize();
    for (int i = 0; i < dim; i ++)
    {
        printf("      k4[%d] = %.10f\n",i,k4[i]);
        y_tmp[i] = y[i] + h * (b5[0] * k1[i] + b5[1] * k2[i] + b5[2] * k3[i] + b5[3] * k4[i]);
    }
    /* k5 */
    function(t + ah[3] * h, y_tmp,k5,dim);
//    cudaDeviceSynchronize();
    for (int i = 0; i < dim; i ++)
    {
        printf("      k5[%d] = %.10f\n",i,k5[i]);
        y_tmp[i] = y[i] + h * (b6[0] * k1[i] + b6[1] * k2[i] + b6[2] * k3[i] + b6[3] * k4[i] + b6[4] * k5[i]);
    }
    /* k6 */
    function(t + ah[4] * h, y_tmp,k6,dim);
//    cudaDeviceSynchronize();
    for (int i = 0; i < dim; i ++)
    {
        printf("      k6[%d] = %.10f\n",i,k6[i]);
        y_tmp[i] = y[i] + h * (b6[0] * k1[i] + b6[1] * k2[i] + b6[2] * k3[i] + b6[3] * k4[i] + b6[4] * k5[i]);
    }
    /* final sum */
    for (int i = 0; i < dim; i ++)
    {
        const double d_i = c1 * k1[i] + c3 * k3[i] + c4 * k4[i] + c5 * k5[i] + c6 * k6[i];
        y[i] += h * d_i;
    }
    /* Derivatives at output */
    function(t + h, y, dydt_out,dim);
//    cudaDeviceSynchronize();
    /* difference between 4th and 5th order */
    for (int i = 0; i < dim; i ++)
    {
        y_err[i] = h * (ec[1] * k1[i] + ec[3] * k3[i] + ec[4] * k4[i] + ec[5] * k5[i] + ec[6] * k6[i]);
    }
    for (int i = 0; i < dim; i++) {
        printf("      y[%d] = %.10f\n",i,y[i]);
    }
    for (int i = 0; i < dim; i++) {
        printf("      y_err[%d] = %.10f\n",i,y_err[i]);
    }
    for (int i = 0; i < dim; i++) {
        printf("      dydt_out[%d] = %.10f\n",i,dydt_out[i]);
    }
    printf("    [step apply] end\n");
    return;
}



__global__
void rk45_gsl_gpu_evolve_apply(double *t, double *t1, double *h,
                               double eps_abs, double eps_rel, double a_y, double a_dydt, unsigned int ord, double scale_abs[],
                               double y[], double y_0[], double y_tmp[], double y_err[], double dydt_out[],
                               double k1[], double k2[], double k3[], double k4[], double k5[], double k6[], double temp[], int *h_adjust_status,
                               const int dim){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    const double t_0 = *t;
    double h_0 = *h;
    double dt = *t1 - t_0;

    printf("  [evolve apply] start\n");

    for (int i = 0; i < dim; i ++){
        y_0[i] = y[i];
    }

    int final_step = 0;

    while(true){
        if ((dt >= 0.0 && h_0 > dt) || (dt < 0.0 && h_0 < dt)) {
            h_0 = dt;
            final_step = 1;
        } else {
            final_step = 0;
        }

        rk45_gsl_gpu_step_apply(t_0, h_0,
                                           y, y_tmp, y_err, dydt_out,
                                           k1, k2, k3, k4, k5, k6, temp,
                                           dim);
//        cudaDeviceSynchronize();

        if (final_step) {
            *t = *t1;
        } else {
            *t = t_0 + h_0;
        }

        double h_old = h_0;

        printf("    after adjust t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f h_old = %.10f\n",*t,t_0,*h,h_0,h_old);

        rk45_gsl_gpu_adjust_h(eps_abs, eps_rel, a_y, a_dydt, ord, scale_abs,
                                                         h, h_0, final_step,
                                                         y, y_err, dydt_out,
                                                         h_adjust_status, dim);
//        cudaDeviceSynchronize();

        //Extra step to get data from *h
        h_0 = *h;

        printf("    after adjust t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f h_old = %.10f\n",*t,t_0,*h,h_0,h_old);

        if (*h_adjust_status == -1)
        {
            double t_curr = (*t);
            double t_next = (*t) + h_0;

            if (fabs(h_0) < fabs(h_old) && t_next != t_curr) {
                /* Step was decreased. Undo step, and try again with new h0. */
                printf("  [evolve apply] step decreased, y = y0\n");
                for (int i = 0; i < dim; i++) {
                    y[i] = y_0[i];
                }
            } else {
                printf("  [evolve apply] step decreased h_0 = h_old\n");
                h_0 = h_old; /* keep current step size */
                break;
            }
        }
        else{
            printf("  [evolve apply] step increased or no change\n");
            break;
        }
    }
    *h = h_0;  /* suggest step size for next time-step */
    printf("    t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f dt = %.10f\n",*t,t_0,*h,h_0,dt);
    printf("    ");
    for (int i = 0; i < dim; i++){
        printf("y[%d] = %.10f\t",i,y[i]);
    }
    printf("\n");
    printf("  [evolve apply] end\n");
    return;
}

#define gpu_thread 1
#define gpu_block 1

bool rk45_gsl_gpu_simulate(){
    const int dim = 2;

    //Default parameters for RK45 in GSL
    double eps_abs = 1e-6;
    double eps_rel = 0.0;
    double a_y = 1.0;
    double a_dydt = 0.0;
    unsigned int ord = 5;
    //End default parameters for RK45

    double* y;
    double* y_0;
    double* y_tmp;
    double* y_err;
    double* dydt_out;
    double* scale_abs;
    double* k1;
    double* k2;
    double* k3;
    double* k4;
    double* k5;
    double* k6;
    double* temp;

    double* t1;
    double* t;
    double* h;
    double* dt;
    int* h_adjust_status;

    // Allocate Unified Memory – accessible from CPU or GPU

    hipMallocManaged(&t1, sizeof(double));
    hipMallocManaged(&t, sizeof(double));
    hipMallocManaged(&h, sizeof(double));
    hipMallocManaged(&dt, sizeof(double));
    hipMallocManaged(&h_adjust_status, sizeof(int));
    hipMallocManaged(&scale_abs, dim * sizeof(double));
    hipMallocManaged(&y, dim * sizeof(double));
    hipMallocManaged(&y_0, dim * sizeof(double));
    hipMallocManaged(&y_tmp, dim * sizeof(double));
    hipMallocManaged(&y_err, dim * sizeof(double));
    hipMallocManaged(&dydt_out, dim * sizeof(double));
    hipMallocManaged(&k1, dim * sizeof(double));
    hipMallocManaged(&k2, dim * sizeof(double));
    hipMallocManaged(&k3, dim * sizeof(double));
    hipMallocManaged(&k4, dim * sizeof(double));
    hipMallocManaged(&k5, dim * sizeof(double));
    hipMallocManaged(&k6, dim * sizeof(double));
    hipMallocManaged(&temp, dim * sizeof(double));

    // initialize x and y arrays on the host
    *t1 = 2.0;
    *t = 0.0;
    *h = 0.2;
    *dt = 0.0;
    *h_adjust_status = 999;
    y[0] = 0.5;
    y[1] = 0.5;
//    y[2] = 0.8;
    for (int i = 0; i < dim; i++) {
        scale_abs[i] = 1.0;
        y_0[i] = 0.0;
        y_tmp[i] = 0.0;
        y_err[i] = 0.0;
        dydt_out[i] = 0.0;
        k1[i] = 0.0;
        k2[i] = 0.0;
        k3[i] = 0.0;
        k4[i] = 0.0;
        k5[i] = 0.0;
        k6[i] = 0.0;
        temp[i] = 0.0;
    }

//    auto start_gpu = std::chrono::high_resolution_clock::now();
    int step_count = 0;
    while(*t < *t1){
        printf ("\n[main gpu] step %d\n", step_count);
        rk45_gsl_gpu_evolve_apply<<<gpu_thread, gpu_block>>>(t, t1, h,
                                             eps_abs, eps_rel, a_y, a_dydt, ord, scale_abs,
                                             y, y_0, y_tmp, y_err, dydt_out,
                                             k1, k2, k3, k4, k5, k6, temp, h_adjust_status,
                                             dim);

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();
        printf ("[main gpu] step %d t = %.10f \t  h = %.10f\n", step_count, *t, *h);
        for (int i = 0; i < dim; i++){
            printf("\t y = %.10f",y[i]);
        }
        printf("\n");
        step_count++;
    }
//    auto stop_gpu = std::chrono::high_resolution_clock::now();
//    auto duration_gpu = std::chrono::duration_cast<std::chrono::microseconds>(stop_gpu - start_gpu);
//    printf("gpu time: %d micro seconds which is %.10f seconds\n",duration_gpu.count(),(duration_gpu.count()/1e6));
    // Free memory
    hipFree(t1);
    hipFree(t);
    hipFree(h);
    hipFree(dt);
    hipFree(h_adjust_status);
    hipFree(scale_abs);
    hipFree(y);
    hipFree(y_0);
    hipFree(y_tmp);
    hipFree(y_err);
    hipFree(dydt_out);
    hipFree(k1);
    hipFree(k2);
    hipFree(k3);
    hipFree(k4);
    hipFree(k5);
    hipFree(k6);
    hipFree(temp);
    return true;
}