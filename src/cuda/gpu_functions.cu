#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <random>

//global parameters in device, will be instanced for each thread.

#define DIM 2

__device__
void function(double t, const double y[], double dydt[]){
    const double m = 5.2;		// Mass of pendulum
    const double g = -9.81;		// g
    const double l = 2;		// Length of pendulum
    const double A = 0.5;		// Amplitude of driving force
    const double wd = 1;		// Angular frequency of driving force
    const double b = 0.5;		// Damping coefficient

    dydt[0] = y[1];
    dydt[1] = -(g / l) * sin(y[0]) + (A * cos(wd * t) - b * y[1]) / (m * l * l);
    return;
}

__device__
void rk45_gpu_adjust_h(double y[], double y_err[], double dydt_out[],
                             double &h, double h_0, int &adjustment_out, int final_step, const int index){
    /* adaptive adjustment */
    /* Available control object constructors.
     *
     * The standard control object is a four parameter heuristic
     * defined as follows:
     *    D0 = eps_abs + eps_rel * (a_y |y| + a_dydt h |y'|)
     *    D1 = |yerr|
     *    q  = consistency order of method (q=4 for 4(5) embedded RK)
     *    S  = safety factor (0.9 say)
     *
     *                      /  (D0/D1)^(1/(q+1))  D0 >= D1
     *    h_NEW = S h_OLD * |
     *                      \  (D0/D1)^(1/q)      D0 < D1
     *
     * This encompasses all the standard error scaling methods.
     *
     * The y method is the standard method with a_y=1, a_dydt=0.
     * The yp method is the standard method with a_y=0, a_dydt=1.
     */
    static double eps_abs = 1e-6;
    static double eps_rel = 0.0;
    static double a_y = 1.0;
    static double a_dydt = 0.0;
    static unsigned int ord = 5;
    const double S = 0.9;
    static double h_old;
    if(final_step){
        h_old = h_0;
    }
    else{
        h_old = h;
    }

    printf("    [adjust h] index = %d begin\n",index);
    for (int i = 0; i < DIM; i ++)
    {
        printf("      y[%d] = %.10f\n",i,y[i]);
    }
    for (int i = 0; i < DIM; i ++)
    {
        printf("      y_err[%d] = %.10f\n",i,y_err[i]);
    }
    for (int i = 0; i < DIM; i ++)
    {
        printf("      dydt_out[%d] = %.10f\n",i,dydt_out[i]);
    }

    double r_max = 2.2250738585072014e-308;
    for (int i = 0; i < DIM; i ++)
    {
        const double D0 = eps_rel * (a_y * fabs(y[i]) + a_dydt * fabs((h_old) * dydt_out[i])) + eps_abs;
        const double r  = fabs(y_err[i]) / fabs(D0);
        printf("      compare r = %.10f r_max = %.10f\n",r,r_max);
        r_max = max(r, r_max);
    }

    printf("      r_max = %.10f\n",r_max);

    if (r_max > 1.1) {
        /* decrease step, no more than factor of 5, but a fraction S more
           than scaling suggests (for better accuracy) */
        double r = S / pow(r_max, 1.0 / ord);

        if (r < 0.2)
            r = 0.2;
        h = r * (h_old);

        printf("      index = %d decrease by %.10f, h_old is %.10f new h is %.10f\n",index, r, h_old, h);
        adjustment_out = -1;
    } else if (r_max < 0.5) {
        /* increase step, no more than factor of 5 */
        double r = S / pow(r_max, 1.0 / (ord + 1.0));

        if (r > 5.0)
            r = 5.0;

        if (r < 1.0)  /* don't allow any decrease caused by S<1 */
            r = 1.0;

        h = r * (h_old);

        printf("      index = %d increase by %.10f, h_old is %.10f new h is %.10f\n",index, r, h_old, h);
        adjustment_out = 1;
    } else {
        /* no change */
        printf("      index = %d no change\n",index);
        adjustment_out = 0;
    }
    printf("    [adjust h] index = %d end\n",index);
    return;
}

__device__
void rk45_gpu_step_apply(double t, double h, double y[], double y_err[], double dydt_out[],const int index)
{
    static const double ah[] = { 1.0/4.0, 3.0/8.0, 12.0/13.0, 1.0, 1.0/2.0 };
    static const double b3[] = { 3.0/32.0, 9.0/32.0 };
    static const double b4[] = { 1932.0/2197.0, -7200.0/2197.0, 7296.0/2197.0};
    static const double b5[] = { 8341.0/4104.0, -32832.0/4104.0, 29440.0/4104.0, -845.0/4104.0};
    static const double b6[] = { -6080.0/20520.0, 41040.0/20520.0, -28352.0/20520.0, 9295.0/20520.0, -5643.0/20520.0};

    static const double c1 = 902880.0/7618050.0;
    static const double c3 = 3953664.0/7618050.0;
    static const double c4 = 3855735.0/7618050.0;
    static const double c5 = -1371249.0/7618050.0;
    static const double c6 = 277020.0/7618050.0;

    static const double ec[] = { 0.0,
                                 1.0 / 360.0,
                                 0.0,
                                 -128.0 / 4275.0,
                                 -2197.0 / 75240.0,
                                 1.0 / 50.0,
                                 2.0 / 55.0
    };

    printf("    [step apply] index = %d start\n",index);
    printf("      t = %.10f h = %.10f\n",t,h);
    for (int i = 0; i < DIM; i ++)
    {
        printf("      y[%d] = %.10f\n",i,y[i]);
        printf("      y_err[%d] = %.10f\n",i,y_err[i]);
        printf("      dydt_out[%d] = %.10f\n",i,dydt_out[i]);
    }

//    double* y_tmp = (double*)malloc(dim);
//    double* k1 = (double*)malloc(dim);
//    double* k2 = (double*)malloc(dim);
//    double* k3 = (double*)malloc(dim);
//    double* k4 = (double*)malloc(dim);
//    double* k5 = (double*)malloc(dim);
//    double* k6 = (double*)malloc(dim);
    double y_tmp[DIM];
    double k1[DIM];
    double k2[DIM];
    double k3[DIM];
    double k4[DIM];
    double k5[DIM];
    double k6[DIM];

    for(int i = 0; i < DIM; i++){
        y_tmp[i] = 0.0;
        y_err[i] = 0.0;
        dydt_out[i] = 0.0;
        k1[i] = 0.0;
        k2[i] = 0.0;
        k3[i] = 0.0;
        k4[i] = 0.0;
        k5[i] = 0.0;
        k6[i] = 0.0;
    }

    /* k1 */
    function(t,y,k1);
    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i ++)
    {
        printf("      k1[%d] = %.10f\n",i,k1[i]);
        y_tmp[i] = y[i] +  ah[0] * h * k1[i];
    }
    /* k2 */
    function(t + ah[0] * h, y_tmp,k2);
    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i ++)
    {
        printf("      k2[%d] = %.10f\n",i,k2[i]);
        y_tmp[i] = y[i] + h * (b3[0] * k1[i] + b3[1] * k2[i]);
    }
    /* k3 */
    function(t + ah[1] * h, y_tmp,k3);
    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i ++)
    {
        printf("      k3[%d] = %.10f\n",i,k3[i]);
        y_tmp[i] = y[i] + h * (b4[0] * k1[i] + b4[1] * k2[i] + b4[2] * k3[i]);
    }
    /* k4 */
    function(t + ah[2] * h, y_tmp,k4);
    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i ++)
    {
        printf("      k4[%d] = %.10f\n",i,k4[i]);
        y_tmp[i] = y[i] + h * (b5[0] * k1[i] + b5[1] * k2[i] + b5[2] * k3[i] + b5[3] * k4[i]);
    }
    /* k5 */
    function(t + ah[3] * h, y_tmp,k5);
    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i ++)
    {
        printf("      k5[%d] = %.10f\n",i,k5[i]);
        y_tmp[i] = y[i] + h * (b6[0] * k1[i] + b6[1] * k2[i] + b6[2] * k3[i] + b6[3] * k4[i] + b6[4] * k5[i]);
    }
    /* k6 */
    function(t + ah[4] * h, y_tmp,k6);
    hipDeviceSynchronize();
    /* final sum */
    for (int i = 0; i < DIM; i ++)
    {
        printf("      k6[%d] = %.10f\n",i,k6[i]);
        const double d_i = c1 * k1[i] + c3 * k3[i] + c4 * k4[i] + c5 * k5[i] + c6 * k6[i];
        y[i] += h * d_i;
    }
    /* Derivatives at output */
    function(t + h, y, dydt_out);
    hipDeviceSynchronize();
    /* difference between 4th and 5th order */
    for (int i = 0; i < DIM; i ++)
    {
        y_err[i] = h * (ec[1] * k1[i] + ec[3] * k3[i] + ec[4] * k4[i] + ec[5] * k5[i] + ec[6] * k6[i]);
    }
    for (int i = 0; i < DIM; i++) {
        printf("      index = %d y[%d] = %.10f\n",index,i,y[i]);
    }
    for (int i = 0; i < DIM; i++) {
        printf("      index = %d y_err[%d] = %.10f\n",index,i,y_err[i]);
    }
    for (int i = 0; i < DIM; i++) {
        printf("      index = %d dydt_out[%d] = %.10f\n",index,i,dydt_out[i]);
    }
    printf("    [step apply] index = %d end\n",index);
    return;
}

__global__
void rk45_gpu_evolve_apply_2(double t1, double t, double h, double *y[], int thread_number){
    int index_gpu = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    double device_t;
    double device_t1;
    double device_h;
    double device_h_0;
    int device_adjustment_out = 999;
    int device_final_step = 0;

//    double* device_y = (double*)malloc(dim);
//    double* device_y_0 = (double*)malloc(dim);
//    double* device_y_err = (double*)malloc(dim);
//    double* device_dydt_out = (double*)malloc(dim);
    double device_y[DIM];
    double device_y_0[DIM];
    double device_y_err[DIM];
    double device_dydt_out[DIM];

    for(int index = index_gpu; index < thread_number; index += stride){
        device_t1 = t1;
        device_t = t;
        device_h = h;

        while(device_t < device_t1){
            const double t_0 = device_t;
            device_h_0 = device_h;
            double dt = device_t1 - t_0;

            printf("\n  [evolve apply] index = %d start\n",index);

            for (int i = 0; i < DIM; i ++){
                device_y[i] = y[index][i];
                device_y_0[i] = device_y[i];
            }

            device_final_step = 0;

            while(true){
                if ((dt >= 0.0 && device_h_0 > dt) || (dt < 0.0 && device_h_0 < dt)) {
                    device_h_0 = dt;
                    device_final_step = 1;
                } else {
                    device_final_step = 0;
                }

                rk45_gpu_step_apply(t_0,device_h_0,device_y,device_y_err,device_dydt_out,index);
                hipDeviceSynchronize();

                if (device_final_step) {
                    device_t = device_t1;
                } else {
                    device_t = t_0 + device_h_0;
                }

                double h_old = device_h_0;

                printf("    before adjust t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f h_old = %.10f\n",device_t,t_0,device_h,device_h_0,h_old);

                rk45_gpu_adjust_h(device_y, device_y_err, device_dydt_out,
                                        device_h, device_h_0, device_adjustment_out, device_final_step,index);
                hipDeviceSynchronize();

                //Extra step to get data from *h
                device_h_0 = device_h;

                printf("    after adjust t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f h_old = %.10f\n",device_t,t_0,device_h,device_h_0,h_old);

                if (device_adjustment_out == -1)
                {
                    double t_curr = (device_t);
                    double t_next = (device_t) + device_h_0;

                    if (fabs(device_h_0) < fabs(h_old) && t_next != t_curr) {
                        /* Step was decreased. Undo step, and try again with new h0. */
                        printf("  [evolve apply] index = %d step decreased, y = y0\n",index);
                        for (int i = 0; i < DIM; i++) {
                            device_y[i] = device_y_0[i];
                        }
                    } else {
                        printf("  [evolve apply] index = %d step decreased h_0 = h_old\n",index);
                        device_h_0 = h_old; /* keep current step size */
                        break;
                    }
                }
                else{
                    printf("  [evolve apply] index = %d step increased or no change\n",index);
                    break;
                }
            }
            device_h = device_h_0;  /* suggest step size for next time-step */
            printf("    index = %d t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f\n",index,device_t,t_0,device_h,device_h_0);
            for (int i = 0; i < DIM; i++){
                printf("    index = %d y[%d][%d] = %.10f\n",index,index,i,device_y[i]);
            }
            printf("  [evolve apply] index = %d end\n",index);
            for (int i = 0; i < DIM; i ++){
                y[index][i] = device_y[i];
            }
            if(device_final_step){
                printf("[output] index = %d t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f\n",index,device_t,t_0,device_h,device_h_0);
                for (int i = 0; i < DIM; i++){
                    printf("[output] index = %d y[%d][%d] = %.10f\n",index,index,i,device_y[i]);
                }
            }
//            hipDeviceSynchronize();
        }
    }
    return;
}

#define gpu_threads2 1

bool rk45_gpu_simulate(const int gpu_threads, const int display_numbers){

    auto start = std::chrono::high_resolution_clock::now();

    double t1 = 2.0;
    double t = 0.0;
    double h = 0.2;

    double **y = new double*[gpu_threads]();
    for (int i = 0; i < gpu_threads; i++)
    {
        y[i] = new double[DIM];
        for(int j = 0; j < DIM; j++){
            y[i][j] = 0.5;
        }
    }
    double **y_d = 0;
    //temp pointers
    double **tmp_ptr = (double**)malloc (gpu_threads * sizeof (double));
    for (int i = 0; i < gpu_threads; i++) {
        hipMalloc ((void **)&tmp_ptr[i], DIM * sizeof (double));
        hipMemcpy(tmp_ptr[i], y[i], DIM * sizeof(double), hipMemcpyHostToDevice);
    }
    //y
    hipMalloc ((void **)&y_d, gpu_threads * sizeof (double));
    hipMemcpy (y_d, tmp_ptr, gpu_threads * sizeof (double), hipMemcpyHostToDevice);
    for (int i = 0; i < gpu_threads; i++) {
        hipMemcpy (tmp_ptr[i], y[i], DIM * sizeof (double), hipMemcpyHostToDevice);
    }

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("Time for allocate mem CPU to GPU: %lld micro seconds which is %.10f seconds\n",duration.count(),(duration.count()/1e6));

    start = std::chrono::high_resolution_clock::now();
//    hipProfilerStart();
    rk45_gpu_evolve_apply_2<<<1,1>>>(t1, t, h,y_d,gpu_threads);

//    hipProfilerStop();
    hipDeviceSynchronize();
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("Time for ODE on GPU: %lld micro seconds which is %.10f seconds\n",duration.count(),(duration.count()/1e6));

    start = std::chrono::high_resolution_clock::now();
    double** host_y_output = (double**)malloc (gpu_threads * sizeof (double));
    for (int i = 0; i < gpu_threads; i++) {
        host_y_output[i] = (double *)malloc (DIM * sizeof (double));
    }
    hipMemcpy (tmp_ptr, y_d, gpu_threads * sizeof (double), hipMemcpyDeviceToHost);
    for (int i = 0; i < gpu_threads; i++) {
        hipMemcpy (host_y_output[i], tmp_ptr[i], DIM * sizeof (double), hipMemcpyDeviceToHost);
    }
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("Time for data transfer GPU to CPU: %lld micro seconds which is %.10f seconds\n",duration.count(),(duration.count()/1e6));

    start = std::chrono::high_resolution_clock::now();
    std::random_device rd; // obtain a random number from hardware
    std::mt19937 gen(rd()); // seed the generator
    std::uniform_int_distribution<> distr(0, gpu_threads); // define the range

    for(int i = 0; i < display_numbers; i++){
        int random_index = distr(gen);
        for(int index = 0; index < DIM; index++){
            printf("thread %d y[%d][%d] = %.10f\n",random_index,random_index,index,host_y_output[random_index][index]);
        }
        printf("\n");
    }
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("Time for display random results on CPU: %lld micro seconds which is %.10f seconds\n",duration.count(),(duration.count()/1e6));
    // Free memory
    hipFree(y);
    hipFree(y_d);
    return true;
}