#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_rk45.h"

__device__
void gpu_func_test(float t, const float y[], float f[], int index, int day, GPU_Parameters* gpu_params){

//    if(index == 0){
//        printf("Here's the info on params: \n");
//        printf("beta1 = %1.9f \n", gpu_params->beta[0]);
//        printf("beta2 = %1.9f \n", gpu_params->beta[1]);
//        printf("beta3 = %1.9f \n", gpu_params->beta[2]);
//        printf("a = %1.3f \n", gpu_params->v_d[gpu_params->i_amp]);
//        printf("sigma_H1B = %1.3f \n", gpu_params->sigma[0][1]);
//        printf("sigma_BH3 = %1.3f \n", gpu_params->sigma[1][2]);
//        printf("sigma_H1H3 = %1.3f \n", gpu_params->sigma[0][2]);
//
//        printf("phis_length = %d\n",gpu_params->phis_d_length);
//        for(int i=0; i<gpu_params->phis_d_length; i++){
//            printf("phi = %5.1f \n", gpu_params->phis_d[i]);
//        }
//    }

    // the transition rate among R-classes
    float trr = fdividef(NUMR, gpu_params->v_d[gpu_params->i_immune_duration]);
//    float stf = gpu_params->phis_d_length == 0 ? 1.0 : gpu_params->stf_d[day];
//    float stf = seasonal_transmission_factor(gpu_params,day);
    float stf = seasonal_transmission_factor(gpu_params,t);
//    float stf = gpu_params->stf;
//    float stf = 1.0;

//    if(index < STARTS)
//    {
//        printf("[function] IN y[%d] = %.20f f[%d] = %.20f\n",index,y[index],index,f[index]);
//        if(index == 0){
//            printf("\n");
//        }
//    }

//    printf("y[%d] = y %f\n",index,y[index]);

    const unsigned int START_I  = int(STARTI);
    const unsigned int START_J  = int(STARTJ);
    const unsigned int START_S  = int(STARTS);
    const unsigned int NUM_LOC  = int(NUMLOC);
    const unsigned int NUM_SEROTYPES  = int(NUMSEROTYPES);
    const unsigned int NUM_R  = int(NUMR);

    f[index] = 0.0;
    if(index < START_I){
//        int zDirection = i % zLength;
//        int yDirection = (i / zLength) % yLength;
//        int xDirection = i / (yLength * zLength);
        f[index] = index;
        int loc = index / (NUM_SEROTYPES * NUM_R);
        int vir = (index / NUM_R) % NUM_SEROTYPES;
        int stg = index % NUM_R;
        f[ index ] = - trr * y[ index ];
        if(index % NUM_R == 0){
            f[ index ] += gpu_params->v_d[gpu_params->i_nu] * y[ START_I + NUM_SEROTYPES*loc + vir ];
        }
        else{
            f[ index ] += trr * y[ NUM_SEROTYPES*NUM_R*loc + NUM_R*vir + stg - 1 ];
        }
        float sum_foi = 0.0;
        const int start_index = index * NUM_LOC*NUM_SEROTYPES;
        const int end_index = start_index + (NUM_LOC*NUM_SEROTYPES);

        for(int k = start_index; k < end_index; k++){
            sum_foi +=   gpu_params->sum_foi_sbe[k]
                        * stf
                        * y[gpu_params->sum_foi_y_index[k]];
        }

        f[index] +=  -(sum_foi) * y[index];
    }
    if(index >= START_I && index < START_J){
        int loc = (index - START_I) / NUM_SEROTYPES;
        int vir = (index - START_I) % NUM_SEROTYPES;
        f[ START_I + NUM_SEROTYPES*loc + vir ] = 0.0;
        f[ START_J + NUM_SEROTYPES*loc + vir ] = 0.0;
        float foi_on_susc_single_virus = 0.0;

        for(int l = 0; l<NUM_LOC; l++){
            foi_on_susc_single_virus += gpu_params->eta[loc][l]
                                        * stf
                                        * gpu_params->beta[vir]
                                        * y[START_I + NUM_SEROTYPES * l + vir];
        }

        f[ START_I + NUM_SEROTYPES*loc + vir ] += y[ START_S + loc ] * foi_on_susc_single_virus;
        f[ START_J + NUM_SEROTYPES*loc + vir ] += y[ START_S + loc ] * foi_on_susc_single_virus;

        const int start_index = (index % (NUM_LOC*NUM_SEROTYPES*NUM_R)) * (NUM_LOC*NUM_SEROTYPES*NUM_R);
        const int end_index = start_index + (NUM_LOC*NUM_SEROTYPES*NUM_R);

        float inflow_from_recovereds = 0.0;
        for(int k = start_index; k < end_index; k++){
            inflow_from_recovereds +=   gpu_params->inflow_from_recovereds_sbe[k]
                                        * stf
                                        * y[gpu_params->inflow_from_recovereds_y1_index[k]]
                                        * y[gpu_params->inflow_from_recovereds_y2_index[k]];
        }
        f[ START_I + NUM_SEROTYPES*loc + vir ] += inflow_from_recovereds;
        f[ START_J + NUM_SEROTYPES*loc + vir ] += inflow_from_recovereds;

        // add the recovery rate - NOTE only for I-classes
        f[ START_I + NUM_SEROTYPES*loc + vir ] += - gpu_params->v_d[gpu_params->i_nu] * y[ START_I + NUM_SEROTYPES*loc + vir ];
    }
    if(index >= START_S && index < gpu_params->dimension)
    {
        unsigned int loc = index - START_S;
        float foi_on_susc_all_viruses = 0.0;

        const int start_index = loc * NUM_LOC*NUM_SEROTYPES;
        const int end_index = start_index + (NUM_LOC*NUM_SEROTYPES);

        for(int k = start_index; k < end_index; k++){
            foi_on_susc_all_viruses +=   gpu_params->foi_on_susc_all_viruses_eb[k]
                                         * stf
                                         * y[gpu_params->foi_on_susc_all_viruses_y_index[k]];
        }

        f[ index ] = ( - foi_on_susc_all_viruses ) * y[ index ];
        for(int vir = 0; vir<NUM_SEROTYPES; vir++)
        {
            // add to dS/dt the inflow of recovereds from the final R-stage
            f[ index ] += trr * y[ NUM_SEROTYPES*NUM_R*(loc) + NUM_R*vir + (NUM_R - 1) ]; // "NUM_R-1" gets you the final R-stage only
        }
    }

//    if(index < START_S)
//    {
//        printf("[function] OUT y[%d] = %.20f f[%d] = %.20f\n",index,y[index],index,f[index]);
//        if(index == 0){
//            printf("\n");
//        }
//    }

    return;
}
