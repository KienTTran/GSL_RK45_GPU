#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_rk45.h"

__device__ int get_1d_index_from_5(const int loc,const  int vir,const  int stg,const  int l,const  int v){
    return loc*NUMSEROTYPES*NUMR*NUMLOC*NUMSEROTYPES + vir*NUMR*NUMLOC*NUMSEROTYPES + stg*NUMLOC*NUMSEROTYPES + l*NUMSEROTYPES + v;
}

__device__ int get_1d_index_start_from_3(const int loc,const  int vir,const  int stg){
    return loc*NUMSEROTYPES*NUMR*NUMLOC*NUMSEROTYPES + vir*NUMR*NUMLOC*NUMSEROTYPES + stg*NUMLOC*NUMSEROTYPES;
}

__device__ int get_1d_index_end_from_3(const int loc,const  int vir,const  int stg){
    return loc*NUMSEROTYPES*NUMR*NUMLOC*NUMSEROTYPES + vir*NUMR*NUMLOC*NUMSEROTYPES + stg*NUMLOC*NUMSEROTYPES + (NUMLOC*NUMSEROTYPES);
}

__device__ double get_sum_foi_sbe_from_1(const int index_1d, const int offset, GPU_Parameters* gpu_params){
//    printf("      sum_foi_sbe[%d] = %f\n",index_1d,gpu_params->sum_foi_sbe[index_1d]);
    return gpu_params->sum_foi_sbe[index_1d];
}

__device__ double get_sum_foi_sbe_from_5(const int loc,const  int vir,const  int stg,const  int l,const  int v, GPU_Parameters* gpu_params){
    return gpu_params->sum_foi_sbe[get_1d_index_from_5(loc, vir, stg, l, v)];
}

__device__ double get_pass1_y_I(const int index, const double y[]){
    return y[STARTI + index];
}

__device__ double get_sum_foi_sbe_from_3(const int loc, const int vir, const int stg,  const double y[], GPU_Parameters* gpu_params){
    cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
    double sum_foi_sbe = 0.0;
//    printf("  loc = %d vir = %d stg = %d sum from %d to %d\n",loc,vir,stg,get_1d_index_start_from_3(loc,vir,stg),get_1d_index_end_from_3(loc,vir,stg));
    for(int i = get_1d_index_start_from_3(loc,vir,stg); i < get_1d_index_end_from_3(loc,vir,stg); i++){
        sum_foi_sbe += get_sum_foi_sbe_from_1(i,i - get_1d_index_start_from_3(loc,vir,stg),gpu_params) * get_pass1_y_I(i - get_1d_index_start_from_3(loc,vir,stg),y);
//        printf("    loc = %d vir = %d stg = %d sum_foi_sbe index = %d y I index is %d y = %f\n",loc,vir,stg,i,STARTI + (i - get_1d_index_start_from_3(loc,vir,stg)),get_pass1_y_I(i - get_1d_index_start_from_3(loc,vir,stg),y));
    }
//    printf("  loc = %d vir = %d stg = %d sum_foi = %f\n",loc,vir,stg, sum_foi);
    block.sync();
    return sum_foi_sbe;
}

__device__
void gpu_func_test(double t, const double y[], double f[], int index, void *params){

    //    printf("gpu_function start\n");
    // just to be safe, cast the void-pointer to convert it to a prms-pointer
    GPU_Parameters* gpu_params = (GPU_Parameters*) params;

//    if(index == 0){
//        printf("Here's the info on params: \n");
//        printf("beta1 = %1.9f \n", gpu_params->beta[0]);
//        printf("beta2 = %1.9f \n", gpu_params->beta[1]);
//        printf("beta3 = %1.9f \n", gpu_params->beta[2]);
//        printf("a = %1.3f \n", gpu_params->v_d[gpu_params->i_amp]);
//        printf("sigma_H1B = %1.3f \n", gpu_params->sigma[0][1]);
//        printf("sigma_BH3 = %1.3f \n", gpu_params->sigma[1][2]);
//        printf("sigma_H1H3 = %1.3f \n", gpu_params->sigma[0][2]);
//
//        printf("phis_length = %d\n",gpu_params->phis_d_length);
//        for(int i=0; i<gpu_params->phis_d_length; i++){
//            printf("phi = %5.1f \n", gpu_params->phis_d[i]);
//        }
//    }

    // the transition rate among R-classes
    double trr = ((double)NUMR) / gpu_params->v_d[gpu_params->i_immune_duration];

    double stf = gpu_params->phis_d_length == 0 ? 1.0 : gpu_params->stf_d[static_cast<int>(t)];
//    double stf = seasonal_transmission_factor(gpu_params,t);

//    R -  i = 0 f[0] = 0-R1a
//    R -  i = 1 f[1] = 0-R1b
//    R -  i = 2 f[2] = 0-R1c
//    R -  i = 3 f[3] = 0-R1d
//    R -  i = 4 f[4] = 0-R2a
//    R -  i = 5 f[5] = 0-R2b
//    R -  i = 6 f[6] = 0-R2c
//    R -  i = 7 f[7] = 0-R2d
//    R -  i = 8 f[8] = 0-R3a
//    R -  i = 9 f[9] = 0-R3b
//    R -  i = 10 f[10] = 0-R3c
//    R -  i = 11 f[11] = 0-R3d
//    I -  i = 12 f[12] = 0-I1
//    I -  i = 13 f[13] = 0-I2
//    I -  i = 14 f[14] = 0-I3
//    J -  i = 15 f[15] = 0-J1
//    J -  i = 16 f[16] = 0-J2
//    J -  i = 17 f[17] = 0-J3
//    S -  i = 18 f[18] = 0-S

    __shared__ bool step_I_done[DIM];
    __shared__ bool step_IJ_done[DIM];

//    printf("y[%d] = y %f\n",index,y[index]);
    if(index < STARTI){
//        int zDirection = i % zLength;
//        int yDirection = (i / zLength) % yLength;
//        int xDirection = i / (yLength * zLength);
        int loc = index / (NUMSEROTYPES * NUMR);
        int vir = (index / NUMR) % NUMSEROTYPES;
        int stg = index % NUMR;
//        printf("index = %d index = %d Loc %d R vir %d stg %d\n",index,index,loc,vir,stg);
//        if(index == 11)
        {
            f[ index ] = - trr * y[ index ];
            if(index % NUMR == 0){
    //            printf("  Index %d stg == 0\n",index);
                f[ index ] += gpu_params->v_d[gpu_params->i_nu] * y[ STARTI + NUMSEROTYPES*loc + vir ];
            }
            else{
                f[ index ] += trr * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg - 1 ];
            }
            double sum_foi_3 = get_sum_foi_sbe_from_3(loc,vir,stg,y,gpu_params);
            f[ index ] += ( - sum_foi_3)
                          * stf
                          * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ];
    //        printf("loc = %d vir = %d stg = %d flat y[%d] = %f\n",loc,vir,stg,index,y[ index ]);
    //        printf("loc = %d vir = %d stg = %d flat sum_foi = %f\n",loc,vir,stg,sum_foi_3);
    //        printf("loc = %d vir = %d stg = %d flat f[%d] = %f\n",loc,vir,stg,index,f[ index ]);
    //        printf("\n");
    //        if(index == STARTI - 1){
    //            printf("\n");
    //        }
        }
    }
    else if(index < STARTS){
        int loc = ((index - STARTJ) / (NUMSEROTYPES)) % NUMLOC;
        int vir = (index - NUMSEROTYPES*NUMR*NUMLOC) % NUMSEROTYPES;
//        printf("index = %d Loc %d I vir %d\n",index,loc,vir);
//        if(index == 29)
        {
            if(index < STARTJ){
                f[STARTI + NUMSEROTYPES * loc + vir] = 0.0;
//            f[STARTJ + NUMSEROTYPES * loc + vir] = 0.0;
                double foi_on_susc_single_virus = 0.0;
//                cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
                for(int l = 0; l < NUMLOC; l++){
                    double foi_on_susc_single_virus_eb = gpu_params->eta[loc][l]
                                                         * stf
                                                         * gpu_params->beta[vir];
                    foi_on_susc_single_virus += foi_on_susc_single_virus_eb * y[STARTI + NUMSEROTYPES * l + vir];
//                    printf("  loc = %d vir = %d l = %d y I index is %d y = %f\n",loc,vir,l,STARTI + NUMSEROTYPES * l + vir,y[STARTI + NUMSEROTYPES * l + vir]);
//                    printf("  loc = %d vir = %d l = %d flat foi_on_susc_single_virus_eb = %f\n",loc,vir,l,foi_on_susc_single_virus_eb);
//                    printf("  loc = %d vir = %d l = %d flat foi_on_susc_single_virus = %f\n",loc,vir,l,foi_on_susc_single_virus);
                }
                f[ STARTI + NUMSEROTYPES*loc + vir ] += y[ STARTS + loc ] * foi_on_susc_single_virus;
//            f[ STARTJ + NUMSEROTYPES*loc + vir ] += y[ STARTS + loc ] * foi_on_susc_single_virus;
//                printf("loc = %d vir = %d y I index is %d y = %f\n",loc,vir,STARTS + loc,y[STARTS + loc]);
//                printf("loc = %d vir = %d flat foi_on_susc_single_virus = %f\n",loc,vir,foi_on_susc_single_virus);
//                printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTI + NUMSEROTYPES*loc + vir,f[STARTI + NUMSEROTYPES*loc + vir]);
//            printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTJ + NUMSEROTYPES*loc + vir,f[STARTJ + NUMSEROTYPES*loc + vir]);
//                printf("\n");

                double inflow_from_recovereds = 0.0;
                for(int i = 0; i < NUMLOC*NUMSEROTYPES*NUMR; i++){
                    int multiplier = index - (NUMLOC*NUMSEROTYPES*NUMR);
                    int inflow_index = (multiplier)*NUMLOC*NUMSEROTYPES*NUMR + i;
                    inflow_from_recovereds +=   gpu_params->inflow_from_recovereds_sbe[inflow_index]
                                                * stf
                                                * y[STARTI + NUMSEROTYPES * loc + vir] * y[i];
//                    printf("loc = %d vir = %d i = %d y index = %d y = %f\n",loc,vir,i,STARTI + NUMSEROTYPES * loc + vir,y[STARTI + NUMSEROTYPES * loc + vir]);
//                    printf("loc = %d vir = %d i = %d y index = %d y = %f\n",loc,vir,i,i,y[i]);
//                    printf("loc = %d vir = %d i = %d multiplier = %d inflow_from_recovereds_sbe index = %d inflow_from_recovereds_sbe = %f\n",loc,vir,i,multiplier,inflow_index,gpu_params->inflow_from_recovereds_sbe[inflow_index]);
//                    printf("loc = %d vir = %d i = %d inflow_from_recovereds = %f\n",loc,vir,i,inflow_from_recovereds);
//                    printf("\n");
                }

//                printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTI + NUMSEROTYPES * loc + vir,f[STARTI + NUMSEROTYPES * loc + vir]);
//            printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTJ + NUMSEROTYPES * loc + vir,f[STARTJ + NUMSEROTYPES * loc + vir]);

                f[ STARTI + NUMSEROTYPES*loc + vir ] += inflow_from_recovereds;
//            f[ STARTJ + NUMSEROTYPES*loc + vir ] += inflow_from_recovereds;
//                printf("loc = %d vir = %d flat inflow_from_recovereds = %f\n", loc, vir, inflow_from_recovereds);
//                printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTI + NUMSEROTYPES * loc + vir,f[STARTI + NUMSEROTYPES * loc + vir]);
//            printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTJ + NUMSEROTYPES * loc + vir,f[STARTJ + NUMSEROTYPES * loc + vir]);

//                step_I_done[index] = true;
//                block.sync();
            }
            else {//MOve J to here after I completed
//                cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
                //Wait for I completed
//                while(!step_I_done[index - NUMLOC*NUMSEROTYPES]){
//                    __syncthreads();
//                }
                f[index] = index;
                f[ index ] = f[ index - (NUMLOC * NUMSEROTYPES)];
//            // add the recovery rate - NOTE only for I-classes
                f[ index - (NUMLOC * NUMSEROTYPES) ] += - gpu_params->v_d[gpu_params->i_nu] * y[ index - (NUMLOC * NUMSEROTYPES) ];

//                printf("loc = %d vir = %d flat f[%d] I only = %f\n", loc, vir, index - (NUMLOC * NUMSEROTYPES),f[index - (NUMLOC * NUMSEROTYPES)]);
//                printf("f[%d] = f[%d]\n",index,index - (NUMLOC*NUMSEROTYPES));
//                step_IJ_done[index] = true;
//                block.sync();
            }
        }
    }
    else{
//        cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
        //Wait for IJ completed
//        while(!step_IJ_done[index - NUMLOC*NUMSEROTYPES]){
//            __syncthreads();
//        }
//    if(index == 36)
    {
        double foi_on_susc_all_viruses = 0.0;
        for(int i = 0; i < NUMLOC*NUMLOC*NUMSEROTYPES; i++){
            if(i < NUMLOC*NUMSEROTYPES){
                foi_on_susc_all_viruses +=  gpu_params->foi_on_susc_all_viruses_eb[i]
                                            * stf
                                            * y[STARTI + (i % (NUMLOC*NUMSEROTYPES))];
//                printf("< %d i = %d foi_on_susc_all_viruses_eb[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,i,gpu_params->foi_on_susc_all_viruses_eb[i]);
//                printf("< %d i = %d y[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,STARTI + (i % (NUMLOC*NUMSEROTYPES)),y[STARTI + (i % (NUMLOC*NUMSEROTYPES))]);
//                printf("< %d i = %d f[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,index + (i / (NUMLOC*NUMSEROTYPES)),f[index + (i / (NUMLOC*NUMSEROTYPES))]);
                f[index + (i / (NUMLOC*NUMSEROTYPES))] = (-foi_on_susc_all_viruses) * y[index + (i / (NUMLOC*NUMSEROTYPES))];
            }
            else{
                if(i / (NUMLOC*NUMSEROTYPES) == 0){
                    foi_on_susc_all_viruses = 0.0;
                }
                foi_on_susc_all_viruses +=  gpu_params->foi_on_susc_all_viruses_eb[i]
                                            * stf
                                            * y[STARTI + (i % (NUMLOC*NUMSEROTYPES))];
//                printf(">= %d i = %d foi_on_susc_all_viruses_eb[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,i,gpu_params->foi_on_susc_all_viruses_eb[i]);
//                printf(">= %d i = %d y[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,STARTI + (i % (NUMLOC*NUMSEROTYPES)),y[STARTI + (i % (NUMLOC*NUMSEROTYPES))]);
//                printf(">= %d i = %d f[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,index + (i / (NUMLOC*NUMSEROTYPES)),f[index + (i / (NUMLOC*NUMSEROTYPES))]);
                f[index + (i / (NUMLOC*NUMSEROTYPES))] = (-foi_on_susc_all_viruses) * y[index + (i / (NUMLOC*NUMSEROTYPES))];
            }
        }
        for(int i = 0; i < NUMLOC*NUMSEROTYPES; i++){
            f[index + (i / NUMSEROTYPES)] += trr * y[ ((i + 1) * NUMR) - 1];
        }
    }
//        f[index] =  - (stf * gpu_params->beta[0] * y[12] * y[index])
//                    - (stf * gpu_params->beta[1] * y[13] * y[index])
//                    - (stf * gpu_params->beta[2] * y[14] * y[index])
//                    + (trr * y[3]) + (trr * y[7]) + (trr * y[11]);
//        block.sync();
    }
//    if(index == 0 || index == DIM - 1){
//        printf("        [function] y[%d] = %.20f f[%d] = %.20f\n",index,y[index],index,f[index]);
//    }

    return;
}
