#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_rk45.h"

__device__ int get_1d_index_from_5(const int loc,const  int vir,const  int stg,const  int l,const  int v){
    return loc*NUMSEROTYPES*NUMR*NUMLOC*NUMSEROTYPES + vir*NUMR*NUMLOC*NUMSEROTYPES + stg*NUMLOC*NUMSEROTYPES + l*NUMSEROTYPES + v;
}

__device__ int get_1d_index_start_from_3(const int loc,const  int vir,const  int stg){
    return loc*NUMSEROTYPES*NUMR*NUMLOC*NUMSEROTYPES + vir*NUMR*NUMLOC*NUMSEROTYPES + stg*NUMLOC*NUMSEROTYPES;
}

__device__ int get_1d_index_end_from_3(const int loc,const  int vir,const  int stg){
    return loc*NUMSEROTYPES*NUMR*NUMLOC*NUMSEROTYPES + vir*NUMR*NUMLOC*NUMSEROTYPES + stg*NUMLOC*NUMSEROTYPES + (NUMLOC*NUMSEROTYPES);
}

__device__ double get_sum_foi_sbe_from_1(const int index_1d, const int offset, GPU_Parameters* gpu_params){
//    printf("      sum_foi_sbe[%d] = %f\n",index_1d,gpu_params->sum_foi_sbe[index_1d]);
    return gpu_params->sum_foi_sbe[index_1d];
}

__device__ double get_sum_foi_sbe_from_5(const int loc,const  int vir,const  int stg,const  int l,const  int v, GPU_Parameters* gpu_params){
    return gpu_params->sum_foi_sbe[get_1d_index_from_5(loc, vir, stg, l, v)];
}

__device__ double get_pass1_y_I(const int index, const double y[]){
    return y[STARTI + index];
}

__device__ double get_sum_foi_sbe_from_3(const int loc, const int vir, const int stg,  const double y[], GPU_Parameters* gpu_params){
    cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
    double sum_foi_sbe = 0.0;
//    printf("  loc = %d vir = %d stg = %d sum from %d to %d\n",loc,vir,stg,get_1d_index_start_from_3(loc,vir,stg),get_1d_index_end_from_3(loc,vir,stg));
    for(int i = get_1d_index_start_from_3(loc,vir,stg); i < get_1d_index_end_from_3(loc,vir,stg); i++){
        sum_foi_sbe += get_sum_foi_sbe_from_1(i,i - get_1d_index_start_from_3(loc,vir,stg),gpu_params) * get_pass1_y_I(i - get_1d_index_start_from_3(loc,vir,stg),y);
//        printf("    loc = %d vir = %d stg = %d sum_foi_sbe index = %d y I index is %d y = %f\n",loc,vir,stg,i,STARTI + (i - get_1d_index_start_from_3(loc,vir,stg)),get_pass1_y_I(i - get_1d_index_start_from_3(loc,vir,stg),y));
    }
//    printf("  loc = %d vir = %d stg = %d sum_foi = %f\n",loc,vir,stg, sum_foi);
    block.sync();
    return sum_foi_sbe;
}

__device__
void gpu_func_test2(double t, const double y[], double f[], int index, void *params){

    //    printf("gpu_function start\n");
    // just to be safe, cast the void-pointer to convert it to a prms-pointer
    GPU_Parameters* gpu_params = (GPU_Parameters*) params;

//    if(index == 0){
//        printf("Here's the info on params: \n");
//        printf("beta1 = %1.9f \n", gpu_params->beta[0]);
//        printf("beta2 = %1.9f \n", gpu_params->beta[1]);
//        printf("beta3 = %1.9f \n", gpu_params->beta[2]);
//        printf("a = %1.3f \n", gpu_params->v_d[gpu_params->i_amp]);
//        printf("sigma_H1B = %1.3f \n", gpu_params->sigma[0][1]);
//        printf("sigma_BH3 = %1.3f \n", gpu_params->sigma[1][2]);
//        printf("sigma_H1H3 = %1.3f \n", gpu_params->sigma[0][2]);
//
//        printf("phis_length = %d\n",gpu_params->phis_d_length);
//        for(int i=0; i<gpu_params->phis_d_length; i++){
//            printf("phi = %5.1f \n", gpu_params->phis_d[i]);
//        }
//    }

    // the transition rate among R-classes
    double trr = ((double)NUMR) / gpu_params->v_d[gpu_params->i_immune_duration];

    double stf = gpu_params->phis_d_length == 0 ? 1.0 : gpu_params->stf_d[static_cast<int>(t)];
//    double stf = seasonal_transmission_factor(gpu_params,t);

//    R -  i = 0 f[0] = 0-R1a
//    R -  i = 1 f[1] = 0-R1b
//    R -  i = 2 f[2] = 0-R1c
//    R -  i = 3 f[3] = 0-R1d
//    R -  i = 4 f[4] = 0-R2a
//    R -  i = 5 f[5] = 0-R2b
//    R -  i = 6 f[6] = 0-R2c
//    R -  i = 7 f[7] = 0-R2d
//    R -  i = 8 f[8] = 0-R3a
//    R -  i = 9 f[9] = 0-R3b
//    R -  i = 10 f[10] = 0-R3c
//    R -  i = 11 f[11] = 0-R3d
//    I -  i = 12 f[12] = 0-I1
//    I -  i = 13 f[13] = 0-I2
//    I -  i = 14 f[14] = 0-I3
//    J -  i = 15 f[15] = 0-J1
//    J -  i = 16 f[16] = 0-J2
//    J -  i = 17 f[17] = 0-J3
//    S -  i = 18 f[18] = 0-S

    __shared__ bool step_I_done[DIM];
    __shared__ bool step_IJ_done[DIM];

//    printf("y[%d] = y %f\n",index,y[index]);
    if(index < STARTI){
//        int zDirection = i % zLength;
//        int yDirection = (i / zLength) % yLength;
//        int xDirection = i / (yLength * zLength);
        int loc = index / (NUMSEROTYPES * NUMR);
        int vir = (index / NUMR) % NUMSEROTYPES;
        int stg = index % NUMR;
//        printf("index = %d index = %d Loc %d R vir %d stg %d\n",index,index,loc,vir,stg);
//        if(index == 11)
        {
            f[ index ] = - trr * y[ index ];
            if(index % NUMR == 0){
    //            printf("  Index %d stg == 0\n",index);
                f[ index ] += gpu_params->v_d[gpu_params->i_nu] * y[ STARTI + NUMSEROTYPES*loc + vir ];
            }
            else{
                f[ index ] += trr * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg - 1 ];
            }
            double sum_foi_3 = get_sum_foi_sbe_from_3(loc,vir,stg,y,gpu_params);
            f[ index ] += ( - sum_foi_3)
                          * stf
                          * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ];
    //        printf("loc = %d vir = %d stg = %d flat y[%d] = %f\n",loc,vir,stg,index,y[ index ]);
    //        printf("loc = %d vir = %d stg = %d flat sum_foi = %f\n",loc,vir,stg,sum_foi_3);
    //        printf("loc = %d vir = %d stg = %d flat f[%d] = %f\n",loc,vir,stg,index,f[ index ]);
    //        printf("\n");
    //        if(index == STARTI - 1){
    //            printf("\n");
    //        }
        }
    }
    else if(index < STARTS){
        int loc = ((index - STARTJ) / (NUMSEROTYPES)) % NUMLOC;
        int vir = (index - NUMSEROTYPES*NUMR*NUMLOC) % NUMSEROTYPES;
//        printf("index = %d Loc %d I vir %d\n",index,loc,vir);
//        if(index == 29)
        {
            if(index < STARTJ){
                f[STARTI + NUMSEROTYPES * loc + vir] = 0.0;
//            f[STARTJ + NUMSEROTYPES * loc + vir] = 0.0;
                double foi_on_susc_single_virus = 0.0;
//                cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
                for(int l = 0; l < NUMLOC; l++){
                    double foi_on_susc_single_virus_eb = gpu_params->eta[loc][l]
                                                         * stf
                                                         * gpu_params->beta[vir];
                    foi_on_susc_single_virus += foi_on_susc_single_virus_eb * y[STARTI + NUMSEROTYPES * l + vir];
//                    printf("  loc = %d vir = %d l = %d y I index is %d y = %f\n",loc,vir,l,STARTI + NUMSEROTYPES * l + vir,y[STARTI + NUMSEROTYPES * l + vir]);
//                    printf("  loc = %d vir = %d l = %d flat foi_on_susc_single_virus_eb = %f\n",loc,vir,l,foi_on_susc_single_virus_eb);
//                    printf("  loc = %d vir = %d l = %d flat foi_on_susc_single_virus = %f\n",loc,vir,l,foi_on_susc_single_virus);
                }
                f[ STARTI + NUMSEROTYPES*loc + vir ] += y[ STARTS + loc ] * foi_on_susc_single_virus;
//            f[ STARTJ + NUMSEROTYPES*loc + vir ] += y[ STARTS + loc ] * foi_on_susc_single_virus;
//                printf("loc = %d vir = %d y I index is %d y = %f\n",loc,vir,STARTS + loc,y[STARTS + loc]);
//                printf("loc = %d vir = %d flat foi_on_susc_single_virus = %f\n",loc,vir,foi_on_susc_single_virus);
//                printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTI + NUMSEROTYPES*loc + vir,f[STARTI + NUMSEROTYPES*loc + vir]);
//            printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTJ + NUMSEROTYPES*loc + vir,f[STARTJ + NUMSEROTYPES*loc + vir]);
//                printf("\n");

                double inflow_from_recovereds = 0.0;
                for(int i = 0; i < NUMLOC*NUMSEROTYPES*NUMR; i++){
                    int multiplier = index - (NUMLOC*NUMSEROTYPES*NUMR);
                    int inflow_index = (multiplier)*NUMLOC*NUMSEROTYPES*NUMR + i;
                    inflow_from_recovereds +=   gpu_params->inflow_from_recovereds_sbe[inflow_index]
                                                * stf
                                                * y[STARTI + NUMSEROTYPES * loc + vir] * y[i];
//                    printf("loc = %d vir = %d i = %d y index = %d y = %f\n",loc,vir,i,STARTI + NUMSEROTYPES * loc + vir,y[STARTI + NUMSEROTYPES * loc + vir]);
//                    printf("loc = %d vir = %d i = %d y index = %d y = %f\n",loc,vir,i,i,y[i]);
//                    printf("loc = %d vir = %d i = %d multiplier = %d inflow_from_recovereds_sbe index = %d inflow_from_recovereds_sbe = %f\n",loc,vir,i,multiplier,inflow_index,gpu_params->inflow_from_recovereds_sbe[inflow_index]);
//                    printf("loc = %d vir = %d i = %d inflow_from_recovereds = %f\n",loc,vir,i,inflow_from_recovereds);
//                    printf("\n");
                }

//                printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTI + NUMSEROTYPES * loc + vir,f[STARTI + NUMSEROTYPES * loc + vir]);
//            printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTJ + NUMSEROTYPES * loc + vir,f[STARTJ + NUMSEROTYPES * loc + vir]);

                f[ STARTI + NUMSEROTYPES*loc + vir ] += inflow_from_recovereds;
//            f[ STARTJ + NUMSEROTYPES*loc + vir ] += inflow_from_recovereds;
//                printf("loc = %d vir = %d flat inflow_from_recovereds = %f\n", loc, vir, inflow_from_recovereds);
//                printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTI + NUMSEROTYPES * loc + vir,f[STARTI + NUMSEROTYPES * loc + vir]);
//            printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, STARTJ + NUMSEROTYPES * loc + vir,f[STARTJ + NUMSEROTYPES * loc + vir]);

//                step_I_done[index] = true;
//                block.sync();
            }
            else {//MOve J to here after I completed
//                cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
                //Wait for I completed
//                while(!step_I_done[index - NUMLOC*NUMSEROTYPES]){
//                    __syncthreads();
//                }
                f[index] = index;
                f[ index ] = f[ index - (NUMLOC * NUMSEROTYPES)];
//            // add the recovery rate - NOTE only for I-classes
                f[ index - (NUMLOC * NUMSEROTYPES) ] += - gpu_params->v_d[gpu_params->i_nu] * y[ index - (NUMLOC * NUMSEROTYPES) ];

//                printf("loc = %d vir = %d flat f[%d] I only = %f\n", loc, vir, index - (NUMLOC * NUMSEROTYPES),f[index - (NUMLOC * NUMSEROTYPES)]);
//                printf("f[%d] = f[%d]\n",index,index - (NUMLOC*NUMSEROTYPES));
//                step_IJ_done[index] = true;
//                block.sync();
            }
        }
    }
    else{
//        cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
        //Wait for IJ completed
//        while(!step_IJ_done[index - NUMLOC*NUMSEROTYPES]){
//            __syncthreads();
//        }
//    if(index == 36)
    {
        double foi_on_susc_all_viruses = 0.0;
        for(int i = 0; i < NUMLOC*NUMLOC*NUMSEROTYPES; i++){
            if(i < NUMLOC*NUMSEROTYPES){
                foi_on_susc_all_viruses +=  gpu_params->foi_on_susc_all_viruses_eb[i]
                                            * stf
                                            * y[STARTI + (i % (NUMLOC*NUMSEROTYPES))];
//                printf("< %d i = %d foi_on_susc_all_viruses_eb[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,i,gpu_params->foi_on_susc_all_viruses_eb[i]);
//                printf("< %d i = %d y[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,STARTI + (i % (NUMLOC*NUMSEROTYPES)),y[STARTI + (i % (NUMLOC*NUMSEROTYPES))]);
//                printf("< %d i = %d f[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,index + (i / (NUMLOC*NUMSEROTYPES)),f[index + (i / (NUMLOC*NUMSEROTYPES))]);
                f[index + (i / (NUMLOC*NUMSEROTYPES))] = (-foi_on_susc_all_viruses) * y[index + (i / (NUMLOC*NUMSEROTYPES))];
            }
            else{
                if(i / (NUMLOC*NUMSEROTYPES) == 0){
                    foi_on_susc_all_viruses = 0.0;
                }
                foi_on_susc_all_viruses +=  gpu_params->foi_on_susc_all_viruses_eb[i]
                                            * stf
                                            * y[STARTI + (i % (NUMLOC*NUMSEROTYPES))];
//                printf(">= %d i = %d foi_on_susc_all_viruses_eb[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,i,gpu_params->foi_on_susc_all_viruses_eb[i]);
//                printf(">= %d i = %d y[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,STARTI + (i % (NUMLOC*NUMSEROTYPES)),y[STARTI + (i % (NUMLOC*NUMSEROTYPES))]);
//                printf(">= %d i = %d f[%d] = %f\n",NUMLOC*NUMSEROTYPES,i,index + (i / (NUMLOC*NUMSEROTYPES)),f[index + (i / (NUMLOC*NUMSEROTYPES))]);
                f[index + (i / (NUMLOC*NUMSEROTYPES))] = (-foi_on_susc_all_viruses) * y[index + (i / (NUMLOC*NUMSEROTYPES))];
            }
        }
        for(int i = 0; i < NUMLOC*NUMSEROTYPES; i++){
            f[index + (i / NUMSEROTYPES)] += trr * y[ ((i + 1) * NUMR) - 1];
        }
    }
//        f[index] =  - (stf * gpu_params->beta[0] * y[12] * y[index])
//                    - (stf * gpu_params->beta[1] * y[13] * y[index])
//                    - (stf * gpu_params->beta[2] * y[14] * y[index])
//                    + (trr * y[3]) + (trr * y[7]) + (trr * y[11]);
//        block.sync();
    }
//    if(index == 0 || index == DIM - 1){
//        printf("        [function] y[%d] = %.20f f[%d] = %.20f\n",index,y[index],index,f[index]);
//    }

    return;
}


__device__
void gpu_func_test(double t, const double y[], double f[], int index, GPU_Parameters* gpu_params){

    //    printf("gpu_function start\n");
    // just to be safe, cast the void-pointer to convert it to a prms-pointer

//    if(index == 0){
//        printf("Here's the info on params: \n");
//        printf("beta1 = %1.9f \n", gpu_params->beta[0]);
//        printf("beta2 = %1.9f \n", gpu_params->beta[1]);
//        printf("beta3 = %1.9f \n", gpu_params->beta[2]);
//        printf("a = %1.3f \n", gpu_params->v_d[gpu_params->i_amp]);
//        printf("sigma_H1B = %1.3f \n", gpu_params->sigma[0][1]);
//        printf("sigma_BH3 = %1.3f \n", gpu_params->sigma[1][2]);
//        printf("sigma_H1H3 = %1.3f \n", gpu_params->sigma[0][2]);
//
//        printf("phis_length = %d\n",gpu_params->phis_d_length);
//        for(int i=0; i<gpu_params->phis_d_length; i++){
//            printf("phi = %5.1f \n", gpu_params->phis_d[i]);
//        }
//    }

    // the transition rate among R-classes
    double trr = ((double)NUMR) / gpu_params->v_d[gpu_params->i_immune_duration];
    double stf = gpu_params->phis_d_length == 0 ? 1.0 : gpu_params->stf_d[static_cast<int>(t)];

//    R -  i = 0 f[0] = 0-R1a
//    R -  i = 1 f[1] = 0-R1b
//    R -  i = 2 f[2] = 0-R1c
//    R -  i = 3 f[3] = 0-R1d
//    R -  i = 4 f[4] = 0-R2a
//    R -  i = 5 f[5] = 0-R2b
//    R -  i = 6 f[6] = 0-R2c
//    R -  i = 7 f[7] = 0-R2d
//    R -  i = 8 f[8] = 0-R3a
//    R -  i = 9 f[9] = 0-R3b
//    R -  i = 10 f[10] = 0-R3c
//    R -  i = 11 f[11] = 0-R3d
//    I -  i = 12 f[12] = 0-I1
//    I -  i = 13 f[13] = 0-I2
//    I -  i = 14 f[14] = 0-I3
//    J -  i = 15 f[15] = 0-J1
//    J -  i = 16 f[16] = 0-J2
//    J -  i = 17 f[17] = 0-J3
//    S -  i = 18 f[18] = 0-S

    __shared__ bool step_I_done[DIM];
    __shared__ bool step_IJ_done[DIM];

//    if(index < STARTS)
//    {
//        printf("[function] IN y[%d] = %.20f f[%d] = %.20f\n",index,y[index],index,f[index]);
//        if(index == 0){
//            printf("\n");
//        }
//    }

//    printf("y[%d] = y %f\n",index,y[index]);
    const unsigned int START_I  = int(STARTI);
    const unsigned int START_J  = int(STARTJ);
    const unsigned int START_S  = int(STARTS);
    const unsigned int NUM_LOC  = int(NUMLOC);
    const unsigned int NUM_SEROTYPES  = int(NUMSEROTYPES);
    const unsigned int NUM_R  = int(NUMR);

    if(index < START_I){
//        int zDirection = i % zLength;
//        int yDirection = (i / zLength) % yLength;
//        int xDirection = i / (yLength * zLength);
        int loc = index / (NUM_SEROTYPES * NUM_R);
        int vir = (index / NUM_R) % NUM_SEROTYPES;
        int stg = index % NUM_R;
//        printf("index = %d index = %d Loc %d R vir %d stg %d\n",index,index,loc,vir,stg);
        f[ index ] = - trr * y[ index ];
        if(index % NUM_R == 0){
//            printf("  Index %d stg == 0\n",index);
            f[ index ] += gpu_params->v_d[gpu_params->i_nu] * y[ START_I + NUM_SEROTYPES*loc + vir ];
        }
        else{
            f[ index ] += trr * y[ NUM_SEROTYPES*NUM_R*loc + NUM_R*vir + stg - 1 ];
        }
        double sum_foi = 0.0;
//        for(int l = 0; l < NUM_LOC; l++){
//            sum_foi += gpu_params->sigma[vir][0] * gpu_params->beta[0] * stf * gpu_params->eta[loc][l] * y[START_I + NUM_SEROTYPES*l + 0] +
//                       gpu_params->sigma[vir][1] * gpu_params->beta[1] * stf * gpu_params->eta[loc][l] * y[START_I + NUM_SEROTYPES*l + 1] +
//                       gpu_params->sigma[vir][2] * gpu_params->beta[2] * stf * gpu_params->eta[loc][l] * y[START_I + NUM_SEROTYPES*l + 2];
//        }
        sum_foi =   gpu_params->sigma[vir][0] * gpu_params->beta[0] * stf * gpu_params->eta[0][0] * y[12] +
                    gpu_params->sigma[vir][1] * gpu_params->beta[1] * stf * gpu_params->eta[0][0] * y[13] +
                    gpu_params->sigma[vir][2] * gpu_params->beta[2] * stf * gpu_params->eta[0][0] * y[14];
        f[index] +=  -(sum_foi) * y[index];
    }
    else if(index < START_S){
        int vir = (index - NUM_SEROTYPES*NUM_R*NUM_LOC) % NUM_SEROTYPES;
        if(index < START_J){
            int loc = ((index - START_J) / (NUM_SEROTYPES)) % NUM_LOC;
//        printf("index = %d Loc %d I vir %d\n",index,loc,vir);
            f[ index ] = 0.0;
            double foi_on_susc_single_virus = 0.0;
//            cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//            for(int l = 0; l<NUM_LOC; l++){
//                foi_on_susc_single_virus += gpu_params->eta[loc][l]
//                                            * stf
//                                            * gpu_params->beta[vir]
//                                            * y[START_I + NUM_SEROTYPES * l + vir];
//            }
            foi_on_susc_single_virus =  gpu_params->eta[0][0] * stf * gpu_params->beta[vir] * y[index];

            f[ index ] += y[ START_S + loc ] * foi_on_susc_single_virus;

            double inflow_from_recovereds = 0.0;
//            for(int l = 0; l < NUM_LOC; l++){
//                for(int v = 0; v < NUM_SEROTYPES; v++){
//                    inflow_from_recovereds +=   gpu_params->sigma[vir][v] * stf * gpu_params->beta[vir] * gpu_params->eta[loc][l] * y[START_I + NUM_SEROTYPES*l + vir] * y[NUM_SEROTYPES*NUM_R*loc + NUM_R*v + 0] +
//                                                gpu_params->sigma[vir][v] * stf * gpu_params->beta[vir] * gpu_params->eta[loc][l] * y[START_I + NUM_SEROTYPES*l + vir] * y[NUM_SEROTYPES*NUM_R*loc + NUM_R*v + 1] +
//                                                gpu_params->sigma[vir][v] * stf * gpu_params->beta[vir] * gpu_params->eta[loc][l] * y[START_I + NUM_SEROTYPES*l + vir] * y[NUM_SEROTYPES*NUM_R*loc + NUM_R*v + 2] +
//                                                gpu_params->sigma[vir][v] * stf * gpu_params->beta[vir] * gpu_params->eta[loc][l] * y[START_I + NUM_SEROTYPES*l + vir] * y[NUM_SEROTYPES*NUM_R*loc + NUM_R*v + 3];
//                }
//            }
            inflow_from_recovereds +=   gpu_params->sigma[vir][0] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[0] +
                                        gpu_params->sigma[vir][0] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[1] +
                                        gpu_params->sigma[vir][0] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[2] +
                                        gpu_params->sigma[vir][0] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[3] +
                                        gpu_params->sigma[vir][1] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[4] +
                                        gpu_params->sigma[vir][1] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[5] +
                                        gpu_params->sigma[vir][1] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[6] +
                                        gpu_params->sigma[vir][1] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[7] +
                                        gpu_params->sigma[vir][2] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[8] +
                                        gpu_params->sigma[vir][2] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[9] +
                                        gpu_params->sigma[vir][2] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[10] +
                                        gpu_params->sigma[vir][2] * stf * gpu_params->beta[vir] * gpu_params->eta[0][0] * y[index] * y[11];
            f[ index ] += inflow_from_recovereds;
//            printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, index,f[index]);
//            step_I_done[index] = true;
//            block.sync();
        }
        else {
//            int loc = ((index - START_S) / (NUM_SEROTYPES)) % NUM_LOC;
//            cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
            //Wait for I completed
//            while(!step_I_done[index - NUM_LOC*NUM_SEROTYPES]){
//                __syncthreads();
//            }
            f[index] = index;
            f[index] = f[index - (NUM_LOC * NUM_SEROTYPES)];
//            // add the recovery rate - NOTE only for I-classes
            f[index - (NUM_LOC * NUM_SEROTYPES)] += - gpu_params->v_d[gpu_params->i_nu] * y[index - (NUM_LOC * NUM_SEROTYPES)];
//            printf("loc = %d vir = %d flat f[%d] = %f\n", loc, vir, index,f[index]);
//            printf("loc = %d vir = %d flat f[%d] I only = %f\n", loc, vir, index - (NUM_LOC * NUM_SEROTYPES),f[index - (NUM_LOC * NUM_SEROTYPES)]);
//            printf("f[%d] = f[%d]\n",index,index - (NUM_LOC*NUM_SEROTYPES));
//            step_IJ_done[index] = true;
//            block.sync();
        }
    }
    else{
////        cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//        //Wait for IJ completed
////        cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
////        while(!step_IJ_done[index - NUM_LOC*NUM_SEROTYPES]){
////            __syncthreads();
////        }
//        unsigned int loc = index - START_S;
////        printf("index = %d Loc %d\n",index,loc);
//        double foi_on_susc_all_viruses = 0.0;
//        for(int l = 0; l < NUM_LOC; l++) {
//            foi_on_susc_all_viruses += gpu_params->eta[loc][l] * stf * gpu_params->beta[0] * y[START_I + NUM_SEROTYPES*l + 0] +
//                                       gpu_params->eta[loc][l] * stf * gpu_params->beta[1] * y[START_I + NUM_SEROTYPES*l + 1] +
//                                       gpu_params->eta[loc][l] * stf * gpu_params->beta[2] * y[START_I + NUM_SEROTYPES*l + 2];
////            printf("loop l-v index %d loc %d foi_on_susc_all_viruses = %f\n",index,loc,foi_on_susc_all_viruses);
//        }
////        printf("index %d loc %d foi_on_susc_all_viruses = %f\n",index,loc,foi_on_susc_all_viruses);
////        printf("index %d loc %d y[%d] = %f\n",index,loc,index,y[index]);
//        f[ index ] = ( - foi_on_susc_all_viruses ) * y[ index ];
////        printf("index %d loc %d f[%d] = %f\n",index,loc,index,f[index]);
//        for(int vir = 0; vir<NUM_SEROTYPES; vir++)
//        {
//            // add to dS/dt the inflow of recovereds from the final R-stage
//            f[ index ] += trr * y[ NUM_SEROTYPES*NUM_R*(loc) + NUM_R*vir + (NUM_R - 1) ]; // "NUM_R-1" gets you the final R-stage only
////            printf("loop vir index %d loc %d f[%d] = %f\n",index,loc,index,f[index]);
//        }
////        block.sync();

        f[index] = -(stf * gpu_params->beta[0] * y[12] * y[index]) -
                (stf * gpu_params->beta[1] * y[13] * y[index]) -
                (stf * gpu_params->beta[2] * y[14] * y[index]) +
                (trr * y[3]) + (trr * y[7]) + (trr * y[11]);

    }

//    if(index < START_S)
//    {
//        printf("[function] OUT y[%d] = %.20f f[%d] = %.20f\n",index,y[index],index,f[index]);
//        if(index == 0){
//            printf("\n");
//        }
//    }

    return;
}
