#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu_rk45.h"

__host__ __device__
void seasonal_transmission_factor(GPU_Parameters* gpu_params, double t, double &factor )
{
    /*


        We're gonna make this thing go for 40 years. 30 years of burn in and 10 years of real modeling.
        We're creating a "10-year model cycle" and need the code below to find a time point's "place" in the "cycle"
        modulus (denoted with % in C++) only works with integers, so need the acrobatics below

     */

    // This is some code that's needed to create the 10-year "cycles" in transmission.

    if(gpu_params->phis_d == nullptr){
        factor = 1.0;
        return;
    }

    int x = (int)t; // This is now to turn a double into an integer
    double remainder = t - (double)x;
    int xx = x % 3650; // int xx = x % NUMDAYSOUTPUT;
    double yy = (double)xx + remainder;
    // put yy into the sine function, let it return the beta value
    t = yy;
    double sine_function_value = 0.0;

    int phis_length = sizeof(gpu_params->phis_d)/ sizeof(gpu_params->phis_d[0]);

    for(int i=0; i<phis_length; i++)
    {
        if( fabs( t - gpu_params->phis_d[i] ) < (gpu_params->v_d[gpu_params->i_epidur] / 2))
        {
            // sine_function_value = sin( 2.0 * 3.141592653589793238 * (phis[i]-t+91.25) / 365.0 );
            sine_function_value = sin( 2.0 * 3.141592653589793238 * (gpu_params->phis_d[i]-t+(gpu_params->v_d[gpu_params->i_epidur] / 2)) / (gpu_params->v_d[gpu_params->i_epidur] * 2));
            // printf("\n\t\t\t %1.3f %1.3f %1.3f \n\n", t, phis[i], sine_function_value );
        }
    }
    factor = 1.0 + gpu_params->v_d[gpu_params->i_amp] * sine_function_value;
    return;
}

__device__
void gpu_func_test(double t, const double y[], double f[],
                   double* sum_foi, double* foi_on_susc_single_virus,
                   double* inflow_from_recovereds, double* foi_on_susc_all_viruses,
                   int index, void *params){
    //    printf("gpu_function start\n");
    // just to be safe, cast the void-pointer to convert it to a prms-pointer

    GPU_Parameters* gpu_params = (GPU_Parameters*) params;

    // everything will be indexed by location (loc), the infecting subtype/serotype (vir), and the stage of recovery (stg) in the R-classes
    int loc, vir, stg;

    // force of infection
    // double foi = gpu_params->v_d[i_beta] * y[NUMR+2];

    // the transition rate among R-classes
    double trr = ((double)NUMR) / gpu_params->v_d[gpu_params->i_immune_duration];

    double stf = 0.0;

    //for(int k=0; k<DIM; k++) f[k] = 0.0;

    //
    // ###  1.  COMPUTE THE FORCES OF INFECTION (NOTE maybe this is not necessary)
    //

    // force of infection on location loc, on immune status i, by virus vir
    /*double foi_partial[NUMLOC][NUMSEROTYPES][NUMSEROTYPES];
    for(loc=0; loc<NUMLOC; loc++)
    {
        for(vir=0; vir<NUMSEROTYPES; vir++)
        {
            for(stg=0; stg<NUMR; stg++)
            {

            }
        }
    }*/


    //
    // ###  2.  WRITE DOWN THE DERIVATIVES FOR ALL THE RECOVERED CLASSES
    //
//    cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//    for(loc=0; loc<NUMLOC; loc++)
//    {
//        for(vir=0; vir<NUMSEROTYPES; vir++)
//        {
//            for(stg=0; stg<NUMR; stg++)
//            {
//                if(index == NUMSEROTYPES*NUMR*loc + NUMR*vir + stg){
//                    // first add the rate at which individuals are transitioning out of the R class
//                    f[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ] = - trr * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ];
//
//                    // now add the rates of individuals coming in
//                    if( stg == 0 )
//                    {
//                        // if this is the first R-class, add the recovery term for individuals coming from I
//                        f[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ] += gpu_params->v_d[gpu_params->i_nu] * y[ STARTI + NUMSEROTYPES*loc + vir ];
//                    }
//                    else
//                    {
//                        // if this is not the first R-class, add a simple transition from the previous R-stage
//                        f[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ] += trr * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg - 1 ];
//                    }
//
//                    // now sum over all locations and serotypes to get the force of infection that is removing
//                    // individuals from this R-class
//
//                    sum_foi[index] = 0.0;
//                    for(int l=0; l<NUMLOC; l++) {
//                        for (int v = 0; v < NUMSEROTYPES; v++) {
//                            seasonal_transmission_factor(gpu_params,t,stf);
//                            sum_foi[index] += gpu_params->sigma[vir][v]
//                                       * stf
//                                       * gpu_params->beta[v] * gpu_params->eta[loc][l] * y[STARTI + NUMSEROTYPES * l + v];
//                        }
//                    }
//                    // now add the term to dR/dt that accounts for the force of infection removing some R-individuals
//                    f[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ] += ( -sum_foi[index] ) * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ];
//                }
//            }
//        }
//    }
//    block.sync();

    //
    // ###  3.  WRITE DOWN THE DERIVATIVES FOR ALL THE INFECTED CLASSES and the J-CLASSES
    //
    cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
    for(loc=0; loc<NUMLOC; loc++)
    {
        for(vir=0; vir<NUMSEROTYPES; vir++)
        {
            // initialize these derivatives to zero
            f[ STARTI + NUMSEROTYPES*loc + vir ] = 0.0;
            f[ STARTJ + NUMSEROTYPES*loc + vir ] = 0.0;

////            // sum over locations to get the force of infection of virus vir on susceptibles in location loc
//            foi_on_susc_single_virus[index] = 0.0;
//            for(int l=0; l<NUMLOC; l++){
//                seasonal_transmission_factor(gpu_params,t,stf);
//                foi_on_susc_single_virus[index] +=
//                        gpu_params->eta[loc][l]
//                        * stf
//                        * gpu_params->beta[vir] * y[STARTI + NUMSEROTYPES * l + vir];
//            }
//            // add the in-flow of new infections from the susceptible class
//            f[ STARTI + NUMSEROTYPES*loc + vir ] += y[ STARTS + loc ] * foi_on_susc_single_virus[index];
//            f[ STARTJ + NUMSEROTYPES*loc + vir ] += y[ STARTS + loc ] * foi_on_susc_single_virus[index];

//            // sum over locations and different types of recovered individuals to get the inflow of recovered
//            // individuals that are becoming re-infected
//            inflow_from_recovereds[index] = 0.0;
//            for(int l=0; l<NUMLOC; l++) {          // sum over locations
//                for (int v = 0; v < NUMSEROTYPES; v++) {  // sum over recent immunity
//                    for (int s = 0; s < NUMR; s++) {    // sum over R stage
//                        seasonal_transmission_factor(gpu_params,t,stf);
//                        inflow_from_recovereds[index] +=
//                                gpu_params->sigma[vir][v]
//                                * stf
//                                * gpu_params->beta[vir] * gpu_params->eta[loc][l] *
//                                y[STARTI + NUMSEROTYPES * l + vir] * y[NUMSEROTYPES * NUMR * loc + NUMR * v + s];
//                    }
//                }
//            }
//            // add the in-flow of new infections from the recovered classes (all histories, all stages)
//            f[ STARTI + NUMSEROTYPES*loc + vir ] += inflow_from_recovereds[index];
//            f[ STARTJ + NUMSEROTYPES*loc + vir ] += inflow_from_recovereds[index];

            // add the recovery rate - NOTE only for I-classes
            f[ STARTI + NUMSEROTYPES*loc + vir ] += - gpu_params->v_d[gpu_params->i_nu] * y[ STARTI + NUMSEROTYPES*loc + vir ];

        }
    }
    block.sync();
//    //
//    // ###  4.  WRITE DOWN THE DERIVATIVES FOR ALL THE SUSCEPTIBLE CLASSES
//    //
//    for(loc=0; loc<NUMLOC; loc++)
//    {
//        // compute the force of infection of all viruses at all locations on the susceptibles at the location loc
//        foi_on_susc_all_viruses[index] = 0.0;
//        for(int l=0; l<NUMLOC; l++) {
//            for (int v = 0; v < NUMSEROTYPES; v++) {
//                seasonal_transmission_factor(gpu_params,t,stf);
//                foi_on_susc_all_viruses[index] +=
//                        gpu_params->eta[loc][l]
//                        * stf
//                        * gpu_params->beta[v] * y[STARTI + NUMSEROTYPES * l + v];
//            }
//        }
//        // add to ODE dS/dt equation the removal of susceptibles by all types of infection
//        f[ STARTS + loc ] = ( - foi_on_susc_all_viruses[index] ) * y[ STARTS + loc ];
//
//        // now loop through all the recovered classes in this location (different histories, final stage only)
//        for(int vir=0; vir<NUMSEROTYPES; vir++)
//        {
//            // add to dS/dt the inflow of recovereds from the final R-stage
//            f[ STARTS + loc ] += trr * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + (NUMR-1) ]; // "NUMR-1" gets you the final R-stage only
//        }
//    }
    return;
}

//__host__ __device__
//void gpu_func_test(double t, const double y[], double f[], int index, void *params){
//    const double m = 5.2;		// Mass of pendulum
//    const double g = -9.81;		// g
//    const double l = 2;		// Length of pendulum
//    const double A = 0.5;		// Amplitude of driving force
//    const double wd = 1;		// Angular frequency of driving force
//    const double b = 0.5;		// Damping coefficient
//
////    printf("before y[%d] = %.10f f[%d] = %.10f\n",index,y[index],index,f[index]);
////    f[0] = y[1];
////    f[1] = -(g / l) * sin(y[0]) + (A * cos(wd * t) - b * y[1]) / (m * l * l);
//    f[index] = -(g / l) * sin(y[index]) + (A * cos(wd * t) - b * y[index]) / (m * l * l);
////    printf("after y[%d] = %.10f f[%d] = %.10f\n",index,y[index],index,f[index]);
//    return;
//}
