
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <complex>
#include "hip/hip_complex.h"

#define N  (2)
#define M  (4)

typedef std::complex<float> T;

__global__ void print_device_matrix (hipComplex** mat)
{
    printf ("matrix on device:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            printf ("(%f, %f)  ", hipCrealf (mat[i][j]), hipCimagf (mat[i][j]));
        }
        printf ("\n");
    }
}

int test_cuda()
{
    /* allocate host "matrix" */
    T **mat = (T**)malloc (N * sizeof (mat[0]));
    for (int i = 0; i < N; i++) {
        mat[i] = (T *)malloc (M * sizeof (mat[0][0]));
    }

    /* fill in host "matrix" */
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            mat[i][j] = T (float(i)+1, float(j)+1);
        }
    }

    /* print host "matrix" */
    printf ("matrix on host:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            printf ("(%f, %f)  ", real(mat[i][j]), imag(mat[i][j]));
        }
        printf ("\n");
    }

    /* allocate device "matrix" */
    T **tmp = (T**)malloc (N * sizeof (tmp[0]));
    for (int i = 0; i < N; i++) {
        hipMalloc ((void **)&tmp[i], M * sizeof (tmp[0][0]));
    }
    hipComplex **matD = 0;
    hipMalloc ((void **)&matD, N * sizeof (matD[0]));

    /* copy "matrix" from host to device */
    hipMemcpy (matD, tmp, N * sizeof (matD[0]), hipMemcpyHostToDevice);
    for (int i = 0; i < N; i++) {
        hipMemcpy (tmp[i], mat[i], M * sizeof (matD[0][0]), hipMemcpyHostToDevice);
    }
    free (tmp);

    /* print device "matrix" */
    print_device_matrix<<<1,1>>> (matD);

    /* free host "matrix" */
    for (int i = 0; i < N; i++) {
        free (mat[i]);
    }
    free (mat);

    /* free device "matrix" */
    tmp = (T**)malloc (N * sizeof (tmp[0]));
    hipMemcpy (tmp, matD, N * sizeof (matD[0]), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        hipFree (tmp[i]);
    }
    free (tmp);
    hipFree (matD);

    return EXIT_SUCCESS;
}