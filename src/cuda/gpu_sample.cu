
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <complex>
#include "hip/hip_complex.h"

#include <stdio.h>

//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//


#include <iostream>
#include <math.h>

#define N 100

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int test_cuda_1()
{
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    add<<<1, 1>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}

#define N2  (2)
#define M  (4)

typedef std::complex<float> T;

__global__ void print_device_matrix (hipComplex** mat)
{
    printf ("matrix on device:\n");
    for (int i = 0; i < N2; i++) {
        for (int j = 0; j < M; j++) {
            printf ("(%f, %f)  ", hipCrealf (mat[i][j]), hipCimagf (mat[i][j]));
        }
        printf ("\n");
    }
}

int test_cuda_2()
{
    /* allocate host "matrix" */
    T **mat = (T**)malloc (N2 * sizeof (mat[0]));
    for (int i = 0; i < N2; i++) {
        mat[i] = (T *)malloc (M * sizeof (mat[0][0]));
    }

    /* fill in host "matrix" */
    for (int i = 0; i < N2; i++) {
        for (int j = 0; j < M; j++) {
            mat[i][j] = T (float(i)+1, float(j)+1);
        }
    }

    /* print host "matrix" */
    printf ("matrix on host:\n");
    for (int i = 0; i < N2; i++) {
        for (int j = 0; j < M; j++) {
            printf ("(%f, %f)  ", real(mat[i][j]), imag(mat[i][j]));
        }
        printf ("\n");
    }

    /* allocate device "matrix" */
    T **tmp = (T**)malloc (N2 * sizeof (tmp[0]));
    for (int i = 0; i < N2; i++) {
        hipMalloc ((void **)&tmp[i], M * sizeof (tmp[0][0]));
    }
    hipComplex **matD = 0;
    hipMalloc ((void **)&matD, N2 * sizeof (matD[0]));

    /* copy "matrix" from host to device */
    hipMemcpy (matD, tmp, N2 * sizeof (matD[0]), hipMemcpyHostToDevice);
    for (int i = 0; i < N2; i++) {
        hipMemcpy (tmp[i], mat[i], M * sizeof (matD[0][0]), hipMemcpyHostToDevice);
    }
    free (tmp);

    /* print device "matrix" */
    print_device_matrix<<<1,1>>> (matD);

    /* free host "matrix" */
    for (int i = 0; i < N2; i++) {
        free (mat[i]);
    }
    free (mat);

    /* free device "matrix" */
    tmp = (T**)malloc (N2 * sizeof (tmp[0]));
    hipMemcpy (tmp, matD, N2 * sizeof (matD[0]), hipMemcpyDeviceToHost);
    for (int i = 0; i < N2; i++) {
        hipFree (tmp[i]);
    }
    free (tmp);
    hipFree (matD);

    return EXIT_SUCCESS;
}