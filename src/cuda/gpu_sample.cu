
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <complex>
#include "hip/hip_complex.h"

#include <stdio.h>

#include <stdio.h>

//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

#define N 10

__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int test_cuda_1() {
    //
    // Create int arrays on the CPU.
    // ('h' stands for "host".)
    //
    int ha[N], hb[N];

    //
    // Create corresponding int arrays on the GPU.
    // ('d' stands for "device".)
    //
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    //
    // Initialise the input data on the CPU.
    //
    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }

    //
    // Copy input data to array on GPU.
    //
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    //
    // Launch GPU code with N threads, one per
    // array element.
    //
    add<<<N, 1>>>(da, db);

    //
    // Copy output array from GPU back to CPU.
    //
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}

#define N2  (2)
#define M  (4)

typedef std::complex<float> T;

__global__ void print_device_matrix (hipComplex** mat)
{
    printf ("matrix on device:\n");
    for (int i = 0; i < N2; i++) {
        for (int j = 0; j < M; j++) {
            printf ("(%f, %f)  ", hipCrealf (mat[i][j]), hipCimagf (mat[i][j]));
        }
        printf ("\n");
    }
}

int test_cuda_2()
{
    /* allocate host "matrix" */
    T **mat = (T**)malloc (N2 * sizeof (mat[0]));
    for (int i = 0; i < N2; i++) {
        mat[i] = (T *)malloc (M * sizeof (mat[0][0]));
    }

    /* fill in host "matrix" */
    for (int i = 0; i < N2; i++) {
        for (int j = 0; j < M; j++) {
            mat[i][j] = T (float(i)+1, float(j)+1);
        }
    }

    /* print host "matrix" */
    printf ("matrix on host:\n");
    for (int i = 0; i < N2; i++) {
        for (int j = 0; j < M; j++) {
            printf ("(%f, %f)  ", real(mat[i][j]), imag(mat[i][j]));
        }
        printf ("\n");
    }

    /* allocate device "matrix" */
    T **tmp = (T**)malloc (N2 * sizeof (tmp[0]));
    for (int i = 0; i < N2; i++) {
        hipMalloc ((void **)&tmp[i], M * sizeof (tmp[0][0]));
    }
    hipComplex **matD = 0;
    hipMalloc ((void **)&matD, N2 * sizeof (matD[0]));

    /* copy "matrix" from host to device */
    hipMemcpy (matD, tmp, N2 * sizeof (matD[0]), hipMemcpyHostToDevice);
    for (int i = 0; i < N2; i++) {
        hipMemcpy (tmp[i], mat[i], M * sizeof (matD[0][0]), hipMemcpyHostToDevice);
    }
    free (tmp);

    /* print device "matrix" */
    print_device_matrix<<<1,1>>> (matD);

    /* free host "matrix" */
    for (int i = 0; i < N2; i++) {
        free (mat[i]);
    }
    free (mat);

    /* free device "matrix" */
    tmp = (T**)malloc (N2 * sizeof (tmp[0]));
    hipMemcpy (tmp, matD, N2 * sizeof (matD[0]), hipMemcpyDeviceToHost);
    for (int i = 0; i < N2; i++) {
        hipFree (tmp[i]);
    }
    free (tmp);
    hipFree (matD);

    return EXIT_SUCCESS;
}