
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <complex>
#include "hip/hip_complex.h"

#include <stdio.h>

//
// Nearly minimal CUDA example.
// Compile with:
//
// nvcc -o example example.cu
//

//
// A function marked __global__
// runs on the GPU but can be called from
// the CPU.
//
// This function multiplies the elements of an array
// of ints by 2.
//
// The entire computation can be thought of as running
// with one thread per array element with blockIdx.x
// identifying the thread.
//
// The comparison i<N is because often it isn't convenient
// to have an exact 1-1 correspondence between threads
// and array elements. Not strictly necessary here.
//
// Note how we're mixing GPU and CPU code in the same source
// file. An alternative way to use CUDA is to keep
// C/C++ code separate from CUDA code and dynamically
// compile and load the CUDA code at runtime, a little
// like how you compile and load OpenGL shaders from
// C/C++ code.
//

#define N 1

__global__
void add(int *a, int *b) {
    int i = threadIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int test_cuda_1() {
    //
    // Create int arrays on the CPU.
    // ('h' stands for "host".)
    //
    int ha[N], hb[N];

    //
    // Create corresponding int arrays on the GPU.
    // ('d' stands for "device".)
    //
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    //
    // Initialise the input data on the CPU.
    //
    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }

    //
    // Copy input data to array on GPU.
    //
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    //
    // Launch GPU code with N threads, one per
    // array element.
    //
    add<<<N, 1>>>(da, db);

    //
    // Copy output array from GPU back to CPU.
    //
    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    //
    // Free up the arrays on the GPU.
    //
    hipFree(da);
    hipFree(db);

    return 0;
}

#define N2  (2)
#define M  (4)

typedef std::complex<float> T;

__global__ void print_device_matrix (hipComplex** mat)
{
    printf ("matrix on device:\n");
    for (int i = 0; i < N2; i++) {
        for (int j = 0; j < M; j++) {
            printf ("(%f, %f)  ", hipCrealf (mat[i][j]), hipCimagf (mat[i][j]));
        }
        printf ("\n");
    }
}

int test_cuda_2()
{
    /* allocate host "matrix" */
    T **mat = (T**)malloc (N2 * sizeof (mat[0]));
    for (int i = 0; i < N2; i++) {
        mat[i] = (T *)malloc (M * sizeof (mat[0][0]));
    }

    /* fill in host "matrix" */
    for (int i = 0; i < N2; i++) {
        for (int j = 0; j < M; j++) {
            mat[i][j] = T (float(i)+1, float(j)+1);
        }
    }

    /* print host "matrix" */
    printf ("matrix on host:\n");
    for (int i = 0; i < N2; i++) {
        for (int j = 0; j < M; j++) {
            printf ("(%f, %f)  ", real(mat[i][j]), imag(mat[i][j]));
        }
        printf ("\n");
    }

    /* allocate device "matrix" */
    T **tmp = (T**)malloc (N2 * sizeof (tmp[0]));
    for (int i = 0; i < N2; i++) {
        hipMalloc ((void **)&tmp[i], M * sizeof (tmp[0][0]));
    }
    hipComplex **matD = 0;
    hipMalloc ((void **)&matD, N2 * sizeof (matD[0]));

    /* copy "matrix" from host to device */
    hipMemcpy (matD, tmp, N2 * sizeof (matD[0]), hipMemcpyHostToDevice);
    for (int i = 0; i < N2; i++) {
        hipMemcpy (tmp[i], mat[i], M * sizeof (matD[0][0]), hipMemcpyHostToDevice);
    }
    free (tmp);

    /* print device "matrix" */
    print_device_matrix<<<1,1>>> (matD);

    /* free host "matrix" */
    for (int i = 0; i < N2; i++) {
        free (mat[i]);
    }
    free (mat);

    /* free device "matrix" */
    tmp = (T**)malloc (N2 * sizeof (tmp[0]));
    hipMemcpy (tmp, matD, N2 * sizeof (matD[0]), hipMemcpyDeviceToHost);
    for (int i = 0; i < N2; i++) {
        hipFree (tmp[i]);
    }
    free (tmp);
    hipFree (matD);

    return EXIT_SUCCESS;
}