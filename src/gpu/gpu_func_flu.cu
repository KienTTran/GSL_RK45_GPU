#include <hip/hip_runtime.h>
#include "gpu_flu.cuh"
#include "gpu_ode.cuh"


__device__
void gpu_func_flu(double t, const double y[], double f[], double stf, int index, FluParameters* flu_params){
    // everything will be indexed by location (loc), the infecting subtype/serotype (vir), and the stage of recovery (stg) in the R-classes
    int loc, vir, stg;

//    if((NUMODE == 1  || (index > 0 && index % (NUMODE / 2) == 0)) && t == 0){
//        printf("gpu_func_flu flu_params: \n");
//        printf("  beta1 = %1.9f \n", flu_params->beta[index*NUMSEROTYPES + 0]);
//        printf("  beta2 = %1.9f \n", flu_params->beta[index*NUMSEROTYPES + 1]);
//        printf("  beta3 = %1.9f \n", flu_params->beta[index*NUMSEROTYPES + 2]);
//        printf("  v_d_i_amp = %1.5f \n", flu_params->v_d_i_amp);
//        printf("  sigma_H1B = %1.5f \n", flu_params->sigma2d[0][1]);
//        printf("  sigma_BH3 = %1.5f \n", flu_params->sigma2d[1][2]);
//        printf("  sigma_H1H3 = %1.5f \n", flu_params->sigma2d[0][2]);
//        printf("  eta = %1.5f \n", flu_params->eta[0][0]);
//        printf("  eta = %1.5f \n", flu_params->eta[1][1]);
//        printf("  eta = %1.5f \n", flu_params->eta[2][2]);
//        printf("  trr = %1.5f \n", flu_params->trr);
//        printf("  v_d_i_nu = %1.5f \n", flu_params->v_d_i_nu);
//        printf("phis_length = %d\n",SAMPLE_PHI_LENGTH);
//        for(int i=0; i<SAMPLE_PHI_LENGTH; i++){
//            printf("  phi = %5.1f \n", flu_params->phi[index*SAMPLE_PHI_LENGTH + i]);
//        }
//    }

//    for(int j=0;j<DIM;j++) {
////        if(j == 0 || j == DIM -1)
//        {
//            printf("[function] IN y[%d] = %f \n",j,y[j]);
//        }
//    }
//    printf("\n");

    //for(int k=0; k<DIM; k++) f[k] = 0.0;

    //
    // ###  1.  COMPUTE THE FORCES OF INFECTION (NOTE maybe this is not necessary)
    //

    // force of infection on location loc, on immune status i, by virus vir
    /*double foi_partial[NUMLOC][NUMSEROTYPES][NUMSEROTYPES];
    for(loc=0; loc<NUMLOC; loc++)
    {
        for(vir=0; vir<NUMSEROTYPES; vir++)
        {
            for(stg=0; stg<NUMR; stg++)
            {

            }
        }
    }*/


    //
    // ###  2.  WRITE DOWN THE DERIVATIVES FOR ALL THE RECOVERED CLASSES
    //


    for(loc=0; loc<NUMLOC; loc++)
    {
        for(vir=0; vir<NUMSEROTYPES; vir++)
        {
            for(stg=0; stg<NUMR; stg++)
            {
                // first add the rate at which individuals are transitioning out of the R class
                f[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ] = - flu_params->trr * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ];

                // now add the rates of individuals coming in
                if( stg==0 )
                {
                    // if this is the first R-class, add the recovery term for individuals coming from I
                    f[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ] += flu_params->v_d_i_nu * y[ STARTI + NUMSEROTYPES*loc + vir ];
                }
                else
                {
                    // if this is not the first R-class, add a simple transition from the previous R-stage
                    f[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ] += flu_params->trr * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg - 1 ];
                }

                // now sum over all locations and serotypes to get the force of infection that is removing
                // individuals from this R-class
                double sum_foi = 0.0;
                for(int l=0; l<NUMLOC; l++)
                    for(int v=0; v<NUMSEROTYPES; v++)
                        sum_foi += flu_params->sigma2d[vir][v]
                                   * flu_params->beta[index*NUMSEROTYPES + v]
                                   * stf
                                   * flu_params->eta[loc][l]
                                   * y[ STARTI + NUMSEROTYPES*l + v ];

//                printf("index %d sum_foi = %f\n", NUMSEROTYPES*NUMR*loc + NUMR*vir + stg,sum_foi);
                // now add the term to dR/dt that accounts for the force of infection removing some R-individuals
                f[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ] += ( -sum_foi ) * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + stg ];

            }
        }
    }

    //
    // ###  3.  WRITE DOWN THE DERIVATIVES FOR ALL THE INFECTED CLASSES and the J-CLASSES
    //


    for(loc=0; loc<NUMLOC; loc++)
    {
        for(vir=0; vir<NUMSEROTYPES; vir++)
        {
            // initialize these derivatives to zero
            f[ STARTI + NUMSEROTYPES*loc + vir ] = 0.0;
            f[ STARTJ + NUMSEROTYPES*loc + vir ] = 0.0;

            // sum over locations to get the force of infection of virus vir on susceptibles in location loc
            double foi_on_susc_single_virus = 0.0;
            for(int l=0; l<NUMLOC; l++) {
                foi_on_susc_single_virus +=
                        flu_params->eta[loc][l]
                        * stf
                        * flu_params->beta[index*NUMSEROTYPES + vir]
                        * y[STARTI + NUMSEROTYPES * l + vir];
//                printf("index %d foi_on_susc_single_virus += flu_params->eta[%d][%d]"
//                       " * stf"
//                       " * flu_params->beta[index*NUMSEROTYPES + %d]"
//                       " * y[%d] = %f\n",
//                       STARTI + NUMSEROTYPES*loc + vir, loc,l,vir,STARTI + NUMSEROTYPES*l + vir,foi_on_susc_single_virus);
//                printf("index %d loc %d vir %d l %d Y[%d] = %f\n",STARTI + NUMSEROTYPES*loc + vir,loc,vir,l,STARTI + NUMSEROTYPES * l + vir,y[STARTI + NUMSEROTYPES * l + vir]);
            }

//            printf("index %d foi_on_susc_single_virus = %f\n", STARTI + NUMSEROTYPES*loc + vir,foi_on_susc_single_virus);
//            printf("index %d loc %d vir %d Y[%d] = %f\n",STARTI + NUMSEROTYPES*loc + vir,loc,vir,STARTS + loc,y[STARTS + loc]);
            // add the in-flow of new infections from the susceptible class
            f[ STARTI + NUMSEROTYPES*loc + vir ] += y[ STARTS + loc ] * foi_on_susc_single_virus;
            f[ STARTJ + NUMSEROTYPES*loc + vir ] += y[ STARTS + loc ] * foi_on_susc_single_virus;

            // sum over locations and different types of recovered individuals to get the inflow of recovered
            // individuals that are becoming re-infected
            double inflow_from_recovereds = 0.0;
            for(int l=0; l<NUMLOC; l++){          // sum over locations
                for (int v = 0; v < NUMSEROTYPES; v++) { // sum over recent immunity
                    for (int s = 0; s < NUMR; s++) {       // sum over R stage
                        inflow_from_recovereds += flu_params->sigma2d[vir][v]
                                                  * stf
                                                  * flu_params->beta[index*NUMSEROTYPES + vir]
                                                  * flu_params->eta[loc][l]
                                                  * y[STARTI + NUMSEROTYPES * l + vir]
                                                  * y[NUMSEROTYPES * NUMR * loc + NUMR * v + s];
//                        printf("index = %d inflow_from_recovereds += inflow_from_recovereds_sbe = %f * y[%d] = %f * y[%d] = %f\n",STARTI + NUMSEROTYPES*loc + vir,flu_params->sigma2d[vir][v]
//                                                                                                                                                              * stf
//                                                                                                                                                              * flu_params->beta[index*NUMSEROTYPES + vir]
//                                                                                                                                                              * flu_params->eta[loc][l],
//                               STARTI + NUMSEROTYPES * l + vir,NUMSEROTYPES * NUMR * loc + NUMR * v + s,
//                                y[STARTI + NUMSEROTYPES * l + vir],y[NUMSEROTYPES * NUMR * loc + NUMR * v + s]);
                    }
                }
            }

//            printf("index %d inflow_from_recovereds = %f\n",STARTI + NUMSEROTYPES*loc + vir,inflow_from_recovereds);
            // add the in-flow of new infections from the recovered classes (all histories, all stages)
            f[ STARTI + NUMSEROTYPES*loc + vir ] += inflow_from_recovereds;
            f[ STARTJ + NUMSEROTYPES*loc + vir ] += inflow_from_recovereds;

            // add the recovery rate - NOTE only for I-classes
            f[ STARTI + NUMSEROTYPES*loc + vir ] += - flu_params->v_d_i_nu * y[ STARTI + NUMSEROTYPES*loc + vir ];

        }
    }



    //
    // ###  4.  WRITE DOWN THE DERIVATIVES FOR ALL THE SUSCEPTIBLE CLASSES
    //


    for(loc=0; loc<NUMLOC; loc++)
    {
        // compute the force of infection of all viruses at all locations on the susceptibles at the location loc
        double foi_on_susc_all_viruses = 0.0;
        for(int l=0; l<NUMLOC; l++) {
            for (int v = 0; v < NUMSEROTYPES; v++) {
                foi_on_susc_all_viruses += flu_params->eta[loc][l] * stf * flu_params->beta[index*NUMSEROTYPES + v] *
                                           y[STARTI + NUMSEROTYPES * l + v];
//                printf(" loop l-v index %d loc %d foi_on_susc_all_viruses = %f\n",STARTS + loc,loc,foi_on_susc_all_viruses);
            }
        }


//        printf("index %d loc %d foi_on_susc_all_viruses = %f\n",STARTS + loc,loc,foi_on_susc_all_viruses);
//        printf("index %d loc %d y[%d] = %f\n",STARTS + loc,loc,STARTS + loc,y[STARTS + loc]);
        // add to ODE dS/dt equation the removal of susceptibles by all types of infection
        f[ STARTS + loc ] = ( - foi_on_susc_all_viruses ) * y[ STARTS + loc ];

//        printf("index %d loc %d f[%d] = %f\n",STARTS + loc,loc,STARTS + loc,f[STARTS + loc]);
        // now loop through all the recovered classes in this location (different histories, final stage only)
        for(int vir=0; vir<NUMSEROTYPES; vir++)
        {
            // add to dS/dt the inflow of recovereds from the final R-stage
            f[ STARTS + loc ] += flu_params->trr * y[ NUMSEROTYPES*NUMR*loc + NUMR*vir + (NUMR-1) ]; // "NUMR-1" gets you the final R-stage only
//            printf("loop vir index %d loc %d f[%d] = %f\n",STARTS + loc,loc,STARTS + loc,f[STARTS + loc]);
        }
    }

//    for(int j=0;j<DIM;j++) {
//        if(j == 0 || j == DIM -1)
//        {
//            printf("[function] OUT f[%d] = %f\n",j,f[j]);
//        }
//    }

    return;
}
