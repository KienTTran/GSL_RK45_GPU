#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <random>
#include "gpu_rk45.h"

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
    return result;
}

GPU_RK45::GPU_RK45(){
    params = new GPU_Parameters();
}

GPU_RK45::~GPU_RK45(){
    params = nullptr;
}

void GPU_RK45::set_parameters(GPU_Parameters* params_) {
    params = &(*params_);
}

__device__
double seasonal_transmission_factor(GPU_Parameters* gpu_params, double t)
{
    /*


        We're gonna make this thing go for 40 years. 30 years of burn in and 10 years of real modeling.
        We're creating a "10-year model cycle" and need the code below to find a time point's "place" in the "cycle"
        modulus (denoted with % in C++) only works with integers, so need the acrobatics below

     */

    // This is some code that's needed to create the 10-year "cycles" in transmission.

    if(gpu_params->phis_d_length == 0){
        return 1.0;
    }

    int x = (int)t; // This is now to turn a double into an integer
    double remainder = t - (double)x;
    int xx = x % 3650; // int xx = x % NUMDAYSOUTPUT;
    double yy = (double)xx + remainder;
    // put yy into the sine function, let it return the beta value
    t = yy;
    double sine_function_value = 0.0;

    for(int i=0; i < gpu_params->phis_d_length; i++)
    {
        if( fabs( t - gpu_params->phis_d[i] ) < (gpu_params->v_d_i_epidur_d2))
        {
            // sine_function_value = sin( 2.0 * 3.141592653589793238 * (phis[i]-t+91.25) / 365.0);
            sine_function_value = sin( gpu_params->pi_x2 * (gpu_params->phis_d[i] - t + (gpu_params->v_d_i_epidur_d2)) / (gpu_params->v_d_i_epidur_x2));
//            printf("      in loop %1.3f %d  %1.3f %1.3f\n", t, i, gpu_params->phis_d[i], sine_function_value);
        }
    }
//    printf("    %f sine_function_value %1.3f\n",t,sine_function_value);
//    printf("    %f return %1.3f\n",t,1.0 + v[i_amp] * sine_function_value);
    return 1.0 + gpu_params->v_d_i_amp * sine_function_value;
}

__device__
double pop_sum( double yy[] )
{
    double sum=0.0;
    for(int i=0; i<DIM; i++) sum += yy[i];

    for(int i=STARTJ; i<STARTJ+NUMLOC*NUMSEROTYPES; i++) sum -= yy[i];
    return sum;
}

__device__
void rk45_gpu_adjust_h(double y[], double y_err[], double dydt_out[],
                             double &h, double h_0, int &adjustment_out, int final_step,
                             const int index){
    /* adaptive adjustment */
    /* Available control object constructors.
     *
     * The standard control object is a four parameter heuristic
     * defined as follows:
     *    D0 = eps_abs + eps_rel * (a_y |y| + a_dydt h |y'|)
     *    D1 = |yerr|
     *    q  = consistency order of method (q=4 for 4(5) embedded RK)
     *    S  = safety factor (0.9 say)
     *
     *                      /  (D0/D1)^(1/(q+1))  D0 >= D1
     *    h_NEW = S h_OLD * |
     *                      \  (D0/D1)^(1/q)      D0 < D1
     *
     * This encompasses all the standard error scaling methods.
     *
     * The y method is the standard method with a_y=1, a_dydt=0.
     * The yp method is the standard method with a_y=0, a_dydt=1.
     */
    static double eps_abs = 1e-6;
    static double eps_rel = 0.0;
    static double a_y = 1.0;
    static double a_dydt = 0.0;
    static unsigned int ord = 5;
    const double S = 0.9;
    double h_old;
    if(final_step){
        h_old = h_0;
    }
    else{
        h_old = h;
    }

//    printf("    [adjust h] index = %d begin\n",index);
//    for (int i = 0; i < DIM; i ++)
//    {
//        printf("      y[%d] = %.10f\n",i,y[i]);
//    }
//    for (int i = 0; i < DIM; i ++)
//    {
//        printf("      y_err[%d] = %.10f\n",i,y_err[i]);
//    }
//    for (int i = 0; i < DIM; i ++)
//    {
//        printf("      dydt_out[%d] = %.10f\n",i,dydt_out[i]);
//    }

    double r_max = 2.2250738585072014e-308;
    for (int i = 0; i < DIM; i ++)
    {
        const double D0 = eps_rel * (a_y * fabs(y[i]) + a_dydt * fabs((h_old) * dydt_out[i])) + eps_abs;
        const double r  = fabs(y_err[i]) / fabs(D0);
//        printf("      compare r = %.10f r_max = %.10f\n",r,r_max);
        r_max = max(r, r_max);
    }

//    printf("      r_max = %.10f\n",r_max);

    if (r_max > 1.1) {
        /* decrease step, no more than factor of 5, but a fraction S more
           than scaling suggests (for better accuracy) */
        double r = S / pow(r_max, 1.0 / ord);

        if (r < 0.2)
            r = 0.2;
        h = r * (h_old);

//        printf("      index = %d decrease by %.10f, h_old is %.10f new h is %.10f\n",index, r, h_old, h);
        adjustment_out = -1;
    } else if (r_max < 0.5) {
        /* increase step, no more than factor of 5 */
        double r = S / pow(r_max, 1.0 / (ord + 1.0));

        if (r > 5.0)
            r = 5.0;

        if (r < 1.0)  /* don't allow any decrease caused by S<1 */
            r = 1.0;

        h = r * (h_old);

//        printf("      index = %d increase by %.10f, h_old is %.10f new h is %.10f\n",index, r, h_old, h);
        adjustment_out = 1;
    } else {
        /* no change */
//        printf("      index = %d no change\n",index);
        adjustment_out = 0;
    }
//    printf("    [adjust h] index = %d end\n",index);
    return;
}

__device__
void rk45_gpu_step_apply(double t, double h, double y[], double y_err[], double dydt_out[],
                         const int index, GPU_Parameters* params)
{
    static const double ah[] = { 1.0/4.0, 3.0/8.0, 12.0/13.0, 1.0, 1.0/2.0 };
    static const double b3[] = { 3.0/32.0, 9.0/32.0 };
    static const double b4[] = { 1932.0/2197.0, -7200.0/2197.0, 7296.0/2197.0};
    static const double b5[] = { 8341.0/4104.0, -32832.0/4104.0, 29440.0/4104.0, -845.0/4104.0};
    static const double b6[] = { -6080.0/20520.0, 41040.0/20520.0, -28352.0/20520.0, 9295.0/20520.0, -5643.0/20520.0};

    static const double c1 = 902880.0/7618050.0;
    static const double c3 = 3953664.0/7618050.0;
    static const double c4 = 3855735.0/7618050.0;
    static const double c5 = -1371249.0/7618050.0;
    static const double c6 = 277020.0/7618050.0;

    static const double ec[] = { 0.0,
                                 1.0 / 360.0,
                                 0.0,
                                 -128.0 / 4275.0,
                                 -2197.0 / 75240.0,
                                 1.0 / 50.0,
                                 2.0 / 55.0
    };

//    printf("    [step apply] index = %d start\n",index);
//    printf("      t = %.10f h = %.10f\n",t,h);

//    double* y_tmp = (double*)malloc(dim);
//    double* k1 = (double*)malloc(dim);
//    double* k2 = (double*)malloc(dim);
//    double* k3 = (double*)malloc(dim);
//    double* k4 = (double*)malloc(dim);
//    double* k5 = (double*)malloc(dim);
//    double* k6 = (double*)malloc(dim);
    double y_tmp[DIM];
    double k1[DIM];
    double k2[DIM];
    double k3[DIM];
    double k4[DIM];
    double k5[DIM];
    double k6[DIM];

    for(int i = 0; i < DIM; i++){
        y_tmp[i] = 0.0;
        y_err[i] = 0.0;
        dydt_out[i] = 0.0;
        k1[i] = 0.0;
        k2[i] = 0.0;
        k3[i] = 0.0;
        k4[i] = 0.0;
        k5[i] = 0.0;
        k6[i] = 0.0;
    }

//    for (int i = 0; i < DIM; i ++)
//    {
//        printf("      y[%d] = %.10f\n",i,y[i]);
//        printf("      y_tmp[%d] = %.10f\n",i,y_tmp[i]);
//        printf("      y_err[%d] = %.10f\n",i,y_err[i]);
//        printf("      dydt_out[%d] = %.10f\n",i,dydt_out[i]);
//    }

    /* k1 */
    gpu_func_test(t,y,k1, index, params);
//    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i ++)
    {
//        printf("      k1[%d] = %.10f\n",i,k1[i]);
        y_tmp[i] = y[i] +  ah[0] * h * k1[i];
    }
    /* k2 */
    gpu_func_test(t + ah[0] * h, y_tmp,k2, index, params);
//    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i ++)
    {
//        printf("      k2[%d] = %.10f\n",i,k2[i]);
        y_tmp[i] = y[i] + h * (b3[0] * k1[i] + b3[1] * k2[i]);
    }
    /* k3 */
    gpu_func_test(t + ah[1] * h, y_tmp,k3, index, params);
//    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i ++)
    {
//        printf("      k3[%d] = %.10f\n",i,k3[i]);
        y_tmp[i] = y[i] + h * (b4[0] * k1[i] + b4[1] * k2[i] + b4[2] * k3[i]);
    }
    /* k4 */
    gpu_func_test(t + ah[2] * h, y_tmp,k4, index, params);
//    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i ++)
    {
//        printf("      k4[%d] = %.10f\n",i,k4[i]);
        y_tmp[i] = y[i] + h * (b5[0] * k1[i] + b5[1] * k2[i] + b5[2] * k3[i] + b5[3] * k4[i]);
    }
    /* k5 */
    gpu_func_test(t + ah[3] * h, y_tmp,k5, index, params);
//    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i ++)
    {
//        printf("      k5[%d] = %.10f\n",i,k5[i]);
        y_tmp[i] = y[i] + h * (b6[0] * k1[i] + b6[1] * k2[i] + b6[2] * k3[i] + b6[3] * k4[i] + b6[4] * k5[i]);
    }
    /* k6 */
    gpu_func_test(t + ah[4] * h, y_tmp,k6, index, params);
//    hipDeviceSynchronize();
    /* final sum */
    for (int i = 0; i < DIM; i ++)
    {
//        printf("      k6[%d] = %.10f\n",i,k6[i]);
        const double d_i = c1 * k1[i] + c3 * k3[i] + c4 * k4[i] + c5 * k5[i] + c6 * k6[i];
        y[i] += h * d_i;
    }
    /* Derivatives at output */
    gpu_func_test(t + h, y, dydt_out, index, params);
//    hipDeviceSynchronize();
    /* difference between 4th and 5th order */
    for (int i = 0; i < DIM; i ++)
    {
        y_err[i] = h * (ec[1] * k1[i] + ec[3] * k3[i] + ec[4] * k4[i] + ec[5] * k5[i] + ec[6] * k6[i]);
    }
    //debug printout
//    for (int i = 0; i < DIM; i++) {
//        printf("      index = %d y[%d] = %.10f\n",index,i,y[i]);
//    }
//    for (int i = 0; i < DIM; i++) {
//        printf("      index = %d y_err[%d] = %.10f\n",index,i,y_err[i]);
//    }
//    for (int i = 0; i < DIM; i++) {
//        printf("      index = %d dydt_out[%d] = %.10f\n",index,i,dydt_out[i]);
//    }
//    printf("    [step apply] index = %d end\n",index);
    return;
}

__device__
void rk45_gpu_evolve_apply(double t, double t_target, double t_delta, double h, double* y[], double* y_output[], int index, GPU_Parameters* params){
    double device_y[DIM];
    double device_y_0[DIM];
    double device_y_err[DIM];
    double device_dydt_out[DIM];
    while(t < t_target)
    {
      double device_t;
      double device_t1;
      double device_h;
      double device_h_0;
      double device_dt;
      int device_adjustment_out = 999;
      device_t = t;
      device_t1 = device_t + t_delta;
      device_h = h;

      int day = t;
//      printf("day %d\t", day);
//      for (int i = 0; i < params->dimension; i ++) {
//        printf("y[%d][%d] = %.1f\t", index, i, y[index][i]);
//        if(i == (params->dimension - 1)){
//          printf("\n");
//        }
//      }
      for (int i = 0; i < params->display_dimension; i ++) {
        const int y_output_index = day * params->display_dimension + i;
        if(y_output_index % params->display_dimension == 0){
          //First column
          y_output[index][y_output_index] = day*1.0;
//          printf("First day = %d index = %d i = %d y_output_index = %d y_output[%d][%d] = %.5f\n",
//                 day, index, i, y_output_index, index, y_output_index, y_output[index][y_output_index]);
        }
        else if(y_output_index % params->display_dimension == 1){
          //Second column
          y_output[index][y_output_index] = seasonal_transmission_factor(params,t);
//          printf("Second day = %d index = %d i = %d y_output_index = %d y_output[%d][%d] = %.5f\n",
//                 day, index, i, y_output_index, index, y_output_index, y_output[index][y_output_index]);
        }
        else if(y_output_index % params->display_dimension == 2){
          //Third column
          y_output[index][y_output_index] = pop_sum(y[index]);
//          printf("Third day = %d index = %d i = %d y_output_index = %d y_output[%d][%d] = %.5f\n",
//                 day, index, i, y_output_index, index, y_output_index, y_output[index][y_output_index]);
        }
        else if(day > 0 && (y_output_index % (params->display_dimension)) - (params->display_dimension - 1) == -2){
          //INC1 - last column -2
          printf("y_output_index = %d - inc1\n",y_output_index);
          y_output[index][y_output_index] = y_output[index][y_output_index - 4];
        }
        else if(day > 0 && (y_output_index % (params->display_dimension)) - (params->display_dimension - 1) == -1){
          //INC2 - last column -1
          printf("y_output_index = %d - inc2\n",y_output_index);
          y_output[index][y_output_index] = y_output[index][y_output_index - 4];
        }
        else if(day > 0 && (y_output_index % (params->display_dimension)) - (params->display_dimension - 1) == 0){
          //INC3 - last column
          printf("y_output_index = %d - inc3\n",y_output_index);
          y_output[index][y_output_index] = y_output[index][y_output_index - 4];
        }
        else{
          //Forth column onward
          const int y_index = (y_output_index - 3) % params->display_dimension;
          y_output[index][y_output_index] = y[index][y_index];
//          printf("day = %d index = %d i = %d y_output_index = %d y[%d][%d] = y[%d][%d] = %.5f\n",
//                 day, index, i, y_output_index, index, y_output_index, index, y_index, y[index][y_index]);
        }
      }

      while(device_t < device_t1)
      {
        int device_final_step = 0;
        const double device_t_0 = device_t;
        device_h_0 = device_h;
        device_dt = device_t1 - device_t_0;
        //                if(index == 0){
        //                    printf("\n  [evolve apply] index = %d start\n",index);
        //                    printf("    t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f dt = %.10f\n",device_t,device_t_0,device_h,device_h_0,device_dt);
        //                }

        for (int i = 0; i < params->dimension; i ++){
          device_y[i] = y[index][i];
          device_y_0[i] = device_y[i];
        }

        device_final_step = 0;

        while(true){
          if ((device_dt >= 0.0 && device_h_0 > device_dt) || (device_dt < 0.0 && device_h_0 < device_dt)) {
            device_h_0 = device_dt;
            device_final_step = 1;
          } else {
            device_final_step = 0;
          }

          rk45_gpu_step_apply(device_t_0,device_h_0,device_y,device_y_err,device_dydt_out,
                              index, params);

          if (device_final_step) {
            device_t = device_t1;
          } else {
            device_t = device_t_0 + device_h_0;
          }

          double h_old = device_h_0;

          //                    printf("    before adjust t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f h_old = %.10f\n",device_t,device_t_0,device_h,device_h_0,h_old);

          rk45_gpu_adjust_h(device_y, device_y_err, device_dydt_out,
                            device_h, device_h_0, device_adjustment_out, device_final_step,index);

          //Extra step to get data from h
          device_h_0 = device_h;

          //                    printf("    after adjust t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f h_old = %.10f\n",device_t,device_t_0,device_h,device_h_0,h_old);

          if (device_adjustment_out == -1)
          {
            double t_curr = (device_t);
            double t_next = (device_t) + device_h_0;

            if (fabs(device_h_0) < fabs(h_old) && t_next != t_curr) {
              /* Step was decreased. Undo step, and try again with new h0. */
              //                            printf("  [evolve apply] index = %d step decreased, y = y0\n",index);
              for (int i = 0; i < DIM; i++) {
                device_y[i] = device_y_0[i];
              }
            } else {
              //                            printf("  [evolve apply] index = %d step decreased h_0 = h_old\n",index);
              device_h_0 = h_old; /* keep current step size */
              break;
            }
          }
          else{
            //                        printf("  [evolve apply] index = %d step increased or no change\n",index);
            break;
          }
        }
        device_h = device_h_0;  /* suggest step size for next time-step */
        h = device_h;
        for (int i = 0; i < DIM; i++){
          y[index][i] = device_y[i];
        }
        //                printf("    index = %d t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f\n",index,device_t,device_t_0,device_h,device_h_0);
        //                for (int i = 0; i < DIM; i++){
        //                    printf("    index = %d y[%d][%d] = %.10f\n",index,index,i,device_y[i]);
        //                }
        //                printf("  [evolve apply] index = %d end\n",index);
        //                if(device_final_step){
        //                    printf("[output] index = %d t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f\n",index,device_t,device_t_0,device_h,device_h_0);
        //                    for (int i = 0; i < DIM; i++){
        //                        printf("[output] index = %d y[%d] = %.10f\n",index,i,device_y[i]);
        //                    }
        //                }
        //                device_t = device_t_0 + device_h_0;
      }
      //            if(index == 0) {
      //                printf("[evolve apply] Index = %d t = %f h = %f end one day\n", index, t, h);
      //            }
      t += t_delta;
    }
    //        if(index == 0){
    //            for (int i = 0; i < DIM; i++){
    //                printf("[output] index = %d y[%d] = %1.5f\n",index,i,device_y[i]);
    //            }
    //        }
}

__device__
void solve_ode(double* y_d[], double* y_output_d[], int index, GPU_Parameters* params){
    rk45_gpu_evolve_apply(params->t0, params->t_target, params->step, params->h, y_d, y_output_d, index, params);
    return;
}

__device__
void mcmc(double* y_output_d[], int index, GPU_Parameters* params){
    for(int i = 0; i < NUMDAYSOUTPUT * params->display_dimension; i++){
      printf("%.1f\t", y_output_d[index][i]);
      if(i > 0 && (i + 1) % params->display_dimension == 0){
        printf("\n");
      }
    }
    return;
}

__global__
void solve_ode_mcmc(double* y_d[], double* y_output_d[], GPU_Parameters* params){
    int index_gpu = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;


    for(int index = index_gpu; index < NUMODE; index += stride)
    {
      for(int iter = 0; iter < 1; iter++){
        solve_ode(y_d, y_output_d, index, params);


//        mcmc(y_output_d, index, params);
      }
    }
    return;
}

void GPU_RK45::run(){
    int num_SMs;
    checkCuda(hipDeviceGetAttribute(&num_SMs, hipDeviceAttributeMultiprocessorCount, 0));
    //    int numBlocks = 32*num_SMs; //multiple of 32
    params->block_size = 256; //max is 1024
    params->num_blocks = (NUMODE + params->block_size - 1) / params->block_size;
    printf("[GSL GPU] block_size = %d num_blocks = %d\n",params->block_size,params->num_blocks);

    auto start = std::chrono::high_resolution_clock::now();
    double **y_d = 0;
    //temp pointers
    double **tmp_ptr = (double**)malloc (NUMODE * sizeof (double));
    for (int i = 0; i < NUMODE; i++) {
        checkCuda(hipMalloc ((void **)&tmp_ptr[i], params->dimension * sizeof (double)));
        checkCuda(hipMemcpy(tmp_ptr[i], params->y[i], params->dimension * sizeof(double), hipMemcpyHostToDevice));
    }
    //y_d
    checkCuda(hipMalloc ((void **)&y_d, NUMODE * sizeof (double)));
    checkCuda(hipMemcpy (y_d, tmp_ptr, NUMODE * sizeof (double), hipMemcpyHostToDevice));

    double **y_output_d = 0;
    //temp pointers
    for (int i = 0; i < NUMODE; i++) {
      checkCuda(hipMalloc ((void **)&tmp_ptr[i], NUMDAYSOUTPUT * params->display_dimension * sizeof (double)));
      checkCuda(hipMemcpy(tmp_ptr[i], params->y_output[i], NUMDAYSOUTPUT * params->display_dimension * sizeof(double), hipMemcpyHostToDevice));
    }
    //y_output_d
    checkCuda(hipMalloc ((void **)&y_output_d, NUMODE * sizeof (double)));
    checkCuda(hipMemcpy (y_output_d, tmp_ptr, NUMODE * sizeof (double), hipMemcpyHostToDevice));

    //params_d
    GPU_Parameters* params_d;
    checkCuda(hipMalloc((void **) &params_d, sizeof(GPU_Parameters)));
    checkCuda(hipMemcpy(params_d, params, sizeof(GPU_Parameters), hipMemcpyHostToDevice));

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for allocate mem CPU to GPU: %ld micro seconds which is %.10f seconds\n",duration.count(),(duration.count()/1e6));

    start = std::chrono::high_resolution_clock::now();
//    hipProfilerStart();
    solve_ode_mcmc<<<params->num_blocks, params->block_size>>>(y_d, y_output_d, params_d);

//    hipProfilerStop();
    checkCuda(hipDeviceSynchronize());
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for calculating %d ODE with %d parameters on GPU: %ld micro seconds which is %.10f seconds\n",NUMODE,DIM,duration.count(),(duration.count()/1e6));

    start = std::chrono::high_resolution_clock::now();
    tmp_ptr = (double**)malloc (NUMODE * sizeof (double));
    double** y_output_h = (double**)malloc (NUMODE * sizeof (double));
    for (int i = 0; i < NUMODE; i++) {
      y_output_h[i] = (double *)malloc (NUMDAYSOUTPUT * params->display_dimension * sizeof (double));
    }
    checkCuda(hipMemcpy (tmp_ptr, y_output_d, NUMODE * sizeof (double), hipMemcpyDeviceToHost));
    for (int i = 0; i < NUMODE; i++) {
        checkCuda(hipMemcpy (y_output_h[i], tmp_ptr[i], NUMDAYSOUTPUT * params->display_dimension * sizeof (double), hipMemcpyDeviceToHost));
    }
    checkCuda(hipDeviceSynchronize());
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for data transfer GPU to CPU: %ld micro seconds which is %.10f seconds\n",duration.count(),(duration.count()/1e6));

    start = std::chrono::high_resolution_clock::now();
    std::random_device rd; // obtain a random number from hardware
    std::mt19937 gen(rd()); // seed the generator
    std::uniform_int_distribution<> distr(0, NUMODE); // define the range

    for(int i = 0; i < params->display_number; i++){
        int random_index = 0;
        if(NUMODE == 1){
            random_index = 0;
        }
        else{
            random_index = distr(gen);
        }
        printf("Display y_output_h[%d]\n",random_index);
        for(int index = 0; index < NUMDAYSOUTPUT * params->display_dimension; index++){
          printf("%.1f\t", y_output_h[random_index][index]);
          if(index > 0 && (index + 1) % params->display_dimension == 0){
            printf("\n");
          }
        }
    }
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for display random results on CPU: %ld micro seconds which is %.10f seconds\n",duration.count(),(duration.count()/1e6));
    printf("\n");
    // Free memory
    checkCuda(hipFree(y_d));
    return;
}