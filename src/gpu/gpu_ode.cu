#include "hip/hip_runtime.h"
//
// Created by kient on 5/2/2022.
//

#include "gpu_ode.cuh"

__device__
double pop_sum(double yy[]) {
    double sum = 0.0;
    for (int i = 0; i < DIM; i++) sum += yy[i];

    for (int i = STARTJ; i < STARTJ + NUMLOC * NUMSEROTYPES; i++) sum -= yy[i];
    return sum;
}

__device__
void rk45_gpu_adjust_h(double y[], double y_err[], double dydt_out[],
                       double &h, double h_0, int &adjustment_out, int final_step,
                       const int index) {
    /* adaptive adjustment */
    /* Available control object constructors.
       *
       * The standard control object is a four parameter heuristic
       * defined as follows:
       *    D0 = eps_abs + eps_rel * (a_y |y| + a_dydt h |y'|)
       *    D1 = |yerr|
       *    q  = consistency order of method (q=4 for 4(5) embedded RK)
       *    S  = safety factor (0.9 say)
       *
       *                      /  (D0/D1)^(1/(q+1))  D0 >= D1
       *    h_NEW = S h_OLD * |
       *                      \  (D0/D1)^(1/q)      D0 < D1
       *
       * This encompasses all the standard error scaling methods.
       *
       * The y method is the standard method with a_y=1, a_dydt=0.
       * The yp method is the standard method with a_y=0, a_dydt=1.
     */
    static double eps_abs = 1e-6;
    static double eps_rel = 0.0;
    static double a_y = 1.0;
    static double a_dydt = 0.0;
    //static unsigned int ord = 5;
    static double one_over_ord = 0.2;// 1 / ord = 1 / 5
    static double one_over_ord_plus_one = 0.166666667;// 1 / ord = 1/ 6
    const double S = 0.9;
    double h_old;
    if (final_step) {
        h_old = h_0;
    } else {
        h_old = h;
    }

    //    printf("    [adjust h] index = %d begin\n",index);
    //    for (int i = 0; i < DIM; i ++)
    //    {
    //        printf("      y[%d] = %.10f\n",i,y[i]);
    //    }
    //    for (int i = 0; i < DIM; i ++)
    //    {
    //        printf("      y_err[%d] = %.10f\n",i,y_err[i]);
    //    }
    //    for (int i = 0; i < DIM; i ++)
    //    {
    //        printf("      dydt_out[%d] = %.10f\n",i,dydt_out[i]);
    //    }

    float r_max = 1.175494e-38;
//    double r_max = 2.2250738585072014e-308;
    for (int i = 0; i < DIM; i++) {
        const double D0 = eps_rel * (a_y * fabs(y[i]) + a_dydt * fabs((h_old) * dydt_out[i])) + eps_abs;
        const float r = __fdividef(fabs(y_err[i]), fabs(D0));
//        const double r = fabs(y_err[i]) / fabs(D0);
        //        printf("      compare r = %.10f r_max = %.10f\n",r,r_max);
        r_max = max(r, r_max);
    }

    //    printf("      r_max = %.10f\n",r_max);

    if (r_max > 1.1) {
        /* decrease step, no more than factor of 5, but a fraction S more
           than scaling suggests (for better accuracy) */
        float r = __fdividef(S,pow(r_max, one_over_ord));
//        double r = S / pow(r_max, one_over_ord);

        if (r < 0.2)
            r = 0.2;
        h = r * (h_old);

        //        printf("      index = %d decrease by %.10f, h_old is %.10f new h is %.10f\n",index, r, h_old, h);
        adjustment_out = -1;
    } else if (r_max < 0.5) {
        /* increase step, no more than factor of 5 */
        float r = __fdividef(S,pow(r_max, one_over_ord_plus_one));
//        double r = S / pow(r_max, one_over_ord_plus_one);

        if (r > 5.0)
            r = 5.0;

        if (r < 1.0)  /* don't allow any decrease caused by S<1 */
            r = 1.0;

        h = r * (h_old);

        //        printf("      index = %d increase by %.10f, h_old is %.10f new h is %.10f\n",index, r, h_old, h);
        adjustment_out = 1;
    } else {
        /* no change */
        //        printf("      index = %d no change\n",index);
        adjustment_out = 0;
    }
    //    printf("    [adjust h] index = %d end\n",index);
    return;
}

__device__
void rk45_gpu_step_apply(double t, double h, double y[], double y_err[], double dydt_out[], double stf,
                         const int index, FluParameters *flu_params) {
//    static const double ah[] = {1.0 / 4.0, 3.0 / 8.0, 12.0 / 13.0, 1.0, 1.0 / 2.0};
//    static const double b3[] = {3.0 / 32.0, 9.0 / 32.0};
//    static const double b4[] = {1932.0 / 2197.0, -7200.0 / 2197.0, 7296.0 / 2197.0};
//    static const double b5[] = {8341.0 / 4104.0, -32832.0 / 4104.0, 29440.0 / 4104.0, -845.0 / 4104.0};
//    static const double b6[] = {-6080.0 / 20520.0, 41040.0 / 20520.0, -28352.0 / 20520.0, 9295.0 / 20520.0,
//                                -5643.0 / 20520.0};
//
//    static const double c1 = 902880.0 / 7618050.0;
//    static const double c3 = 3953664.0 / 7618050.0;
//    static const double c4 = 3855735.0 / 7618050.0;
//    static const double c5 = -1371249.0 / 7618050.0;
//    static const double c6 = 277020.0 / 7618050.0;
//
//    static const double ec[] = {0.0,
//                                1.0 / 360.0,
//                                0.0,
//                                -128.0 / 4275.0,
//                                -2197.0 / 75240.0,
//                                1.0 / 50.0,
//                                2.0 / 55.0
//    };

    static const double ah[] = {0.25, 0.375, 0.923076923, 1.0, 0.5};
    static const double b3[] = {0.09375, 0.28125};
    static const double b4[] = {0.879380974, -3.277196177, 3.320892126};
    static const double b5[] = {2.032407407, -8.0, 7.173489279, -0.205896686};
    static const double b6[] = {-0.296296296, 2.0, -1.381676413, 0.45297271,
                                -0.275};

    static const double c1 = 0.118518519;
    static const double c3 = 0.518986355;
    static const double c4 = 0.50613149;
    static const double c5 = -0.18;
    static const double c6 = 0.036363636;

    static const double ec[] = {0.0,
                                0.002777778,
                                0.0,
                                -0.02994152,
                                -0.029199894,
                                0.02,
                                0.036363636
    };

    //    printf("    [step apply] index = %d start\n",index);
    //    printf("      t = %.10f h = %.10f\n",t,h);

    //    double* y_tmp = (double*)malloc(dim);
    //    double* k1 = (double*)malloc(dim);
    //    double* k2 = (double*)malloc(dim);
    //    double* k3 = (double*)malloc(dim);
    //    double* k4 = (double*)malloc(dim);
    //    double* k5 = (double*)malloc(dim);
    //    double* k6 = (double*)malloc(dim);
    double y_tmp[DIM];
    double k1[DIM];
    double k2[DIM];
    double k3[DIM];
    double k4[DIM];
    double k5[DIM];
    double k6[DIM];

    for (int i = 0; i < DIM; i++) {
        y_tmp[i] = 0.0;
        y_err[i] = 0.0;
        dydt_out[i] = 0.0;
        k1[i] = 0.0;
        k2[i] = 0.0;
        k3[i] = 0.0;
        k4[i] = 0.0;
        k5[i] = 0.0;
        k6[i] = 0.0;
    }

    //    for (int i = 0; i < DIM; i ++)
    //    {
    //        printf("      y[%d] = %.10f\n",i,y[i]);
    //        printf("      y_tmp[%d] = %.10f\n",i,y_tmp[i]);
    //        printf("      y_err[%d] = %.10f\n",i,y_err[i]);
    //        printf("      dydt_out[%d] = %.10f\n",i,dydt_out[i]);
    //    }

    /* k1 */
    gpu_func_flu(t, y, k1, stf, index, flu_params);
    //    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i++) {
        //        printf("      k1[%d] = %.10f\n",i,k1[i]);
        y_tmp[i] = y[i] + ah[0] * h * k1[i];
    }
    /* k2 */
    gpu_func_flu(t + ah[0] * h, y_tmp, k2, stf, index, flu_params);
    //    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i++) {
        //        printf("      k2[%d] = %.10f\n",i,k2[i]);
        y_tmp[i] = y[i] + h * (b3[0] * k1[i] + b3[1] * k2[i]);
    }
    /* k3 */
    gpu_func_flu(t + ah[1] * h, y_tmp, k3, stf, index, flu_params);
    //    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i++) {
        //        printf("      k3[%d] = %.10f\n",i,k3[i]);
        y_tmp[i] = y[i] + h * (b4[0] * k1[i] + b4[1] * k2[i] + b4[2] * k3[i]);
    }
    /* k4 */
    gpu_func_flu(t + ah[2] * h, y_tmp, k4, stf, index, flu_params);
    //    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i++) {
        //        printf("      k4[%d] = %.10f\n",i,k4[i]);
        y_tmp[i] = y[i] + h * (b5[0] * k1[i] + b5[1] * k2[i] + b5[2] * k3[i] + b5[3] * k4[i]);
    }
    /* k5 */
    gpu_func_flu(t + ah[3] * h, y_tmp, k5, stf, index, flu_params);
    //    hipDeviceSynchronize();
    for (int i = 0; i < DIM; i++) {
        //        printf("      k5[%d] = %.10f\n",i,k5[i]);
        y_tmp[i] = y[i] + h * (b6[0] * k1[i] + b6[1] * k2[i] + b6[2] * k3[i] + b6[3] * k4[i] + b6[4] * k5[i]);
    }
    /* k6 */
    gpu_func_flu(t + ah[4] * h, y_tmp, k6, stf, index, flu_params);
    //    hipDeviceSynchronize();
    /* final sum */
    for (int i = 0; i < DIM; i++) {
        //        printf("      k6[%d] = %.10f\n",i,k6[i]);
        const double d_i = c1 * k1[i] + c3 * k3[i] + c4 * k4[i] + c5 * k5[i] + c6 * k6[i];
        y[i] += h * d_i;
    }
    /* Derivatives at output */
    gpu_func_flu(t + h, y, dydt_out, stf, index, flu_params);
    //    hipDeviceSynchronize();
    /* difference between 4th and 5th order */
    for (int i = 0; i < DIM; i++) {
        y_err[i] = h * (ec[1] * k1[i] + ec[3] * k3[i] + ec[4] * k4[i] + ec[5] * k5[i] + ec[6] * k6[i]);
    }
    //debug printout
    //    for (int i = 0; i < DIM; i++) {
    //        printf("      index = %d y[%d] = %.10f\n",index,i,y[i]);
    //    }
    //    for (int i = 0; i < DIM; i++) {
    //        printf("      index = %d y_err[%d] = %.10f\n",index,i,y_err[i]);
    //    }
    //    for (int i = 0; i < DIM; i++) {
    //        printf("      index = %d dydt_out[%d] = %.10f\n",index,i,dydt_out[i]);
    //    }
    //    printf("    [step apply] index = %d end\n",index);
    return;
}

__device__
void rk45_gpu_evolve_apply(double t, double t_target, double t_delta, double h, double *y[], double *y_output[],
                           double *y_agg_input[],  double *y_agg_output[],  double stf[], int index,
                           GPUParameters *gpu_params, FluParameters* flu_params) {
    double device_y[DIM];
    double device_y_0[DIM];
    double device_y_err[DIM];
    double device_dydt_out[DIM];
    double device_y_yesterday[DIM];
    int week_count = 0;
    double agg_inc_sum[DATADIM_COLS];
    double agg_inc_max[DATADIM_COLS];
    for (int i = 0; i < DATADIM_COLS; i++) {
        agg_inc_sum[i] = 0.0;
        agg_inc_max[i] = 0.0;
    }
    for (int i = 0; i < gpu_params->ode_dimension; i++) {
        device_y[i] = y[index][i];
    }

//    if((NUMODE == 1  || (index > 0 && index % (NUMODE / 2) == 0)) && t == 0){
//        printf("rk45_gpu_evolve_apply flu_params: \n");
//        printf("  beta1 = %1.9f \n", flu_params->beta[index*NUMSEROTYPES + 0]);
//        printf("  beta2 = %1.9f \n", flu_params->beta[index*NUMSEROTYPES + 1]);
//        printf("  beta3 = %1.9f \n", flu_params->beta[index*NUMSEROTYPES + 2]);
//        printf("  v_d_i_amp = %1.5f \n", flu_params->v_d_i_amp);
//        printf("  sigma_H1B = %1.5f \n", flu_params->sigma2d[0][1]);
//        printf("  sigma_BH3 = %1.5f \n", flu_params->sigma2d[1][2]);
//        printf("  sigma_H1H3 = %1.5f \n", flu_params->sigma2d[0][2]);
//        printf("  eta = %1.5f \n", flu_params->eta[0][0]);
//        printf("  eta = %1.5f \n", flu_params->eta[1][1]);
//        printf("  eta = %1.5f \n", flu_params->eta[2][2]);
//        printf("  trr = %1.5f \n", flu_params->trr);
//        printf("  v_d_i_nu = %1.5f \n", flu_params->v_d_i_nu);
//        printf("phis_length = %d\n",flu_params->SAMPLE_PHI_LENGTH);
//        for(int i=0; i<SAMPLE_PHI_LENGTH; i++){
//            printf("  phi = %5.1f \n", flu_params->phi[index*SAMPLE_PHI_LENGTH + i]);
//        }
//    }

    while (t < t_target) {
        double device_t;
        double device_t1;
        double device_h;
        double device_h_0;
        double device_dt;
        int device_adjustment_out = 999;
        device_t = t;
        device_t1 = device_t + t_delta;
        device_h = h;

        int day = t;
        double stf_today = stf[day];

//      printf("day %d\t", day);
//      for (int i = 0; i < gpu_params->ode_dimension; i ++) {
//        printf("y[%d][%d] = %.1f\t", index, i, device_y[i]);
//        if(i == (gpu_params->ode_dimension - 1)){
//          printf("\n");
//        }
//      }
        for (int i = 0; i < gpu_params->ode_dimension; i++) {
            device_y_yesterday[i] = device_y[i];
        }
        while (device_t < device_t1) {
            int device_final_step = 0;
            const double device_t_0 = device_t;
            device_h_0 = device_h;
            device_dt = device_t1 - device_t_0;
//            if(index == 0){
//                printf("\n  [evolve apply] index = %d start\n",index);
//                printf("    t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f dt = %.10f\n",device_t,device_t_0,device_h,device_h_0,device_dt);
//            }

            for (int i = 0; i < gpu_params->ode_dimension; i++) {
                device_y_0[i] = device_y[i];
            }

            device_final_step = 0;

            while (true) {
                if ((device_dt >= 0.0 && device_h_0 > device_dt) || (device_dt < 0.0 && device_h_0 < device_dt)) {
                    device_h_0 = device_dt;
                    device_final_step = 1;
                } else {
                    device_final_step = 0;
                }

                rk45_gpu_step_apply(device_t_0, device_h_0, device_y, device_y_err, device_dydt_out, stf_today,
                                    index, flu_params);

                if (device_final_step) {
                    device_t = device_t1;
                } else {
                    device_t = device_t_0 + device_h_0;
                }

                double h_old = device_h_0;

//              printf("    before adjust t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f h_old = %.10f\n",device_t,device_t_0,device_h,device_h_0,h_old);

                rk45_gpu_adjust_h(device_y, device_y_err, device_dydt_out,
                                  device_h, device_h_0, device_adjustment_out, device_final_step, index);

                //Extra step to get data from h
                device_h_0 = device_h;

//              printf("    after adjust t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f h_old = %.10f\n",device_t,device_t_0,device_h,device_h_0,h_old);

                if (device_adjustment_out == -1) {
                    double t_curr = (device_t);
                    double t_next = (device_t) + device_h_0;

                    if (fabs(device_h_0) < fabs(h_old) && t_next != t_curr) {
                        /* Step was decreased. Undo step, and try again with new h0. */
//                      printf("  [evolve apply] index = %d step decreased, y = y0\n",index);
                        for (int i = 0; i < DIM; i++) {
                            device_y[i] = device_y_0[i];
                        }
                    } else {
                        //                            printf("  [evolve apply] index = %d step decreased h_0 = h_old\n",index);
                        device_h_0 = h_old; /* keep current step size */
                        break;
                    }
                } else {
                    //                        printf("  [evolve apply] index = %d step increased or no change\n",index);
                    break;
                }
            }
            device_h = device_h_0;  /* suggest step size for next time-step */
            h = device_h;
//            printf("    index = %d t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f\n",index,device_t,device_t_0,device_h,device_h_0);
//            for (int i = 0; i < DIM; i++){
//                printf("    index = %d y[%d][%d] = %.10f\n",index,index,i,device_y[i]);
//            }
//            printf("  [evolve apply] index = %d end\n",index);
//            if(device_final_step){
//                printf("[output] index = %d t = %.10f t_0 = %.10f  h = %.10f h_0 = %.10f\n",index,device_t,device_t_0,device_h,device_h_0);
//                for (int i = 0; i < DIM; i++){
//                    printf("[output] index = %d y[%d] = %.10f\n",index,i,device_y[i]);
//                }
//            }
//            device_t = device_t_0 + device_h_0;
        }
//        if(index == 0) {
//            printf("[evolve apply] Index = %d t = %f h = %f end one day\n", index, t, h);
//        }
        if(NUMODE == 1  && t == NUMDAYSOUTPUT - 1 || (index > 0 && index % (NUMODE / 2) == 0) && t == NUMDAYSOUTPUT - 1) {
            printf("ODE %d t = %f h = %f end, y[%d][%d] = %.5f\n", index, t, h, index,gpu_params->ode_dimension - 4, device_y[gpu_params->ode_dimension - 4]);
        }
        t += t_delta;

        /* y_ode_output_d*/
//        for (int i = 0; i < gpu_params->display_dimension; i ++) {
//          const int y_output_index = day * gpu_params->display_dimension + i;
//          if(y_output_index % gpu_params->display_dimension == 0){
//            //First column
//            y_output[index][y_output_index] = day*1.0;
//            //          printf("First day = %d index = %d i = %d y_output_index = %d y_output[%d][%d] = %.5f\n",
//            //                 day, index, i, y_output_index, index, y_output_index, y_output[index][y_output_index]);
//          }
//          else if(y_output_index % gpu_params->display_dimension == 1){
//            //Second column
//            y_output[index][y_output_index] = stf_today;
//            //          printf("Second day = %d index = %d i = %d y_output_index = %d y_output[%d][%d] = %.5f\n",
//            //                 day, index, i, y_output_index, index, y_output_index, y_output[index][y_output_index]);
//          }
//          else if(y_output_index % gpu_params->display_dimension >= 2 && y_output_index % gpu_params->display_dimension < gpu_params->display_dimension - 1){
//            //Third column to column next to last column
//            const int y_index = (y_output_index - 2) % gpu_params->display_dimension;
//            y_output[index][y_output_index] = device_y_yesterday[y_index];
//            //          printf("day = %d index = %d i = %d y_output_index = %d y[%d][%d] = y[%d][%d] = %.5f\n",
//            //                 day, index, i, y_output_index, index, y_output_index, index, y_index, device_y[y_index]);
//          }
//          else{
//            //Last column
////            y_output[index][y_output_index] = pop_sum(device_y);
//            y_output[index][y_output_index] = pop_sum(device_y);
//            //          printf("Third day = %d index = %d i = %d y_output_index = %d y_output[%d][%d] = %.5f\n",
//            //                 day, index, i, y_output_index, index, y_output_index, y_output[index][y_output_index]);
//          }
//        }

        /* y_ode_agg_d*/
        /* AGG Output 1-6 */
        for (int i = 0; i < gpu_params->data_params.cols; i++) {
            const int y_output_agg_index = (day + 1) * gpu_params->agg_dimension + i;
            const int y_output_agg_to_sum_index = (day) * gpu_params->agg_dimension + i;
            const int y_ode_index = gpu_params->ode_dimension - 4 + i;
            y_agg_output[index][y_output_agg_index] = y_agg_input[index][y_output_agg_index];
            if(day == 0) {
                y_agg_output[index][y_output_agg_to_sum_index] = 0.0;
            }
            y_agg_output[index][y_output_agg_index] = device_y[y_ode_index] - device_y_yesterday[y_ode_index];
            agg_inc_sum[i] += y_agg_output[index][y_output_agg_to_sum_index];
        }

        if ((day+1) % 7 == 0 || day == gpu_params->ode_output_day - 1) {
            for(int i = 0; i < gpu_params->data_params.cols; i++){
                //Col 3 4 5
                const int y_output_agg_col = (3 + i) + week_count * gpu_params->agg_dimension;
                y_agg_output[index][y_output_agg_col] = agg_inc_sum[i];
                if(agg_inc_sum[i] >= agg_inc_max[i]) agg_inc_max[i] = agg_inc_sum[i];
                agg_inc_sum[i] = 0.0;
            }
            week_count++;
        }

        //Write max agg inc to first line
        if(day == gpu_params->ode_output_day - 1){
            for(int i = 0; i < DATADIM_COLS; i++){
                //Col 1 2 3
                y_agg_output[index][i] = agg_inc_max[i];
            }
        }
//        if(index == 0 && t == NUMDAYSOUTPUT - 1) {
//            printf("[evolve apply agg] Index = %d t = %f h = %f end, agg_inc_max[0] = %.5f\n", index, t, h, agg_inc_max[0]);
//        }
    }
//    if(index == 0){
//        for (int i = 0; i < DIM; i++){
//            printf("[output] index = %d y[%d] = %1.5f\n",index,i,device_y[i]);
//        }
//    }
}

__device__
void solve_ode_one(double *y_ode_input_d[], double *y_ode_output_d[], double *y_agg_input_d[], double *y_agg_output_d[],  double stf[], int index, GPUParameters *gpu_params, FluParameters* flu_params) {
    rk45_gpu_evolve_apply(gpu_params->ode_t0, gpu_params->ode_t_target, gpu_params->ode_step, gpu_params->ode_h, y_ode_input_d, y_ode_output_d, y_agg_input_d, y_agg_output_d, stf, index, gpu_params, flu_params);
    return;
}

__global__
void solve_ode_n(double *y_ode_input_d[], double *y_ode_output_d[], double *y_agg_input_d[], double *y_agg_output_d[], double* stf[], GPUParameters *gpu_params, FluParameters* flu_params) {
    int index_gpu = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int index = index_gpu; index < gpu_params->ode_number; index += stride) {
//        if(index % 32 == 0){
//            printf("ODE %d will be solved by thread index = %d blockIdx.x = %d\n", index, index, blockIdx.x);
//        }
        solve_ode_one(y_ode_input_d, y_ode_output_d, y_agg_input_d, y_agg_output_d, stf[index], index, gpu_params, flu_params);
    }
    return;
}

__global__
void calculate_stf(double* stf_d[], GPUParameters* gpu_params, FluParameters* flu_params){
    int index_gpu = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int index = index_gpu; index < gpu_params->ode_number * gpu_params->ode_output_day; index += stride) {
        const int ode_index = index / gpu_params->ode_output_day;
        const int day_index = index % gpu_params->ode_output_day;
        double t = day_index*1.0;

//        if((NUMODE == 1  || (index > 0 && index % (NUMODE / 2) == 0)) && t == 0) {
//            printf("\nSTF ODE %d Old phi: ",ode_index);
//            for(int i = 0; i < flu_params[ode_index]->SAMPLE_PHI_LENGTH; i++){
//                printf("%.2f\t",flu_params[ode_index]->phi[i]);
//            }
//            printf("\nSTF ODE %d Old flu_params[%d]->phi_0 = %.5f\n", ode_index, ode_index, flu_params[ode_index]->phi_0);
//            printf("STF ODE %d Old flu_params[%d]->beta[0] = %.10f\n", ode_index, ode_index, flu_params[ode_index]->beta[0]);
//            printf("STF ODE %d Old flu_params[%d]->beta[1] = %.10f\n", ode_index, ode_index, flu_params[ode_index]->beta[1]);
//            printf("STF ODE %d Old flu_params[%d]->beta[2] = %.10f\n", ode_index, ode_index, flu_params[ode_index]->beta[2]);
//            printf("STF ODE %d Old flu_params[%d]->sigma[0] = %.5f\n", ode_index, ode_index, flu_params[ode_index]->sigma[0]);
//            printf("STF ODE %d Old flu_params[%d]->sigma[1] = %.5f\n", ode_index, ode_index, flu_params[ode_index]->sigma[1]);
//            printf("STF ODE %d Old flu_params[%d]->sigma[2] = %.5f\n", ode_index, ode_index, flu_params[ode_index]->sigma[2]);
//            printf("STF ODE %d Old flu_params[%d]->amp = %.5f\n", ode_index, ode_index, flu_params[ode_index]->amp);
//            printf("STF ODE %d Old flu_params[%d]->nu_denom = %.5f\n", ode_index, ode_index, flu_params[ode_index]->nu_denom);
//            printf("STF ODE %d Old flu_params[%d]->rho_denom = %.5f\n", ode_index, ode_index, flu_params[ode_index]->rho_denom);
//        }

        if (SAMPLE_PHI_LENGTH == 0) {
            stf_d[ode_index][day_index] = 1.0;
        }
        else{
            double remainder = day_index - t;
            int xx = day_index % 3650;
            double yy = (double) xx + remainder;
            // put yy into the sine function, let it return the beta value
            t = yy;
            float sine_function_value = 0.0;

            for (int i = 0; i < SAMPLE_PHI_LENGTH; i++) {
                if (fabs(t - flu_params->phi[ode_index*SAMPLE_PHI_LENGTH + i]) < (flu_params->v_d_i_epidur_d2)) {
                    sine_function_value = __sinf(flu_params->pi_x2 * (flu_params->phi[ode_index*SAMPLE_PHI_LENGTH + i] - t + (flu_params->v_d_i_epidur_d2)) /
                                                 (flu_params->v_d_i_epidur_x2));
                }
            }
//        printf("index %d SAMPLE_PHI_LENGTH %d %f sine_function_value %1.3f\n",index,flu_params->SAMPLE_PHI_LENGTH,t,sine_function_value);
//        printf("index %d day %f return %1.5f\n",index,day_index,t,1.0 + flu_params[ode_index]->v_d_i_amp * sine_function_value);
            stf_d[ode_index][day_index] = 1.0 + flu_params->v_d_i_amp * sine_function_value;
//            if(day_index < 10){
//                printf("index %d ODE %d day %d stf_d[%d][%d] = %.5f\n", index, ode_index, day_index, ode_index, day_index, stf_d[ode_index][day_index]);
//            }
//        printf("%d = %.5f\n", day_index, stf_d[ode_index][day_index]);
        }
    }
}
