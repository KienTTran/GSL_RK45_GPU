#include "hip/hip_runtime.h"
//
// Created by kient on 5/1/2022.
//
#include "gpu_ode_mcmc.h"

static const int NUM_ELEMENTS = 512;

__global__
void reduce_sum(double *input, double* output, int len)
{
    __shared__ double s_data[NUM_ELEMENTS];
    int tid = threadIdx.x;
    int index = tid + blockIdx.x*blockDim.x;
    s_data[tid] = 0.0;
    if (index < len){
        s_data[tid] = input[index];
    }
    __syncthreads();

    for (int s = 2; s <= blockDim.x; s = s * 2){
        if ((tid%s) == 0){
            s_data[tid] += s_data[tid + s / 2];
        }
        __syncthreads();
    }

    if (tid == 0){
//        printf("sum = %.5f\n",s_data[tid]);
        output[blockIdx.x] = s_data[tid];
    }
}

