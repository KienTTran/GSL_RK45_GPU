#include "hip/hip_runtime.h"
#include "gpu_flu.cuh"

inline
hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
      fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
      assert(result == hipSuccess);
    }
#endif
    return result;
}

GPUFlu::GPUFlu() {
}

GPUFlu::~GPUFlu() {
}

void GPUFlu::set_gpu_parameters(GPUParameters *gpu_params_) {
    gpu_params = gpu_params_;
}

void GPUFlu::init(){
    checkCuda(hipEventCreate(&start_event));
    checkCuda(hipEventCreate(&stop_event));
    checkCuda(hipEventCreate(&start_event_all));
    checkCuda(hipEventCreate(&stop_event_all));
    checkCuda(hipEventCreate(&start_one_ode_event));
    checkCuda(hipEventCreate(&stop_one_ode_event));
    checkCuda(hipEventCreate(&start_one_stf_event));
    checkCuda(hipEventCreate(&stop_one_stf_event));
    checkCuda(hipEventCreate(&start_one_mcmc_event));
    checkCuda(hipEventCreate(&stop_one_mcmc_event));
    checkCuda(hipEventCreate(&start_one_update_event));
    checkCuda(hipEventCreate(&stop_one_update_event));
    checkCuda(hipEventCreate(&start_one_iter_event));
    checkCuda(hipEventCreate(&stop_one_iter_event));

    checkCuda(hipEventRecord(start_event_all,0));
    checkCuda(hipEventRecord(start_event,0));

    flu_params = new FluParameters();
    flu_params->init();
    gpu_params->init(flu_params);

    ode_double_size = gpu_params->ode_number* sizeof(double);

    /* stf_d - stf on device */
    stf_d_size = gpu_params->ode_output_day * sizeof(double);
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], stf_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->stf[i], stf_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &stf_d, ode_double_size));
    checkCuda(hipMemcpy(stf_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* y_ode_input_d - device */
    y_ode_input_d_size = gpu_params->ode_dimension * sizeof(double);
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_ode_input_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_ode_input[i], y_ode_input_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_ode_input_d, ode_double_size));
    checkCuda(hipMemcpy(y_ode_input_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* y_ode_output_d - device */
    y_ode_output_d_size = gpu_params->ode_output_day * gpu_params->display_dimension * sizeof(double);
    //y_ode_output_d
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_ode_output_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_ode_output[i],y_ode_output_d_size, hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_ode_output_d, ode_double_size));
    checkCuda(hipMemcpy(y_ode_output_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* y_data_input_d - device */
    y_data_input_d_size = gpu_params->data_dimension * sizeof(double);
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_data_input_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_data_input[i], y_data_input_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_data_input_d, ode_double_size));
    checkCuda(hipMemcpy(y_data_input_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* y_agg_input_d - device */
    y_agg_d_size = gpu_params->ode_output_day * gpu_params->agg_dimension * sizeof(double);
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_agg_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_agg[i], y_agg_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_agg_input_d, ode_double_size));
    checkCuda(hipMemcpy(y_agg_input_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* y_agg_output_d - device */
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_agg_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_agg[i], y_agg_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_agg_output_d, ode_double_size));
    checkCuda(hipMemcpy(y_agg_output_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* dnorm 1 ode with padding - on host */
    mcmc_dnorm_1_ode_padding_size = ceil(gpu_params->data_params.rows/(GPU_REDUCE_THREADS*1.0))*GPU_REDUCE_THREADS - gpu_params->data_params.rows;
    y_mcmc_dnorm_1_ode_h = (double*)malloc(gpu_params->data_params.rows*sizeof(double));
    y_mcmc_dnorm_1_ode_h_size = gpu_params->data_params.rows;
    for (int i = 0; i < y_mcmc_dnorm_1_ode_h_size; i++) {
        y_mcmc_dnorm_1_ode_h[i] = 0.0;
    }
    y_mcmc_dnorm_1_ode_padding_h = (double*)malloc((gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size)*sizeof(double));
    y_mcmc_dnorm_1_ode_padding_h_size = gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size;
    memcpy(y_mcmc_dnorm_1_ode_padding_h, y_mcmc_dnorm_1_ode_h, y_mcmc_dnorm_1_ode_h_size * sizeof(double));
    memset(y_mcmc_dnorm_1_ode_padding_h + y_mcmc_dnorm_1_ode_h_size,0,mcmc_dnorm_1_ode_padding_size* sizeof(double));

    /* dnorm N ode with padding - on host */
    y_mcmc_dnorm_n_ode_padding_h = (double*)malloc(gpu_params->ode_number*(gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size)*sizeof(double));
    y_mcmc_dnorm_n_ode_padding_h_size = gpu_params->ode_number*(gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size);
    for(int ode_index = 0; ode_index < gpu_params->ode_number; ode_index++){
        memcpy(y_mcmc_dnorm_n_ode_padding_h + ode_index*y_mcmc_dnorm_1_ode_padding_h_size, y_mcmc_dnorm_1_ode_padding_h, y_mcmc_dnorm_1_ode_padding_h_size * sizeof(double));
    }

    /* dnorm N ode with padding - on device */
    y_mcmc_dnorm_n_ode_padding_d_size = y_mcmc_dnorm_n_ode_padding_h_size * sizeof(double);
    checkCuda(hipMalloc((void **) &y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_d_size));
    checkCuda(hipMalloc((void **) &y_mcmc_dnorm_n_ode_padding_zero_d, y_mcmc_dnorm_n_ode_padding_d_size));
    checkCuda(hipMemcpy(y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_h, y_mcmc_dnorm_n_ode_padding_d_size,hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(y_mcmc_dnorm_n_ode_padding_zero_d, y_mcmc_dnorm_n_ode_padding_h, y_mcmc_dnorm_n_ode_padding_d_size,hipMemcpyHostToDevice));

    /* gpu_params_d - on device */
    checkCuda(hipMalloc((void **) &gpu_params_d, sizeof(GPUParameters)));
    checkCuda(hipMemcpy(gpu_params_d, gpu_params, sizeof(GPUParameters), hipMemcpyHostToDevice));

    /* flu_params_current_d - on device */
    checkCuda(hipMalloc((void **) &flu_params_current_d, sizeof(FluParameters)));
    checkCuda(hipMemcpy(flu_params_current_d, flu_params, sizeof(FluParameters), hipMemcpyHostToDevice));

    /* flu_params_new_d - on device */
    checkCuda(hipMalloc((void **) &flu_params_new_d, sizeof(FluParameters)));
    checkCuda(hipMemcpy(flu_params_new_d, flu_params, sizeof(FluParameters), hipMemcpyHostToDevice));

    /* r_denom/r_num - on host */
    r_h = (double*)malloc(ode_double_size);
    memset(r_h,0,ode_double_size);

    /* r_denom_d - on device */
    checkCuda(hipMalloc((void **) &r_denom_d, ode_double_size));
    checkCuda(hipMemcpy(r_denom_d, r_h, ode_double_size,hipMemcpyHostToDevice));

    /* r_num_d - on device */
    checkCuda(hipMalloc((void **) &r_num_d, ode_double_size));
    checkCuda(hipMemcpy(r_num_d, r_h, ode_double_size,hipMemcpyHostToDevice));

    /* norm_h - on host */
    norm_size = gpu_params->ode_number * SAMPLE_LENGTH;
    norm_h = (double*)malloc(norm_size * sizeof(double));
    for (int i = 0; i < norm_size; i++) {
        norm_h[i] = 0.0;
    }

    /* norm_d - on device */
    checkCuda(hipMalloc((void **) &norm_d, norm_size * sizeof(double)));
    checkCuda(hipMemcpy(norm_d, norm_h, norm_size * sizeof(double),hipMemcpyHostToDevice));

    /* norm_and_sd_d - on device */
    checkCuda(hipMalloc((void **) &norm_sd_d, norm_size * sizeof(double)));
    checkCuda(hipMemcpy(norm_d, norm_h, norm_size * sizeof(double),hipMemcpyHostToDevice));

    /* curand_state_d - on device */
    checkCuda(hipMalloc((void **)&curand_state_d, norm_size * sizeof(hiprandState)));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    /* Blocks to process other things, must be equal number of ODE */
    gpu_params->block_size = GPU_ODE_THREADS; //max is 1024
    gpu_params->num_blocks = (gpu_params->ode_number + gpu_params->block_size - 1) / gpu_params->block_size;
    /* Blocks to process reduction sum with padding, must be divided by 1024 */
    gpu_reduce_num_block = ceil(prop.maxBlocksPerMultiProcessor * prop.maxThreadsPerBlock / GPU_REDUCE_THREADS);
    printf("GPU reduce threads = %d block = %d\n",prop.maxBlocksPerMultiProcessor * prop.maxThreadsPerBlock, gpu_reduce_num_block);

    checkCuda(hipEventRecord(stop_event, 0));
    checkCuda(hipEventSynchronize(stop_event));
    checkCuda(hipEventElapsedTime(&transfer_h2d_ms, start_event, stop_event));
}

void GPUFlu::run() {
    checkCuda(hipEventRecord(start_event,0));
    //hipProfilerStart();

    /* Setup prng states */
    mcmc_setup_states_for_random<<<gpu_params->num_blocks, gpu_params->block_size>>>(curand_state_d, norm_size);
    for (int iter = 0; iter < gpu_params->mcmc_loop; iter++) {
        checkCuda(hipEventRecord(start_one_iter_event,0));
        if(iter == 0){
            /* Calculate stf */
            calculate_stf<<<gpu_params->num_blocks, gpu_params->block_size>>>(stf_d, gpu_params_d, flu_params_current_d);
            /* Calculate ODE */
            solve_ode_n<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_ode_input_d, y_ode_output_d, y_agg_input_d, y_agg_output_d, stf_d, gpu_params_d, flu_params_current_d);
            /* Calculate dnorm */
            mcmc_dnorm_padding<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_data_input_d, y_agg_output_d, y_mcmc_dnorm_n_ode_padding_d, mcmc_dnorm_1_ode_padding_size, gpu_params_d);
            /* Calculate sum dnorm */
            reduce_sum_padding<<<gpu_reduce_num_block, GPU_REDUCE_THREADS>>>(y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_d, gpu_params_d, y_mcmc_dnorm_n_ode_padding_h_size);
            /* Calculate R_denom */
            mcmc_compute_r<<<gpu_reduce_num_block, GPU_REDUCE_THREADS>>>(y_mcmc_dnorm_n_ode_padding_d, r_denom_d, gpu_params_d);
        }

        //
        // Generate new parameters
        //

        checkCuda(hipEventRecord(start_one_update_event,0));
        /* Reset dnorm vector on device */
        checkCuda(hipMemcpy(y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_zero_d, y_mcmc_dnorm_n_ode_padding_d_size, hipMemcpyDeviceToDevice));
        /* Update new flu parameters */
        mcmc_generate_norm<<<gpu_reduce_num_block, GPU_REDUCE_THREADS>>>(norm_d, norm_size, curand_state_d);
        mcmc_update_parameters<<<gpu_params->num_blocks, gpu_params->block_size>>>(gpu_params_d, flu_params_current_d, flu_params_new_d, curand_state_d);
        checkCuda(hipEventRecord(stop_one_update_event,0));
        checkCuda(hipEventSynchronize(stop_one_update_event));
        checkCuda(hipEventElapsedTime(&one_update_ms, start_one_update_event, stop_one_update_event));

        //
        // Solve ode with new parameters
        //

        /* Calculate stf */
        checkCuda(hipEventRecord(start_one_stf_event,0));
        calculate_stf<<<gpu_params->num_blocks, gpu_params->block_size>>>(stf_d, gpu_params_d, flu_params_new_d);
        checkCuda(hipEventRecord(stop_one_stf_event,0));
        checkCuda(hipEventSynchronize(stop_one_stf_event));
        checkCuda(hipEventElapsedTime(&one_stf_ms, start_one_stf_event, stop_one_stf_event));
        /* Calculate ODE */
        checkCuda(hipEventRecord(start_one_ode_event,0));
        solve_ode_n<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_ode_input_d, y_ode_output_d, y_agg_input_d, y_agg_output_d, stf_d, gpu_params_d, flu_params_new_d);
        checkCuda(hipEventRecord(stop_one_ode_event,0));
        checkCuda(hipEventSynchronize(stop_one_ode_event));
        checkCuda(hipEventElapsedTime(&one_ode_ms, start_one_ode_event, stop_one_ode_event));

        checkCuda(hipEventRecord(start_one_mcmc_event,0));
        /* Calculate dnorm */
        mcmc_dnorm_padding<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_data_input_d, y_agg_output_d, y_mcmc_dnorm_n_ode_padding_d, mcmc_dnorm_1_ode_padding_size, gpu_params_d);
        /* Calculate sum dnorm */
        reduce_sum_padding<<<gpu_reduce_num_block, GPU_REDUCE_THREADS>>>(y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_d, gpu_params_d, y_mcmc_dnorm_n_ode_padding_h_size);
        /* Calculate R_num */
        mcmc_compute_r<<<gpu_reduce_num_block, GPU_REDUCE_THREADS>>>(y_mcmc_dnorm_n_ode_padding_d, r_num_d, gpu_params_d);
        /* Accept or reject new parameters */
        mcmc_check_acceptance<<<gpu_params->num_blocks, gpu_params->block_size>>>(r_denom_d, r_num_d, gpu_params_d, flu_params_current_d, flu_params_new_d, curand_state_d);
        checkCuda(hipEventRecord(stop_one_mcmc_event,0));
        checkCuda(hipEventSynchronize(stop_one_mcmc_event));
        checkCuda(hipEventElapsedTime(&one_mcmc_ms, start_one_mcmc_event, stop_one_mcmc_event));

        checkCuda(hipEventRecord(stop_one_iter_event, 0));
        checkCuda(hipEventSynchronize(stop_one_iter_event));
        checkCuda(hipEventElapsedTime(&one_iter_ms, start_one_iter_event, stop_one_iter_event));
	    checkCuda(hipDeviceSynchronize());
        printf("==== iter %d update done in %f seconds ====\n",iter,(one_update_ms/1e3));
        printf("==== iter %d stf done in %f seconds ====\n",iter,(one_stf_ms/1e3));
        printf("==== iter %d ode done in %f seconds ====\n",iter,(one_ode_ms/1e3));
        printf("==== iter %d mcmc done in %f seconds ====\n",iter,(one_mcmc_ms/1e3));
	    printf("==== iter %d update-stf-ode-mcmc done in %f seconds ====\n\n",iter,(one_iter_ms/1e3));
    }

//    hipProfilerStop();
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipEventRecord(stop_event, 0));
    checkCuda(hipEventSynchronize(stop_event));
    checkCuda(hipEventElapsedTime(&compute_ms, start_event, stop_event));

    checkCuda(hipEventRecord(start_event,0));

    //y_ode_output_h
    y_ode_output_h = (double **) malloc(ode_double_size);
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        y_ode_output_h[i] = (double *) malloc(gpu_params->ode_output_day * gpu_params->display_dimension * sizeof(double));
    }
    checkCuda(hipMemcpy(tmp_ptr, y_ode_output_d, ode_double_size, hipMemcpyDeviceToHost));
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMemcpy(y_ode_output_h[i], tmp_ptr[i], gpu_params->ode_output_day * gpu_params->display_dimension * sizeof(double),
                             hipMemcpyDeviceToHost));
    }

    //y_output_agg_h
    y_output_agg_h = (double **) malloc(ode_double_size);
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        y_output_agg_h[i] = (double *) malloc(gpu_params->ode_output_day * gpu_params->agg_dimension * sizeof(double));
    }
    checkCuda(hipMemcpy(tmp_ptr, y_agg_output_d, ode_double_size, hipMemcpyDeviceToHost));
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMemcpy(y_output_agg_h[i], tmp_ptr[i], gpu_params->ode_output_day * gpu_params->agg_dimension * sizeof(double),
                             hipMemcpyDeviceToHost));
    }
    checkCuda(hipDeviceSynchronize());

    checkCuda(hipEventRecord(stop_event, 0));
    checkCuda(hipEventSynchronize(stop_event));
    checkCuda(hipEventElapsedTime(&transfer_d2h_ms, start_event, stop_event));

    checkCuda(hipEventRecord(stop_event_all, 0));
    checkCuda(hipEventSynchronize(stop_event_all));
    checkCuda(hipEventElapsedTime(&all_ms, start_event_all, stop_event_all));

    std::random_device rd; // obtain a random number from hardware
    std::mt19937 gen(rd()); // seed the generator
    std::uniform_int_distribution<> distr(0, NUMODE); // define the range

//    for (int i = 0; i < gpu_params->display_number; i++) {
//        int random_index = 0;
//        if (NUMODE == 1) {
//            random_index = 0;
//        } else {
//            random_index = distr(gen);
//        }
//        printf("Display y_ode_output_h[%d]\n",random_index);
//        for(int index = 0; index < gpu_params->ode_output_day * gpu_params->display_dimension; index++){
//            const int line_index = (index / gpu_params->display_dimension) % NUMDAYSOUTPUT;
//            if(line_index < 10)
//            {
//                printf("%.5f\t", y_ode_output_h[random_index][index]);
//                if (index > 0 && (index + 1) % gpu_params->display_dimension == 0) {
//                    printf("\n");
//                }
//            }
//        }
//        printf("Display y_output_agg_h[%d]\n", random_index);
//        for (int index = 0; index < gpu_params->ode_output_day * gpu_params->agg_dimension; index++) {
//            const int line_index = (index / gpu_params->agg_dimension);
//            if(line_index < 10)
//            {
//                printf("%d %.5f\t",line_index, y_output_agg_h[random_index][index]);
//                if (index > 0 && (index + 1) % gpu_params->agg_dimension == 0) {
//                    printf("\n");
//                }
//            }
//        }
//    }
//    printf("\n");

    checkCuda(hipDeviceSynchronize());

    printf("[GPU FLU] GPU Time for transfer data from CPU to GPU: %f milliseconds which is %f seconds\n",transfer_h2d_ms,(transfer_h2d_ms/1e3));
    printf("[GPU FLU] GPU Time for compute MCMC %d iteration with %d ODE(s) with %d parameters, step %f in %f days on GPU: %f milliseconds which is %f seconds\n",
           gpu_params->mcmc_loop,gpu_params->ode_number,gpu_params->ode_dimension,gpu_params->h,gpu_params->t_target,compute_ms,(compute_ms/1e3));
    printf("[GPU FLU] GPU Time for transfer data from GPU on CPU: %f milliseconds which is %f seconds\n",transfer_d2h_ms,(transfer_d2h_ms/1e3));
    printf("[GPU FLU] GPU Time for complete MCMC %d iteration with %d ODE(s) with %d parameters: %f milliseconds which is %f seconds\n",
           gpu_params->mcmc_loop,gpu_params->ode_number,gpu_params->ode_dimension,all_ms,(all_ms/1e3));

    // Free memory

    checkCuda(hipEventDestroy(start_event));
    checkCuda(hipEventDestroy(stop_event));
    checkCuda(hipEventDestroy(start_event_all));
    checkCuda(hipEventDestroy(stop_event_all));
    checkCuda(hipEventDestroy(start_one_ode_event));
    checkCuda(hipEventDestroy(stop_one_ode_event));
    checkCuda(hipEventDestroy(start_one_stf_event));
    checkCuda(hipEventDestroy(stop_one_stf_event));
    checkCuda(hipEventDestroy(start_one_mcmc_event));
    checkCuda(hipEventDestroy(stop_one_mcmc_event));
    checkCuda(hipEventDestroy(start_one_update_event));
    checkCuda(hipEventDestroy(stop_one_update_event));
    checkCuda(hipEventDestroy(start_one_iter_event));
    checkCuda(hipEventDestroy(stop_one_iter_event));

    checkCuda(hipFree(y_ode_input_d));
    checkCuda(hipFree(y_ode_output_d));
    checkCuda(hipFree(y_agg_input_d));
    checkCuda(hipFree(y_agg_output_d));
    checkCuda(hipFree(y_data_input_d));
    checkCuda(hipFree(y_mcmc_dnorm_n_ode_padding_d));
    checkCuda(hipFree(gpu_params_d));
    checkCuda(hipFree(flu_params_current_d));
    checkCuda(hipFree(flu_params_new_d));
    checkCuda(hipFree(norm_d));
    checkCuda(hipFree(norm_sd_d));
    checkCuda(hipFree(stf_d));
    checkCuda(hipFree(curand_state_d));
    checkCuda(hipFree(r_denom_d));
    checkCuda(hipFree(r_num_d));
    gpu_params = nullptr;
    flu_params = nullptr;
    delete y_ode_output_h;
    delete y_output_agg_h;
    delete norm_h;
    delete r_h;

    delete [] tmp_ptr;
    return;
}
