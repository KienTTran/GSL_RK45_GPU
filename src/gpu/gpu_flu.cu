#include "hip/hip_runtime.h"
#include "gpu_flu.cuh"

inline
hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
      fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
      assert(result == hipSuccess);
    }
#endif
    return result;
}

GPUFlu::GPUFlu() {
}

GPUFlu::~GPUFlu() {
}

void GPUFlu::set_gpu_parameters(GPUParameters *gpu_params_) {
    gpu_params = gpu_params_;
}

void GPUFlu::init() {
    flu_params = new FluParameters();
    flu_params->init();
    gpu_params->init(flu_params);
}

void GPUFlu::run() {
    auto start = std::chrono::high_resolution_clock::now();
    size_t ode_double_size = gpu_params->ode_number* sizeof(double);
    /* stf_h - stf on host */
    double** stf_h = new double*[gpu_params->ode_number]();
    for(int i = 0; i < gpu_params->ode_number; i++){
        stf_h[i] = new double[gpu_params->ode_output_day];
        for(int j = 0; j < gpu_params->ode_output_day; j++) {
            stf_h[i][j] = 7.0;
        }
    }
    /* stf_d - stf on device */
    double **stf_d = 0;
    size_t stf_d_size = gpu_params->ode_output_day * sizeof(double);
    //temp pointers
    double **tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], stf_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], stf_h[i], stf_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &stf_d, ode_double_size));
    checkCuda(hipMemcpy(stf_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* y_ode_input_d - device */
    double **y_ode_input_d = 0;
    size_t y_ode_input_d_size = gpu_params->ode_dimension * sizeof(double);
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_ode_input_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_ode_input[i], y_ode_input_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_ode_input_d, ode_double_size));
    checkCuda(hipMemcpy(y_ode_input_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* y_ode_output_d - device */
    double **y_ode_output_d = 0;
    size_t y_ode_output_d_size = gpu_params->ode_output_day * gpu_params->display_dimension * sizeof(double);
    //y_ode_output_d
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_ode_output_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_ode_output[i],y_ode_output_d_size, hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_ode_output_d, ode_double_size));
    checkCuda(hipMemcpy(y_ode_output_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* y_data_input_d - device */
    double **y_data_input_d = 0;
    size_t y_data_input_d_size = gpu_params->data_dimension * sizeof(double);
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_data_input_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_data_input[i], y_data_input_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_data_input_d, ode_double_size));
    checkCuda(hipMemcpy(y_data_input_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* y_agg_input_d - device */
    double **y_agg_input_d = 0;
    size_t y_agg_d_size = gpu_params->ode_output_day * gpu_params->agg_dimension * sizeof(double);
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_agg_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_agg[i], y_agg_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_agg_input_d, ode_double_size));
    checkCuda(hipMemcpy(y_agg_input_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* y_agg_output_d - device */
    double **y_agg_output_d = 0;
    //temp pointers
    tmp_ptr = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_agg_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_agg[i], y_agg_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_agg_output_d, ode_double_size));
    checkCuda(hipMemcpy(y_agg_output_d, tmp_ptr, ode_double_size, hipMemcpyHostToDevice));

    /* dnorm 1 ode with padding - on host */
    int mcmc_dnorm_1_ode_padding_size = ceil(gpu_params->data_params.rows/(GPU_REDUCE_THREADS*1.0))*GPU_REDUCE_THREADS - gpu_params->data_params.rows;
    double *y_mcmc_dnorm_1_ode_h = (double*)malloc(gpu_params->data_params.rows*sizeof(double));
    int y_mcmc_dnorm_1_ode_h_size = gpu_params->data_params.rows;
    for (int i = 0; i < y_mcmc_dnorm_1_ode_h_size; i++) {
        y_mcmc_dnorm_1_ode_h[i] = 0.0;
    }
    double *y_mcmc_dnorm_1_ode_padding_h = (double*)malloc((gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size)*sizeof(double));
    int y_mcmc_dnorm_1_ode_padding_h_size = gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size;
    memcpy(y_mcmc_dnorm_1_ode_padding_h, y_mcmc_dnorm_1_ode_h, y_mcmc_dnorm_1_ode_h_size * sizeof(double));
    memset(y_mcmc_dnorm_1_ode_padding_h + y_mcmc_dnorm_1_ode_h_size,0,mcmc_dnorm_1_ode_padding_size* sizeof(double));

    /* dnorm N ode with padding - on host */
    double *y_mcmc_dnorm_n_ode_padding_h = (double*)malloc(gpu_params->ode_number*(gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size)*sizeof(double));
    int y_mcmc_dnorm_n_ode_padding_h_size = gpu_params->ode_number*(gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size);
    for(int ode_index = 0; ode_index < gpu_params->ode_number; ode_index++){
        memcpy(y_mcmc_dnorm_n_ode_padding_h + ode_index*y_mcmc_dnorm_1_ode_padding_h_size, y_mcmc_dnorm_1_ode_padding_h, y_mcmc_dnorm_1_ode_padding_h_size * sizeof(double));
    }

    /* dnorm N ode with padding - on device */
    double* y_mcmc_dnorm_n_ode_padding_d = 0;
    double* y_mcmc_dnorm_n_ode_padding_zero_d;
    size_t y_mcmc_dnorm_n_ode_padding_d_size = y_mcmc_dnorm_n_ode_padding_h_size * sizeof(double);
    checkCuda(hipMalloc((void **) &y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_d_size));
    checkCuda(hipMalloc((void **) &y_mcmc_dnorm_n_ode_padding_zero_d, y_mcmc_dnorm_n_ode_padding_d_size));
    checkCuda(hipMemcpy(y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_h, y_mcmc_dnorm_n_ode_padding_d_size,hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(y_mcmc_dnorm_n_ode_padding_zero_d, y_mcmc_dnorm_n_ode_padding_h, y_mcmc_dnorm_n_ode_padding_d_size,hipMemcpyHostToDevice));

    /* gpu_params_d - on device */
    GPUParameters *gpu_params_d;
    checkCuda(hipMalloc((void **) &gpu_params_d, sizeof(GPUParameters)));
    checkCuda(hipMemcpy(gpu_params_d, gpu_params, sizeof(GPUParameters), hipMemcpyHostToDevice));

    FluParameters *flu_params_current_d;
    checkCuda(hipMalloc((void **) &flu_params_current_d, sizeof(FluParameters)));
    checkCuda(hipMemcpy(flu_params_current_d, flu_params, sizeof(FluParameters), hipMemcpyHostToDevice));

    FluParameters *flu_params_new_d;
    checkCuda(hipMalloc((void **) &flu_params_new_d, sizeof(FluParameters)));
    checkCuda(hipMemcpy(flu_params_new_d, flu_params, sizeof(FluParameters), hipMemcpyHostToDevice));


    /* curand_state_d - on device */
    hiprandState *curand_state_d;
    checkCuda(hipMalloc((void **)&curand_state_d, gpu_params->ode_number * sizeof(hiprandState)));

    /* r_denom/r_num - on host */
    double *r_h = (double*)malloc(gpu_params->ode_number*sizeof(double));
    memset(r_h,0,gpu_params->ode_number*sizeof(double));

    /* r_denom - on device */
    size_t r_d_size = gpu_params->ode_number * sizeof(double);
    double* r_denom_d = 0;
    checkCuda(hipMalloc((void **) &r_denom_d, r_d_size));
    checkCuda(hipMemcpy(r_denom_d, r_h, r_d_size,hipMemcpyHostToDevice));

    /* r_num - on device */
    double* r_num_d = 0;
    checkCuda(hipMalloc((void **) &r_num_d, r_d_size));
    checkCuda(hipMemcpy(r_num_d, r_h, r_d_size,hipMemcpyHostToDevice));

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for allocate mem CPU to GPU: %ld micro seconds which is %.10f seconds\n", duration.count(),
           (duration.count() / 1e6));

    start = std::chrono::high_resolution_clock::now();
    //hipProfilerStart();

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    /* Blocks to process other things, must be equal number of ODE */
    gpu_params->block_size = GPU_ODE_THREADS; //max is 1024
    gpu_params->num_blocks = (gpu_params->ode_number + gpu_params->block_size - 1) / gpu_params->block_size;
    /* Blocks to process reduction sum with padding, must be divided by 1024 */
    int num_block = ceil(prop.maxBlocksPerMultiProcessor * prop.maxThreadsPerBlock / GPU_REDUCE_THREADS);
    printf("max threads = %d block = %d\n",prop.maxBlocksPerMultiProcessor * prop.maxThreadsPerBlock, num_block);

    /* Setup prng states */
    mcmc_setup_states_for_random<<<gpu_params->num_blocks, gpu_params->block_size>>>(curand_state_d);
    for (int iter = 0; iter < gpu_params->mcmc_loop; iter++) {
        if(iter == 0){
            /* Calculate stf */
            calculate_stf<<<gpu_params->num_blocks, gpu_params->block_size>>>(stf_d, gpu_params_d, flu_params_current_d);
            /* Calculate ODE */
            solve_ode<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_ode_input_d, y_ode_output_d, y_agg_input_d, y_agg_output_d, stf_d, gpu_params_d, flu_params_current_d);
            /* Calculate dnorm */
            mcmc_dnorm_padding<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_data_input_d, y_agg_output_d, y_mcmc_dnorm_n_ode_padding_d, mcmc_dnorm_1_ode_padding_size, gpu_params_d);
            /* Calculate sum dnorm */
            reduce_sum_padding<<<num_block, GPU_REDUCE_THREADS>>>(y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_d, gpu_params_d, y_mcmc_dnorm_n_ode_padding_h_size);
            /* Calculate R_denom */
            mcmc_compute_r<<<num_block, GPU_REDUCE_THREADS>>>(y_mcmc_dnorm_n_ode_padding_d, r_denom_d, gpu_params_d);
        }
//        mcmc_print_r<<<gpu_params->num_blocks, gpu_params->block_size>>>(gpu_params_d, r_denom_d);

        //
        // Generate new parameters
        //

        /* Reset dnorm vector on device */
        checkCuda(hipMemcpy(y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_zero_d, y_mcmc_dnorm_n_ode_padding_d_size, hipMemcpyDeviceToDevice));
        /* Update new flu parameters */
        mcmc_update_parameters<<<gpu_params->num_blocks, gpu_params->block_size>>>(gpu_params_d, flu_params_current_d, flu_params_new_d, curand_state_d);

        //
        // Solve ode with new parameters
        //

        /* Calculate stf */
        calculate_stf<<<gpu_params->num_blocks, gpu_params->block_size>>>(stf_d, gpu_params_d, flu_params_new_d);
        /* Calculate ODE */
        solve_ode<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_ode_input_d, y_ode_output_d, y_agg_input_d, y_agg_output_d, stf_d, gpu_params_d, flu_params_new_d);
//        /* Calculate dnorm */
        mcmc_dnorm_padding<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_data_input_d, y_agg_output_d, y_mcmc_dnorm_n_ode_padding_d, mcmc_dnorm_1_ode_padding_size, gpu_params_d);
        /* Calculate sum dnorm */
        reduce_sum_padding<<<num_block, GPU_REDUCE_THREADS>>>(y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_d, gpu_params_d, y_mcmc_dnorm_n_ode_padding_h_size);
        /* Calculate R_num */
        mcmc_compute_r<<<num_block, GPU_REDUCE_THREADS>>>(y_mcmc_dnorm_n_ode_padding_d, r_num_d, gpu_params_d);
//        mcmc_print_r<<<gpu_params->num_blocks, gpu_params->block_size>>>(gpu_params_d, r_num_d);

        /* Accept or reject new parameters */
        mcmc_check_acceptance<<<gpu_params->num_blocks, gpu_params->block_size>>>(r_denom_d, r_num_d, gpu_params_d, flu_params_current_d, flu_params_new_d, curand_state_d);

        checkCuda(hipDeviceSynchronize());
        printf("==== iter %d done ====\n",iter);
    }

    //    hipProfilerStop();
    checkCuda(hipDeviceSynchronize());
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for MCMC %d ODE with %d parameters %d times on GPU: %ld micro seconds which is %.10f seconds\n",
           NUMODE, DIM, gpu_params->mcmc_loop, duration.count(), (duration.count() / 1e6));

    start = std::chrono::high_resolution_clock::now();
    //y_ode_output_h
    tmp_ptr = (double **) malloc(ode_double_size);
    double **y_ode_output_h = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        y_ode_output_h[i] = (double *) malloc(gpu_params->ode_output_day * gpu_params->display_dimension * sizeof(double));
    }
    checkCuda(hipMemcpy(tmp_ptr, y_ode_output_d, ode_double_size, hipMemcpyDeviceToHost));
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMemcpy(y_ode_output_h[i], tmp_ptr[i], gpu_params->ode_output_day * gpu_params->display_dimension * sizeof(double),
                             hipMemcpyDeviceToHost));
    }
    //y_output_agg_h
    tmp_ptr = (double **) malloc(ode_double_size);
    double **y_output_agg_h = (double **) malloc(ode_double_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        y_output_agg_h[i] = (double *) malloc(gpu_params->ode_output_day * gpu_params->agg_dimension * sizeof(double));
    }
    checkCuda(hipMemcpy(tmp_ptr, y_agg_output_d, ode_double_size, hipMemcpyDeviceToHost));
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMemcpy(y_output_agg_h[i], tmp_ptr[i], gpu_params->ode_output_day * gpu_params->agg_dimension * sizeof(double),
                             hipMemcpyDeviceToHost));
    }
    checkCuda(hipDeviceSynchronize());
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for data transfer GPU to CPU: %ld micro seconds which is %.10f seconds\n", duration.count(),
           (duration.count() / 1e6));

    start = std::chrono::high_resolution_clock::now();
    std::random_device rd; // obtain a random number from hardware
    std::mt19937 gen(rd()); // seed the generator
    std::uniform_int_distribution<> distr(0, NUMODE); // define the range

    for (int i = 0; i < gpu_params->display_number; i++) {
        int random_index = 0;
        if (NUMODE == 1) {
            random_index = 0;
        } else {
            random_index = distr(gen);
        }
//        printf("Display y_ode_output_h[%d]\n",random_index);
//        for(int index = 0; index < gpu_params->ode_output_day * gpu_params->display_dimension; index++){
//            const int line_index = (index / gpu_params->display_dimension) % NUMDAYSOUTPUT;
//            if(line_index < 10)
//            {
//                printf("%.5f\t", y_ode_output_h[random_index][index]);
//                if (index > 0 && (index + 1) % gpu_params->display_dimension == 0) {
//                    printf("\n");
//                }
//            }
//        }
        printf("Display y_output_agg_h[%d]\n", random_index);
        for (int index = 0; index < gpu_params->ode_output_day * gpu_params->agg_dimension; index++) {
            const int line_index = (index / gpu_params->agg_dimension);
            if(line_index < 10)
            {
                printf("%d %.5f\t",line_index, y_output_agg_h[random_index][index]);
                if (index > 0 && (index + 1) % gpu_params->agg_dimension == 0) {
                    printf("\n");
                }
            }
        }
    }
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for display random results on CPU: %ld micro seconds which is %.10f seconds\n",
           duration.count(), (duration.count() / 1e6));
    printf("\n");
    checkCuda(hipDeviceSynchronize());
    // Free memory
    checkCuda(hipFree(y_ode_input_d));
    checkCuda(hipFree(y_ode_output_d));
    checkCuda(hipFree(y_agg_input_d));
    checkCuda(hipFree(y_agg_output_d));
    checkCuda(hipFree(y_data_input_d));
    checkCuda(hipFree(y_mcmc_dnorm_n_ode_padding_d));
    checkCuda(hipFree(gpu_params_d));
    checkCuda(hipFree(flu_params_current_d));
    checkCuda(hipFree(flu_params_new_d));
    gpu_params = nullptr;
    flu_params = nullptr;
    delete y_ode_output_h;
    delete y_output_agg_h;
    delete [] stf_h;
    delete [] tmp_ptr;
    return;
}