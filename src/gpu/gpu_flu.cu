#include "hip/hip_runtime.h"
#include "gpu_flu.cuh"

inline
hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
      fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
      assert(result == hipSuccess);
    }
#endif
    return result;
}

GPUFlu::GPUFlu() {
    gpu_params = new GPUParameters();
    flu_params = new FluParameters*[NUMODE]();
}

GPUFlu::~GPUFlu() {
    gpu_params = nullptr;
    for(int i = 0; i < NUMODE; i++){
        flu_params[i] = nullptr;
    }
}

void GPUFlu::set_gpu_parameters(GPUParameters *gpu_params_) {
    gpu_params = &(*gpu_params_);
}

void GPUFlu::set_flu_parameters(FluParameters *flu_params_[]) {
    for(int i = 0; i < NUMODE; i++){
        flu_params[i] = &(*flu_params_[i]);
    }
}

void GPUFlu::init() {
    gpu_params->init(flu_params);
}

void GPUFlu::run() {
    auto start = std::chrono::high_resolution_clock::now();
    size_t ode_size = gpu_params->ode_number* sizeof(double);
    //stf_h
    double stf_h[gpu_params->ode_number][gpu_params->ode_output_day];
    for(int i = 0; i < gpu_params->ode_number; i++){
        for(int j = 0; j < gpu_params->ode_output_day; j++) {
            stf_h[i][j] = 0.0;
        }
    }
    //stf_d
    double **stf_d = 0;
    size_t stf_d_size = gpu_params->ode_dimension * sizeof(double);
    //temp pointers
    double **tmp_ptr = (double **) malloc(ode_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], stf_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], stf_h[i], stf_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &stf_d, ode_size));
    checkCuda(hipMemcpy(stf_d, tmp_ptr, ode_size, hipMemcpyHostToDevice));

    //y_ode_input_d
    double **y_ode_input_d = 0;
    size_t y_ode_input_d_size = gpu_params->ode_dimension * sizeof(double);
    //temp pointers
    tmp_ptr = (double **) malloc(ode_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_ode_input_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_ode_input[i], y_ode_input_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_ode_input_d, ode_size));
    checkCuda(hipMemcpy(y_ode_input_d, tmp_ptr, ode_size, hipMemcpyHostToDevice));

    //y_ode_output_d
    double **y_ode_output_d = 0;
    size_t y_ode_output_d_size = gpu_params->ode_output_day * gpu_params->display_dimension * sizeof(double);
    //y_ode_output_d
    //temp pointers
    tmp_ptr = (double **) malloc(ode_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_ode_output_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_ode_output[i],y_ode_output_d_size, hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_ode_output_d, ode_size));
    checkCuda(hipMemcpy(y_ode_output_d, tmp_ptr, ode_size, hipMemcpyHostToDevice));

    //y_data_input_d
    double **y_data_input_d = 0;
    size_t y_data_input_d_size = gpu_params->data_dimension * sizeof(double);
    //temp pointers
    tmp_ptr = (double **) malloc(ode_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_data_input_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_data_input[i], y_data_input_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_data_input_d, ode_size));
    checkCuda(hipMemcpy(y_data_input_d, tmp_ptr, ode_size, hipMemcpyHostToDevice));

    //y_agg_input_d
    double **y_agg_input_d = 0;
    size_t y_agg_d_size = gpu_params->ode_output_day * gpu_params->agg_dimension * sizeof(double);
    //temp pointers
    tmp_ptr = (double **) malloc(ode_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_agg_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_agg[i], y_agg_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_agg_input_d, ode_size));
    checkCuda(hipMemcpy(y_agg_input_d, tmp_ptr, ode_size, hipMemcpyHostToDevice));

    //y_agg_output_d
    double **y_agg_output_d = 0;
    //temp pointers
    tmp_ptr = (double **) malloc(ode_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &tmp_ptr[i], y_agg_d_size));
        checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_agg[i], y_agg_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &y_agg_output_d, ode_size));
    checkCuda(hipMemcpy(y_agg_output_d, tmp_ptr, ode_size, hipMemcpyHostToDevice));

    //Calculate padding dnorm N ODE
    int mcmc_dnorm_1_ode_padding_size = ceil(gpu_params->data_params.rows/1024.0)*1024 - gpu_params->data_params.rows;
    double y_mcmc_dnorm_1_ode_h[gpu_params->data_params.rows];
    int y_mcmc_dnorm_1_ode_h_size = gpu_params->data_params.rows;
    for (int i = 0; i < y_mcmc_dnorm_1_ode_h_size; i++) {
        y_mcmc_dnorm_1_ode_h[i] = 0.0;
    }
    double y_mcmc_dnorm_1_ode_padding_h[gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size];
    int y_mcmc_dnorm_1_ode_padding_h_size = gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size;
    memcpy(y_mcmc_dnorm_1_ode_padding_h, y_mcmc_dnorm_1_ode_h, y_mcmc_dnorm_1_ode_h_size * sizeof(double));
    memset(y_mcmc_dnorm_1_ode_padding_h + y_mcmc_dnorm_1_ode_h_size,0,mcmc_dnorm_1_ode_padding_size* sizeof(double));
//    for(int i = 0; i < y_mcmc_dnorm_1_ode_padding_h_size; i++){
//        printf("y_mcmc_dnorm_1_ode_padding_h_size[%d] = %.1f\n",i,y_mcmc_dnorm_1_ode_padding_h[i]);
//    }

    double y_mcmc_dnorm_n_ode_padding_h[gpu_params->ode_number*(gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size)];
    int y_mcmc_dnorm_n_ode_padding_h_size = gpu_params->ode_number*(gpu_params->data_params.rows + mcmc_dnorm_1_ode_padding_size);
    for(int ode_index = 0; ode_index < gpu_params->ode_number; ode_index++){
        memcpy(y_mcmc_dnorm_n_ode_padding_h + ode_index*y_mcmc_dnorm_1_ode_padding_h_size, y_mcmc_dnorm_1_ode_padding_h, y_mcmc_dnorm_1_ode_padding_h_size * sizeof(double));
    }
//    for(int i = 0; i < y_mcmc_dnorm_n_ode_padding_h_size; i++){
//        if(y_mcmc_dnorm_n_ode_padding_h[i] != 0.0){
//            printf("y_mcmc_dnorm_n_ode_padding_h[%d] = %.1f\n",i,y_mcmc_dnorm_n_ode_padding_h[i]);
//        }
//    }

    //y_mcmc_dnorm_n_ode_padding_d
    double* y_mcmc_dnorm_n_ode_padding_d;
    size_t y_mcmc_dnorm_n_ode_padding_d_size = y_mcmc_dnorm_n_ode_padding_h_size * sizeof(double);
    checkCuda(hipMalloc((void **) &y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_d_size));
    checkCuda(hipMemcpy(y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_h, y_mcmc_dnorm_n_ode_padding_d_size,hipMemcpyHostToDevice));

    double r_denom_h[gpu_params->ode_number];
    memset(r_denom_h,0,gpu_params->ode_number*sizeof(double));
    double r_num_h[gpu_params->ode_number];
    memset(r_num_h,0,gpu_params->ode_number*sizeof(double));

    //gpu_params_d
    GPUParameters *gpu_params_d;
    checkCuda(hipMalloc((void **) &gpu_params_d, sizeof(GPUParameters)));
    checkCuda(hipMemcpy(gpu_params_d, gpu_params, sizeof(GPUParameters), hipMemcpyHostToDevice));
    
    //flu_params_d
    FluParameters **flu_params_d;
    FluParameters** flu_tmp_ptr = (FluParameters **) malloc(ode_size);
    size_t flu_params_d_size = gpu_params->ode_number * sizeof(FluParameters);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMalloc((void **) &flu_tmp_ptr[i], flu_params_d_size));
        checkCuda(hipMemcpy(flu_tmp_ptr[i], flu_params[i], flu_params_d_size,hipMemcpyHostToDevice));
    }
    checkCuda(hipMalloc((void **) &flu_params_d, ode_size));
    checkCuda(hipMemcpy(flu_params_d, flu_tmp_ptr, ode_size, hipMemcpyHostToDevice));

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for allocate mem CPU to GPU: %ld micro seconds which is %.10f seconds\n", duration.count(),
           (duration.count() / 1e6));

    start = std::chrono::high_resolution_clock::now();
    //hipProfilerStart();

    gpu_params->block_size = GPU_ODE_THREADS; //max is 1024
    gpu_params->num_blocks = (gpu_params->ode_output_day + gpu_params->block_size - 1) / gpu_params->block_size;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int num_block = ceil(prop.maxBlocksPerMultiProcessor * prop.maxThreadsPerBlock / 1024);
//    printf("max threads = %d block = %d\n",prop.maxBlocksPerMultiProcessor * prop.maxThreadsPerBlock, num_block);
//    reduce_sum_n<<<num_block, 1024>>>(y_mcmc_dnorm_n_ode_padding_d,y_mcmc_dnorm_n_ode_padding_d,gpu_params->ode_number,y_mcmc_dnorm_n_ode_padding_h_size);
//    checkCuda(hipMemcpy(y_mcmc_dnorm_n_ode_padding_h,y_mcmc_dnorm_n_ode_padding_d,y_mcmc_dnorm_n_ode_padding_d_size,hipMemcpyDeviceToHost));
//    checkCuda(hipDeviceSynchronize());
//    for(int i = 0; i < y_mcmc_dnorm_n_ode_padding_h_size; i++){
//        if(y_mcmc_dnorm_n_ode_padding_h[i] > 1.0){
//            printf("y_mcmc_dnorm_n_ode_padding_h[%d] = %.1f\n",i,y_mcmc_dnorm_n_ode_padding_h[i]);
//        }
//    }
//    calculate_stf<<<gpu_params->num_blocks, gpu_params->block_size>>>(stf_d,gpu_params_d);
//    solve_ode<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_ode_input_d, y_ode_output_d, y_agg_input_d, y_agg_output_d, stf_d, gpu_params_d);;
//    mcmc_dnorm_padding<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_data_input_d, y_ode_agg_d, y_mcmc_dnorm_n_ode_padding_d, mcmc_dnorm_1_ode_padding_size, gpu_params_d);
//    reduce_sum_padding<<<num_block, 1024>>>(y_mcmc_dnorm_n_ode_padding_d,y_mcmc_dnorm_n_ode_padding_d,gpu_params->ode_number,y_mcmc_dnorm_n_ode_padding_h_size);
//    checkCuda(hipMemcpy(y_mcmc_dnorm_n_ode_padding_h,y_mcmc_dnorm_n_ode_padding_d,y_mcmc_dnorm_n_ode_padding_d_size,hipMemcpyDeviceToHost));
//    checkCuda(hipDeviceSynchronize());
//    for(int i = 0; i < y_mcmc_dnorm_n_ode_padding_h_size; i++){
//        if(i % 2 == 0 && y_mcmc_dnorm_n_ode_padding_h[i] != 0.0){
//            printf("y_mcmc_dnorm_n_ode_padding_h[%d] = %.5f\n",i,y_mcmc_dnorm_n_ode_padding_h[i]);
//        }
//    }

    //Pre-calculate stf
    calculate_stf<<<gpu_params->num_blocks, gpu_params->block_size>>>(stf_d, flu_params_d);
    for (int iter = 0; iter < gpu_params->mcmc_loop; iter++) {
//        calculate_stf<<<gpu_params->num_blocks, gpu_params->block_size>>>(stf_d, flu_params_d);
//        solve_ode<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_ode_input_d, y_ode_output_d, y_agg_input_d, y_agg_output_d, stf_d, gpu_params_d, flu_params_d);
//        mcmc_dnorm_padding<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_data_input_d, y_agg_output_d, y_mcmc_dnorm_n_ode_padding_d, mcmc_dnorm_1_ode_padding_size, gpu_params_d);
//        reduce_sum_padding<<<num_block, 1024>>>(y_mcmc_dnorm_n_ode_padding_d,y_mcmc_dnorm_n_ode_padding_d,gpu_params->ode_number,y_mcmc_dnorm_n_ode_padding_h_size);

//        //
//        // Generate new parameters
//        //
//
////        old_params = params;
////        gpu_params->update();
//
//        //
//        // Copy new parameters to gpu
//        //
//
//        checkCuda(hipMemcpy(gpu_params_d, params, sizeof(GPUParameters), hipMemcpyHostToDevice));
//        calculate_stf<<<gpu_params->num_blocks, gpu_params->block_size>>>(stf_d,gpu_params_d);
//
//        //y_ode_input_d
//        //temp pointers
//        tmp_ptr = (double **) malloc(ode_size);
//        for (int i = 0; i < gpu_params->ode_number; i++) {
//            checkCuda(hipMalloc((void **) &tmp_ptr[i], y_ode_input_d_size));
//            checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_ode_input[i], y_ode_input_d_size,hipMemcpyHostToDevice));
//        }
//        checkCuda(hipMalloc((void **) &y_ode_input_d, ode_size));
//        checkCuda(hipMemcpy(y_ode_input_d, tmp_ptr, ode_size, hipMemcpyHostToDevice));//y_ode_output_d
//
//        //y_ode_agg_d
//        //temp pointers
//        tmp_ptr = (double **) malloc(ode_size);
//        for (int i = 0; i < gpu_params->ode_number; i++) {
//            checkCuda(hipMalloc((void **) &tmp_ptr[i], y_ode_agg_d_size));
//            checkCuda(hipMemcpy(tmp_ptr[i], gpu_params->y_ode_agg[i], y_ode_agg_d_size,hipMemcpyHostToDevice));
//        }
//        checkCuda(hipMalloc((void **) &y_ode_agg_d, ode_size));
//        checkCuda(hipMemcpy(y_ode_agg_d, tmp_ptr, ode_size, hipMemcpyHostToDevice));
//
//        //y_mcmc_dnorm_d - single 1d array for NUMODE dnorm values (NUMODE*data_dimension.rows)
//        checkCuda(hipMemcpy(y_mcmc_dnorm_n_ode_padding_d, y_mcmc_dnorm_n_ode_padding_h, y_mcmc_dnorm_n_ode_padding_d_size,hipMemcpyHostToDevice));
//
//        //
//        // Solve ode with new parameters
//        //

//        calculate_stf<<<gpu_params->num_blocks, gpu_params->block_size>>>(stf_d, flu_params_d);
//        solve_ode<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_ode_input_d, y_ode_output_d, y_agg_input_d, y_agg_output_d, stf_d, gpu_params_d, flu_params_d);
//        mcmc_dnorm_padding<<<gpu_params->num_blocks, gpu_params->block_size>>>(y_data_input_d, y_agg_output_d, y_mcmc_dnorm_n_ode_padding_d, mcmc_dnorm_1_ode_padding_size, gpu_params_d);
//        reduce_sum_padding<<<num_block, 1024>>>(y_mcmc_dnorm_n_ode_padding_d,y_mcmc_dnorm_n_ode_padding_d,gpu_params->ode_number,y_mcmc_dnorm_n_ode_padding_h_size);

//        for(int ode_index = 0; ode_index < gpu_params->ode_number; ode_index++) {
//            double r = r_num_h[ode_index] - r_denom_h[ode_index];
//            if (exp(r) > rand_uniform(0.0, 1.0)) {
//                params = old_params;
//                printf("iter %d ODE %d accept params (r = %.5f)\n", iter, ode_index, r);
//            } else {
//                printf("iter %d ODE %d reject params_temp (r = %.5f)\n", iter, ode_index, r);
//            }
//        }
//        printf("==== iter %d done ====\n",iter);
    }

    //    hipProfilerStop();
    checkCuda(hipDeviceSynchronize());
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for MCMC %d ODE with %d parameters %d times on GPU: %ld micro seconds which is %.10f seconds\n",
           NUMODE, DIM, gpu_params->mcmc_loop, duration.count(), (duration.count() / 1e6));

    start = std::chrono::high_resolution_clock::now();
    //y_ode_output_h
    tmp_ptr = (double **) malloc(ode_size);
    double **y_ode_output_h = (double **) malloc(ode_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        y_ode_output_h[i] = (double *) malloc(gpu_params->ode_output_day * gpu_params->display_dimension * sizeof(double));
    }
    checkCuda(hipMemcpy(tmp_ptr, y_ode_output_d, ode_size, hipMemcpyDeviceToHost));
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMemcpy(y_ode_output_h[i], tmp_ptr[i], gpu_params->ode_output_day * gpu_params->display_dimension * sizeof(double),
                             hipMemcpyDeviceToHost));
    }
    //y_output_agg_h
    tmp_ptr = (double **) malloc(ode_size);
    double **y_output_agg_h = (double **) malloc(ode_size);
    for (int i = 0; i < gpu_params->ode_number; i++) {
        y_output_agg_h[i] = (double *) malloc(gpu_params->ode_output_day * gpu_params->agg_dimension * sizeof(double));
    }
    checkCuda(hipMemcpy(tmp_ptr, y_agg_output_d, ode_size, hipMemcpyDeviceToHost));
    for (int i = 0; i < gpu_params->ode_number; i++) {
        checkCuda(hipMemcpy(y_output_agg_h[i], tmp_ptr[i], gpu_params->ode_output_day * gpu_params->agg_dimension * sizeof(double),
                             hipMemcpyDeviceToHost));
    }
    checkCuda(hipDeviceSynchronize());
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for data transfer GPU to CPU: %ld micro seconds which is %.10f seconds\n", duration.count(),
           (duration.count() / 1e6));

    start = std::chrono::high_resolution_clock::now();
    std::random_device rd; // obtain a random number from hardware
    std::mt19937 gen(rd()); // seed the generator
    std::uniform_int_distribution<> distr(0, NUMODE); // define the range

    for (int i = 0; i < gpu_params->display_number; i++) {
        int random_index = 0;
        if (NUMODE == 1) {
            random_index = 0;
        } else {
            random_index = distr(gen);
        }
//        printf("Display y_ode_output_h[%d]\n",random_index);
//        for(int index = 0; index < gpu_params->ode_output_day * gpu_params->display_dimension; index++){
//            const int line_index = (index / gpu_params->display_dimension) % NUMDAYSOUTPUT;
//            if(line_index < 10)
//            {
//                printf("%.5f\t", y_ode_output_h[random_index][index]);
//                if (index > 0 && (index + 1) % gpu_params->display_dimension == 0) {
//                    printf("\n");
//                }
//            }
//        }
        printf("Display y_output_agg_h[%d]\n", random_index);
        for (int index = 0; index < gpu_params->ode_output_day * gpu_params->agg_dimension; index++) {
            const int line_index = (index / gpu_params->agg_dimension) % NUMDAYSOUTPUT;
            if(line_index < 10)
            {
                printf("%d %.5f\t",line_index, y_output_agg_h[random_index][index]);
                if (index > 0 && (index + 1) % gpu_params->agg_dimension == 0) {
                    printf("\n");
                }
            }
        }
    }
    stop = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    printf("[GSL GPU] Time for display random results on CPU: %ld micro seconds which is %.10f seconds\n",
           duration.count(), (duration.count() / 1e6));
    printf("\n");
    // Free memory
    checkCuda(hipFree(y_ode_input_d));
    checkCuda(hipFree(y_ode_output_d));
    checkCuda(hipFree(y_agg_output_d));
    checkCuda(hipFree(y_data_input_d));
    checkCuda(hipFree(y_mcmc_dnorm_n_ode_padding_d));
    checkCuda(hipFree(gpu_params_d));
    checkCuda(hipFree(flu_params_d));
    delete y_ode_output_h;
    delete y_output_agg_h;
    delete tmp_ptr;
    return;
}